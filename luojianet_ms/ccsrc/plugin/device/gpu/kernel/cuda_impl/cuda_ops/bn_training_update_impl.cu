#include "hip/hip_runtime.h"
/**
 * Copyright 2021, 2022 LuoJiaNET Research and Development Group, Wuhan University
 * Copyright 2021, 2022 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <algorithm>
#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/util.cuh"
#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/bn_training_update_impl.cuh"

__inline__ __device__ float HalfFloatInputConvert(const half val) { return __half2float(val); }
__inline__ __device__ float HalfFloatInputConvert(const float val) { return val; }

template <typename T>
__device__ __forceinline__ T AbsFunc(T x) {
  return abs(x);
}

template <typename T>
__global__ void BNTrainingUpdateKernel(size_t N, size_t C, size_t H, size_t W, T *x, T *y, float *sum,
                                       float *square_sum, float *scale, float *offset, float *mean,
                                       float *variance, float factor, float epsilon, float *mean_output,
                                       float *variance_output, float *save_mean_reduce_output,
                                       float *save_variance_reduce_output) {
  __shared__ float num_rec;
  int num = N * C * H * W;
  int normal_size = N * H * W;
  num_rec = HalfFloatInputConvert(1) / normal_size;
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < num; i += blockDim.x * gridDim.x) {
    int channel_index = i / (H * W) % C;
    float save_mean_reduce = sum[channel_index] * num_rec;
    float variance_div = square_sum[channel_index] * num_rec;
    float variance_square = save_mean_reduce * save_mean_reduce;
    float save_variance_reduce = variance_div - variance_square;

    float multiplier_add = save_variance_reduce + epsilon;
    if(multiplier_add < static_cast<float>(0)) {
      printf("multiplier_add < 0 %f!\n",multiplier_add);
    }

    float multiplier_sqrt = sqrtf(AbsFunc(multiplier_add));
    float multiplier_div = scale[channel_index] / multiplier_sqrt;

    float addend_mul = multiplier_div * save_mean_reduce;
    float addend_sub = offset[channel_index] - addend_mul;
    T res_y = (multiplier_div * HalfFloatInputConvert(x[i])) + addend_sub;

    float batch_var_scaler;
    if (num == 1) {
      batch_var_scaler = 0.0;
    } else {
      batch_var_scaler = static_cast<float>(num) / (num - 1);
    }
    float batch_variance = save_variance_reduce * batch_var_scaler;

    float factor_reverse = 1.0 - factor;
    float mean_mul = save_mean_reduce * factor;
    float mean_mul_rev = mean[channel_index] * factor_reverse;
    float mean = mean_mul + mean_mul_rev;

    float var_mul = batch_variance * factor;
    float var_mul_rev = variance[channel_index] * factor_reverse;
    float variance = var_mul + var_mul_rev;

    mean_output[channel_index] = mean;
    variance_output[channel_index] = variance;
    save_mean_reduce_output[channel_index] = save_mean_reduce;
    save_variance_reduce_output[channel_index] = save_variance_reduce;
    y[i] = res_y;
  }
  return;
}

template <typename T>
void BNTrainingUpdate(size_t N, size_t C, size_t H, size_t W, T *x, T *y, float *sum, float *square_sum, float *scale,
                      float *offset, float *mean, float *variance, float factor, float epsilon, float *mean_output,
                      float *variance_output, float *save_mean_reduce_output, float *save_variance_reduce_output,
                      hipStream_t hip_stream) {
  BNTrainingUpdateKernel<<<C, GET_THREADS, 0, hip_stream>>>(N, C, H, W, x, y, sum, square_sum, scale, offset, mean,
                                                             variance, factor, epsilon, mean_output, variance_output,
                                                             save_mean_reduce_output, save_variance_reduce_output);
  return;
}

template CUDA_LIB_EXPORT void BNTrainingUpdate<half>(size_t N, size_t C, size_t H, size_t W, half *x, half *y,
                                                     float *sum, float *square_sum, float *scale, float *offset,
                                                     float *mean, float *variance, float factor, float epsilon,
                                                     float *mean_output, float *variance_output,
                                                     float *save_mean_reduce_output, float *save_variance_reduce_output,
                                                     hipStream_t hip_stream);

template CUDA_LIB_EXPORT void BNTrainingUpdate<float>(size_t N, size_t C, size_t H, size_t W, float *x, float *y,
                                                      float *sum, float *square_sum, float *scale, float *offset,
                                                      float *mean, float *variance, float factor, float epsilon,
                                                      float *mean_output, float *variance_output,
                                                      float *save_mean_reduce_output,
                                                      float *save_variance_reduce_output, hipStream_t hip_stream);
