#include "hip/hip_runtime.h"
/**
 * Copyright 2021, 2022 LuoJiaNET Research and Development Group, Wuhan University
 * Copyright 2021, 2022 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/tensor_scatter_update.cuh"
#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/util.cuh"

template <typename T, typename S>
__global__ void TensorScatterUpdateKernel(T *input, S *indices, T *update, T *output, const size_t block_size,
                                          const size_t input_size, const size_t output_size, const size_t indices_dim_0,
                                          const size_t indices_dim_1, S *indices_stride, S *work_shape) {
  int i, j;
  for (size_t read_index = blockIdx.x * blockDim.x + threadIdx.x; read_index < input_size;
       read_index += blockDim.x * gridDim.x) {
    size_t write_index = 0;
    bool out_bound = false;

    i = read_index / block_size;
    j = read_index % block_size;

    for (size_t k = 0; k < indices_dim_1; k++) {
      S indices_i = indices[i * indices_dim_1 + k];
      out_bound |= indices_i >= work_shape[k];
      write_index += indices_i * indices_stride[k];
    }

    write_index += j;
    out_bound |= write_index >= output_size;

    if (!out_bound) {
      output[write_index] = update[read_index];
    }
  }
}

template <typename T, typename S>
void TensorScatterUpdate(T *input, S *indices, T *update, T *output, const size_t &block_size, const size_t &input_size,
                         const size_t &output_size, const size_t &indices_dim_0, const size_t &indices_dim_1,
                         S *indices_stride, S *work_shape, hipStream_t stream) {
  TensorScatterUpdateKernel<<<GET_BLOCKS(output_size), GET_THREADS, 0, stream>>>(
    input, indices, update, output, block_size, input_size, output_size, indices_dim_0, indices_dim_1, indices_stride,
    work_shape);
  return;
}

template CUDA_LIB_EXPORT void TensorScatterUpdate<half, int>(half *input, int *indices, half *update, half *output,
                                                             const size_t &block_size, const size_t &input_size,
                                                             const size_t &output_size, const size_t &indices_dim_0,
                                                             const size_t &indices_dim_1, int *indices_stride,
                                                             int *work_shape, hipStream_t stream);
template CUDA_LIB_EXPORT void TensorScatterUpdate<float, int>(float *input, int *indices, float *update, float *output,
                                                              const size_t &block_size, const size_t &input_size,
                                                              const size_t &output_size, const size_t &indices_dim_0,
                                                              const size_t &indices_dim_1, int *indices_stride,
                                                              int *work_shape, hipStream_t stream);
template CUDA_LIB_EXPORT void TensorScatterUpdate<double, int>(double *input, int *indices, double *update,
                                                               double *output, const size_t &block_size,
                                                               const size_t &input_size, const size_t &output_size,
                                                               const size_t &indices_dim_0, const size_t &indices_dim_1,
                                                               int *indices_stride, int *work_shape,
                                                               hipStream_t stream);
template CUDA_LIB_EXPORT void TensorScatterUpdate<char, int>(char *input, int *indices, char *update, char *output,
                                                             const size_t &block_size, const size_t &input_size,
                                                             const size_t &output_size, const size_t &indices_dim_0,
                                                             const size_t &indices_dim_1, int *indices_stride,
                                                             int *work_shape, hipStream_t stream);
template CUDA_LIB_EXPORT void TensorScatterUpdate<unsigned char, int>(unsigned char *input, int *indices,
                                                                      unsigned char *update, unsigned char *output,
                                                                      const size_t &block_size,
                                                                      const size_t &input_size,
                                                                      const size_t &output_size,
                                                                      const size_t &indices_dim_0,
                                                                      const size_t &indices_dim_1, int *indices_stride,
                                                                      int *work_shape, hipStream_t stream);
template CUDA_LIB_EXPORT void TensorScatterUpdate<int, int>(int *input, int *indices, int *update, int *output,
                                                            const size_t &block_size, const size_t &input_size,
                                                            const size_t &output_size, const size_t &indices_dim_0,
                                                            const size_t &indices_dim_1, int *indices_stride,
                                                            int *work_shape, hipStream_t stream);
template CUDA_LIB_EXPORT void TensorScatterUpdate<bool, int>(bool *input, int *indices, bool *update, bool *output,
                                                             const size_t &block_size, const size_t &input_size,
                                                             const size_t &output_size, const size_t &indices_dim_0,
                                                             const size_t &indices_dim_1, int *indices_stride,
                                                             int *work_shape, hipStream_t stream);
template CUDA_LIB_EXPORT void TensorScatterUpdate<bool, int64_t>(bool *input, int64_t *indices, bool *update,
                                                                 bool *output, const size_t &block_size,
                                                                 const size_t &input_size, const size_t &output_size,
                                                                 const size_t &indices_dim_0,
                                                                 const size_t &indices_dim_1, int64_t *indices_stride,
                                                                 int64_t *work_shape, hipStream_t stream);
template CUDA_LIB_EXPORT void TensorScatterUpdate<float, int64_t>(float *input, int64_t *indices, float *update,
                                                                  float *output, const size_t &block_size,
                                                                  const size_t &input_size, const size_t &output_size,
                                                                  const size_t &indices_dim_0,
                                                                  const size_t &indices_dim_1, int64_t *indices_stride,
                                                                  int64_t *work_shape, hipStream_t stream);
template CUDA_LIB_EXPORT void TensorScatterUpdate<double, int64_t>(double *input, int64_t *indices, double *update,
                                                                   double *output, const size_t &block_size,
                                                                   const size_t &input_size, const size_t &output_size,
                                                                   const size_t &indices_dim_0,
                                                                   const size_t &indices_dim_1, int64_t *indices_stride,
                                                                   int64_t *work_shape, hipStream_t stream);
