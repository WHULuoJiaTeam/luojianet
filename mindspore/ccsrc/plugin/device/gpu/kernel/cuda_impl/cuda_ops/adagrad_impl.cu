#include "hip/hip_runtime.h"
/**
 * Copyright 2020 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/adagrad_impl.cuh"
#include "include/hip/hip_fp16.h"

template <typename T>
__device__ __forceinline__ T SqrtFunc(T input) {
  return sqrt(input);
}

template <>
__device__ __forceinline__ half SqrtFunc(half input) {
  return hsqrt(input);
}

template <typename T, typename S, typename G>
__global__ void ApplyAdagradKernel(const size_t size,
                                   const bool update_slots,
                                   const S *learning_rate,
                                   const G *gradient,
                                   T *variable,
                                   T *accumulation) {
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < size; i += gridDim.x * blockDim.x) {
    if (update_slots) {
      accumulation[i] += gradient[i] * gradient[i];
    }
    variable[i] -= learning_rate[0] * gradient[i] / SqrtFunc(accumulation[i]);
  }
}

template <>
__global__ void ApplyAdagradKernel(const size_t size,
                                   const bool update_slots,
                                   const float *learning_rate,
                                   const half *gradient,
                                   half *variable,
                                   half *accumulation) {
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < size; i += gridDim.x * blockDim.x) {
    if (update_slots) {
      accumulation[i] += gradient[i] * gradient[i];
    }
    variable[i] -= __float2half(learning_rate[0]) * gradient[i] / SqrtFunc(accumulation[i]);
  }
}

template <>
__global__ void ApplyAdagradKernel(const size_t size,
                                   const bool update_slots,
                                   const float *learning_rate,
                                   const half *gradient,
                                   float *variable,
                                   float *accumulation) {
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < size; i += gridDim.x * blockDim.x) {
    if (update_slots) {
      accumulation[i] += __half2float(gradient[i]) * __half2float(gradient[i]);
    }
    variable[i] -= learning_rate[0] * __half2float(gradient[i]) / SqrtFunc(accumulation[i]);
  }
}

template <>
__global__ void ApplyAdagradKernel(const size_t size,
                                   const bool update_slots,
                                   const half *learning_rate,
                                   const float *gradient,
                                   float *variable,
                                   float *accumulation) {
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < size; i += gridDim.x * blockDim.x) {
    if (update_slots) {
      accumulation[i] += gradient[i] * gradient[i];
    }
    variable[i] -= __half2float(learning_rate[0]) * gradient[i] / SqrtFunc(accumulation[i]);
  }
}

template <>
__global__ void ApplyAdagradKernel(const size_t size,
                                   const bool update_slots,
                                   const float *learning_rate,
                                   const float *gradient,
                                   half *variable,
                                   half *accumulation) {
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < size; i += gridDim.x * blockDim.x) {
    if (update_slots) {
      accumulation[i] += __float2half(gradient[i]) * __float2half(gradient[i]);
    }
    variable[i] -= __float2half(learning_rate[0]) * __float2half(gradient[i]) / SqrtFunc(accumulation[i]);
  }
}

template <typename T, typename S, typename G>
void ApplyAdagrad(const size_t size,
                  const bool update_slots,
                  const S *learning_rate,
                  const G *gradient,
                  T *variable,
                  T *accumulation,
                  hipStream_t hip_stream) {
  ApplyAdagradKernel<<< GET_BLOCKS(size), GET_THREADS, 0, hip_stream>>>(
          size, update_slots, learning_rate, gradient, variable, accumulation);
}

template CUDA_LIB_EXPORT void ApplyAdagrad<float, float, float>(const size_t size,
                                                                const bool update_slots,
                                                                const float *learning_rate,
                                                                const float *gradient,
                                                                float *variable,
                                                                float *accumulation,
                                                                hipStream_t hip_stream);

template CUDA_LIB_EXPORT void ApplyAdagrad<half, half, half>(const size_t size,
                                                             const bool update_slots,
                                                             const half *learning_rate,
                                                             const half *gradient,
                                                             half *variable,
                                                             half *accumulation,
                                                             hipStream_t hip_stream);

template CUDA_LIB_EXPORT void ApplyAdagrad<half, float, half>(const size_t size,
                                                              const bool update_slots,
                                                              const float *learning_rate,
                                                              const half *gradient,
                                                              half *variable,
                                                              half *accumulation,
                                                              hipStream_t hip_stream);

template CUDA_LIB_EXPORT void ApplyAdagrad<float, float, half>(const size_t size,
                                                               const bool update_slots,
                                                               const float *learning_rate,
                                                               const half *gradient,
                                                               float *variable,
                                                               float *accumulation,
                                                               hipStream_t hip_stream);

template CUDA_LIB_EXPORT void ApplyAdagrad<float, half, float>(const size_t size,
                                                               const bool update_slots,
                                                               const half *learning_rate,
                                                               const float *gradient,
                                                               float *variable,
                                                               float *accumulation,
                                                               hipStream_t hip_stream);

template CUDA_LIB_EXPORT void ApplyAdagrad<half, float, float>(const size_t size,
                                                               const bool update_slots,
                                                               const float *learning_rate,
                                                               const float *gradient,
                                                               half *variable,
                                                               half *accumulation,
                                                               hipStream_t hip_stream);
