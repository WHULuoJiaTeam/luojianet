#include "hip/hip_runtime.h"
/**
 * Copyright 2020 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/adam_impl.cuh"
#include "include/hip/hip_fp16.h"

template <typename T>
__device__ __forceinline__ T SqrtFunc(T input) {
  return sqrt(input);
}

template <>
__device__ __forceinline__ half SqrtFunc(half input) {
  return hsqrt(input);
}

template <typename T>
__global__ void ApplyAdamKernel(const size_t size, const T *gradient, const T *beta1_power, const T *beta2_power,
                                const T *learning_rate, const T *beta1, const T *beta2, const T *epsilon, T *variable,
                                T *m, T *v) {
  const T one = static_cast<T>(1.0);
  const T new_learning_rate = learning_rate[0] * SqrtFunc(one - beta2_power[0]) / (one - beta1_power[0]);

  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < size; i += gridDim.x * blockDim.x) {
    m[i] += (gradient[i] - m[i]) * (one - beta1[0]);
    v[i] += (gradient[i] * gradient[i] - v[i]) * (one - beta2[0]);
    variable[i] -= new_learning_rate * m[i] / (SqrtFunc(v[i]) + epsilon[0]);
  }
}

template <typename T>
__global__ void AdamWeightDecayKernel(const size_t size, const T *gradient, const float *learning_rate,
                                      const float *beta1, const float *beta2, const float *epsilon, const float *decay,
                                      T *variable, T *m, T *v) {
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < size; i += gridDim.x * blockDim.x) {
    T next_m = beta1[0] * m[i] + (1 - beta1[0]) * gradient[i];
    T next_v = beta2[0] * v[i] + (1 - beta2[0]) * gradient[i] * gradient[i];
    T update = next_m / (sqrt(next_v) + epsilon[0]);
    update += decay[0] * variable[i];
    variable[i] -= learning_rate[0] * update;
    m[i] = next_m;
    v[i] = next_v;
  }
}
template <>
__global__ void AdamWeightDecayKernel(const size_t size, const half *gradient, const float *learning_rate,
                                      const float *beta1, const float *beta2, const float *epsilon, const float *decay,
                                      half *variable, half *m, half *v) {
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < size; i += gridDim.x * blockDim.x) {
    half next_m = __float2half(beta1[0]) * m[i] + __float2half(1 - beta1[0]) * gradient[i];
    half next_v = __float2half(beta2[0]) * v[i] + __float2half(1 - beta2[0]) * gradient[i] * gradient[i];
    half update = next_m / (hsqrt(next_v) + __float2half(epsilon[0]));
    update += __float2half(decay[0]) * variable[i];
    variable[i] -= __float2half(learning_rate[0]) * update;
    m[i] = next_m;
    v[i] = next_v;
  }
}
template <typename T>
void ApplyAdam(const size_t size, const T *gradient, const T *beta1_power, const T *beta2_power, const T *learning_rate,
               const T *beta1, const T *beta2, const T *epsilon, T *variable, T *m, T *v, hipStream_t hip_stream) {
  ApplyAdamKernel<<<GET_BLOCKS(size), GET_THREADS, 0, hip_stream>>>(
    size, gradient, beta1_power, beta2_power, learning_rate, beta1, beta2, epsilon, variable, m, v);
}
template <typename T>
void AdamWeightDecayOp(const size_t size, const T *gradient, const float *learning_rate, const float *beta1,
                       const float *beta2, const float *epsilon, const float *decay, T *variable, T *m, T *v,
                       hipStream_t hip_stream) {
  AdamWeightDecayKernel<<<GET_BLOCKS(size), GET_THREADS, 0, hip_stream>>>(size, gradient, learning_rate, beta1, beta2,
                                                                           epsilon, decay, variable, m, v);
}

template CUDA_LIB_EXPORT void ApplyAdam<float>(const size_t size, const float *gradient, const float *beta1_power,
                                               const float *beta2_power, const float *learning_rate, const float *beta1,
                                               const float *beta2, const float *epsilon, float *variable, float *m,
                                               float *v, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void ApplyAdam<half>(const size_t size, const half *gradient, const half *beta1_power,
                                              const half *beta2_power, const half *learning_rate, const half *beta1,
                                              const half *beta2, const half *epsilon, half *variable, half *m, half *v,
                                              hipStream_t hip_stream);
template CUDA_LIB_EXPORT void AdamWeightDecayOp<float>(const size_t size, const float *gradient,
                                                       const float *learning_rate, const float *beta1,
                                                       const float *beta2, const float *epsilon, const float *decay,
                                                       float *variable, float *m, float *v, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void AdamWeightDecayOp<half>(const size_t size, const half *gradient,
                                                      const float *learning_rate, const float *beta1,
                                                      const float *beta2, const float *epsilon, const float *decay,
                                                      half *variable, half *m, half *v, hipStream_t hip_stream);
