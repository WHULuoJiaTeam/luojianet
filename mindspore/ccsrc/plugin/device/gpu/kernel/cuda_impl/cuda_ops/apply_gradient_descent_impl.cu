#include "hip/hip_runtime.h"
/**
 * Copyright 2021 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/apply_gradient_descent_impl.cuh"
#include "include/hip/hip_fp16.h"

template <typename T>
__global__ void ApplyGradientDescent(const size_t size, T *var, const T *alpha, const T *delta, T *output) {
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < size; pos += blockDim.x * gridDim.x) {
      const T alpha_value = alpha[0];
      var[pos] -= alpha_value * delta[pos];
      output[pos] = var[pos];
  }
}

template <typename T>
void CalApplyGradientDescent(const size_t &size, T *var, const T *alpha, const T *delta, T *output,
                             hipStream_t hip_stream) {
  ApplyGradientDescent<<<GET_BLOCKS(size), GET_THREADS, 0, hip_stream>>>(size, var, alpha, delta, output);
}

template CUDA_LIB_EXPORT void CalApplyGradientDescent<float>(const size_t &size, float *var, const float *alpha,
                                                             const float *delta, float *output,
                                                             hipStream_t hip_stream);
template CUDA_LIB_EXPORT void CalApplyGradientDescent<half>(const size_t &size, half *var, const half *alpha,
                                                            const half *delta, half *output, hipStream_t hip_stream);
