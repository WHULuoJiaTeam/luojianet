#include "hip/hip_runtime.h"
/**
 * Copyright 2020 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "argmax_impl.cuh"
template <typename T, typename S>
__global__ void Argmax(const T *input, const S bound, const size_t outer_size,
                       const size_t inner_size, S *output) {
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < outer_size * inner_size;
       pos += gridDim.x * blockDim.x) {
    size_t x = pos / inner_size % outer_size;
    size_t y = pos % inner_size;
    S idx = 0;
    size_t input_offset = x * bound * inner_size + 0 * inner_size + y;
    T max_data = input[input_offset];
    for (S i = 1; i < bound; i++) {
      input_offset = x * bound * inner_size + i * inner_size + y;
      auto input_data = input[input_offset];
      idx = input_data > max_data ? i : idx;
      max_data = input_data > max_data ? input_data : max_data;
    }
    output[pos] = idx;
  }
  return;
}

template <typename T, typename S>
void CalArgmax(const T *input, const S bound, const size_t outer_size, const size_t inner_size,
               S *output, hipStream_t hip_stream) {
  Argmax<<<GET_BLOCKS(outer_size), GET_THREADS, 0, hip_stream>>>(input, bound, outer_size, inner_size,
                                                                  output);
  return;
}

template
CUDA_LIB_EXPORT void CalArgmax<float, int>(const float *input, const int bound, const size_t outer_size,
                                    const size_t inner_size, int *output, hipStream_t hip_stream);
template
CUDA_LIB_EXPORT void CalArgmax<half, int>(const half *input, const int bound, const size_t outer_size,
                                   const size_t inner_size, int *output, hipStream_t hip_stream);
