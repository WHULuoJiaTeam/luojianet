#include "hip/hip_runtime.h"
/**
 * Copyright 2020 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <thrust/device_ptr.h>
#include <thrust/fill.h>
#include <thrust/system/cuda/execution_policy.h>
#include "batchnorm_fold_impl.cuh"

template <typename T>
__global__ void UpdateRunningStd(int channel_size, const double epsilon, T* running_std) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < channel_size; i += blockDim.x * gridDim.x) {
    running_std[i] = sqrtf(running_std[i] + epsilon);
  }
  return;
}

template <typename T>
__global__ void UpdateBatchStd(int channel_size, T* batch_std) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < channel_size; i += blockDim.x * gridDim.x) {
    batch_std[i] = 1 / batch_std[i];
  }
  return;
}

template <typename T>
__global__ void CalDx(const T* d_batch_mean, const T* d_batch_std, const T* x, const T* batch_mean, const T* batch_std,
                      int batch_size, int channel_size, int height, int width, T* dx) {
  int n = batch_size * channel_size * height * width;
  int normal_size = batch_size * height * width;
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
    int channel_index = i / (height * width) % channel_size;
    dx[i] = d_batch_mean[channel_index] / normal_size +
            d_batch_std[channel_index] * (x[i] - batch_mean[channel_index]) / batch_std[channel_index] / normal_size;
  }
  return;
}

template <typename T>
void CalUpdateRunningStd(int channel_size, double epsilon, T* running_std, hipStream_t hip_stream) {
  UpdateRunningStd<<<GET_BLOCKS(channel_size), GET_THREADS, 0, hip_stream>>>(channel_size, epsilon, running_std);
  return;
}

template CUDA_LIB_EXPORT void CalUpdateRunningStd<float>(int channel_size, double epsilon, float* running_std,
                                                         hipStream_t hip_stream);

template <typename T>
void CalUpdateBatchStd(int channel_size, T* batch_std, hipStream_t hip_stream) {
  UpdateBatchStd<<<GET_BLOCKS(channel_size), GET_THREADS, 0, hip_stream>>>(channel_size, batch_std);
  return;
}

template CUDA_LIB_EXPORT void CalUpdateBatchStd<float>(int channel_size, float* batch_std, hipStream_t hip_stream);

template <typename T>
void CalBatchNormFoldGrad(const T* d_batch_mean, const T* d_batch_std, const T* x, const T* batch_mean,
                          const T* batch_std, int batch_size, int channel_size, int height, int width, T* dx,
                          hipStream_t hip_stream) {
  CalDx<<<GET_BLOCKS(batch_size * channel_size * height * width), GET_THREADS, 0, hip_stream>>>(
    d_batch_mean, d_batch_std, x, batch_mean, batch_std, batch_size, channel_size, height, width, dx);
}

template CUDA_LIB_EXPORT void CalBatchNormFoldGrad<float>(const float* d_batch_mean, const float* d_batch_std,
                                                          const float* x, const float* batch_mean,
                                                          const float* batch_std, int batch_size, int channel_size,
                                                          int height, int width, float* dx, hipStream_t hip_stream);

template <typename T>
void ThrustFillWith(T* array, int size, T tofill, hipStream_t hip_stream) {
  thrust::device_ptr<T> dev_ptr(array);
  thrust::fill(thrust::cuda::par.on(hip_stream), dev_ptr, dev_ptr + size, tofill);
}

template CUDA_LIB_EXPORT void ThrustFillWith<float>(float* array, int size, float tofill, hipStream_t hip_stream);

