#include "hip/hip_runtime.h"
/**
 * Copyright 2021 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <hip/hip_runtime.h>
#include "batchtospace_impl.cuh"
#include "include/hip/hip_fp16.h"

template <typename T>
__global__ void BatchToSpace(const size_t size, const T *input, const size_t in,
                             const size_t ih, const size_t iw, const size_t ic,
                             const size_t on, const size_t oh, const size_t ow,
                             const size_t oc, const size_t crop_up, const size_t crop_dn,
                             const size_t crop_lft, const size_t crop_rht, const size_t block_num,
                             T *output) {
  size_t temp_stride = 0;
  size_t temp_pos = 0;
  size_t idx_on = 0;
  size_t idx_oc = 0;
  size_t idx_oh = 0;
  size_t idx_ow = 0;
  size_t idx_in = 0;
  size_t input_pos = 0;
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < size;
       pos += blockDim.x * gridDim.x) {
    temp_stride = oc * oh * ow;
    idx_on = pos / temp_stride;
    temp_pos = pos % temp_stride;

    temp_stride /= oc;
    idx_oc = temp_pos / temp_stride;
    temp_pos = pos % temp_stride;

    temp_stride /= oh;
    idx_oh = temp_pos / temp_stride;
    temp_pos = pos % temp_stride;

    temp_stride /= ow;
    idx_ow = temp_pos / temp_stride;

    idx_in = (((idx_oh + crop_up) % block_num) * block_num + ((idx_ow + crop_lft) % block_num)) * on + idx_on;
    input_pos = idx_in * ic;
    input_pos = (input_pos + idx_oc) * ih;
    input_pos = (input_pos + ((idx_oh + crop_up) - (idx_in / (on * block_num))) / block_num) * iw;
    input_pos = (input_pos + ((idx_ow + crop_lft) - ((idx_in / on) % block_num)) / block_num);
    output[pos] = input[input_pos];
  }
  return;
}

template <typename T>
void CalBatchToSpace(const size_t size, const T *input, const size_t in,
  const size_t ih, const size_t iw, const size_t ic,
  const size_t on, const size_t oh, const size_t ow,
  const size_t oc, const size_t crop_up, const size_t crop_dn,
  const size_t crop_lft, const size_t crop_rht, const size_t block_num,
  T *output, hipStream_t hip_stream) {
  BatchToSpace<<<GET_BLOCKS(size), GET_THREADS, 0, hip_stream>>>(
    size, input, in, ih, iw, ic, on, oh, ow, oc, crop_up, crop_dn, crop_lft, crop_rht, block_num, output);
  return;
}

template CUDA_LIB_EXPORT void CalBatchToSpace<float>(const size_t size, const float *input, const size_t in,
                                                     const size_t ih, const size_t iw, const size_t ic,
                                                     const size_t on, const size_t oh, const size_t ow,
                                                     const size_t oc, const size_t crop_up, const size_t crop_dn,
                                                     const size_t crop_lft, const size_t crop_rht,
                                                     const size_t block_num, float *output, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void CalBatchToSpace<half>(const size_t size, const half *input, const size_t in,
                                                    const size_t ih, const size_t iw, const size_t ic,
                                                    const size_t on, const size_t oh, const size_t ow,
                                                    const size_t oc, const size_t crop_up, const size_t crop_dn,
                                                    const size_t crop_lft, const size_t crop_rht,
                                                    const size_t block_num, half *output, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void CalBatchToSpace<int>(const size_t size, const int *input, const size_t in,
                                                   const size_t ih, const size_t iw, const size_t ic,
                                                   const size_t on, const size_t oh, const size_t ow,
                                                   const size_t oc, const size_t crop_up, const size_t crop_dn,
                                                   const size_t crop_lft, const size_t crop_rht, const size_t block_num,
                                                   int *output, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void CalBatchToSpace<int64_t>(const size_t size, const int64_t *input, const size_t in,
                                                       const size_t ih, const size_t iw, const size_t ic,
                                                       const size_t on, const size_t oh, const size_t ow,
                                                       const size_t oc, const size_t crop_up, const size_t crop_dn,
                                                       const size_t crop_lft, const size_t crop_rht,
                                                       const size_t block_num, int64_t *output,
                                                       hipStream_t hip_stream);
template CUDA_LIB_EXPORT void CalBatchToSpace<int16_t>(const size_t size, const int16_t *input, const size_t in,
                                                       const size_t ih, const size_t iw, const size_t ic,
                                                       const size_t on, const size_t oh, const size_t ow,
                                                       const size_t oc, const size_t crop_up, const size_t crop_dn,
                                                       const size_t crop_lft, const size_t crop_rht,
                                                       const size_t block_num, int16_t *output,
                                                       hipStream_t hip_stream);
template CUDA_LIB_EXPORT void CalBatchToSpace<int8_t>(const size_t size, const int8_t *input, const size_t in,
                                                      const size_t ih, const size_t iw, const size_t ic,
                                                      const size_t on, const size_t oh, const size_t ow,
                                                      const size_t oc, const size_t crop_up, const size_t crop_dn,
                                                      const size_t crop_lft, const size_t crop_rht,
                                                      const size_t block_num, int8_t *output, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void CalBatchToSpace<uint8_t>(const size_t size, const uint8_t *input, const size_t in,
                                                       const size_t ih, const size_t iw, const size_t ic,
                                                       const size_t on, const size_t oh, const size_t ow,
                                                       const size_t oc, const size_t crop_up, const size_t crop_dn,
                                                       const size_t crop_lft, const size_t crop_rht,
                                                       const size_t block_num, uint8_t *output,
                                                       hipStream_t hip_stream);
template CUDA_LIB_EXPORT void CalBatchToSpace<uint16_t>(const size_t size, const uint16_t *input, const size_t in,
                                                        const size_t ih, const size_t iw, const size_t ic,
                                                        const size_t on, const size_t oh, const size_t ow,
                                                        const size_t oc, const size_t crop_up, const size_t crop_dn,
                                                        const size_t crop_lft, const size_t crop_rht,
                                                        const size_t block_num, uint16_t *output,
                                                        hipStream_t hip_stream);
template CUDA_LIB_EXPORT void CalBatchToSpace<uint32_t>(const size_t size, const uint32_t *input, const size_t in,
                                                        const size_t ih, const size_t iw, const size_t ic,
                                                        const size_t on, const size_t oh, const size_t ow,
                                                        const size_t oc, const size_t crop_up, const size_t crop_dn,
                                                        const size_t crop_lft, const size_t crop_rht,
                                                        const size_t block_num, uint32_t *output,
                                                        hipStream_t hip_stream);
template CUDA_LIB_EXPORT void CalBatchToSpace<uint64_t>(const size_t size, const uint64_t *input, const size_t in,
                                                        const size_t ih, const size_t iw, const size_t ic,
                                                        const size_t on, const size_t oh, const size_t ow,
                                                        const size_t oc, const size_t crop_up, const size_t crop_dn,
                                                        const size_t crop_lft, const size_t crop_rht,
                                                        const size_t block_num, uint64_t *output,
                                                        hipStream_t hip_stream);
