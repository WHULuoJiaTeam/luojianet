#include "hip/hip_runtime.h"
/**
 * Copyright 2020-2021 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/broadcast_grad_impl.cuh"
#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/util.cuh"

template <typename T>
struct MinimumGradFunc {
  __device__ __forceinline__ void operator()(const T &x1, const T &x2, const bool &grad_x1, const bool &grad_x2,
                                             const T &dy, T *dx1, T *dx2) {
    if (grad_x1 && x1 < x2) {
      MsAtomicAdd(dx1, dy);
    } else if (grad_x2 && x1 >= x2) {
      MsAtomicAdd(dx2, dy);
    }
  }
};

template <typename T>
struct MaximumGradFunc {
  __device__ __forceinline__ void operator()(const T &x1, const T &x2, const bool &grad_x1, const bool &grad_x2,
                                             const T &dy, T *dx1, T *dx2) {
    if (grad_x1 && x1 > x2) {
      MsAtomicAdd(dx1, dy);
    } else if (grad_x2 && x1 <= x2) {
      MsAtomicAdd(dx2, dy);
    }
  }
};

__device__ __forceinline__ int Index(const int &index, const int &dim) { return dim == 1 ? 0 : index; }

template <typename T, typename Func>
__device__ __forceinline__ void BroadcastGradOperator(const int &l0, const int &l1, const int &l2, const int &l3,
                                                      const int &r0, const int &r1, const int &r2, const int &r3,
                                                      const int &d0, const int &d1, const int &d2, const int &d3,
                                                      const bool &grad_x1, const bool &grad_x2, const T *x1,
                                                      const T *x2, const T *dy, T *dx1, T *dx2) {
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < d0 * d1 * d2 * d3; pos += blockDim.x * gridDim.x) {
    int i = pos / (d1 * d2 * d3) % d0;
    int j = pos / (d2 * d3) % d1;
    int k = pos / d3 % d2;
    int l = pos % d3;

    int l_index = Index(i, l0) * l1 * l2 * l3 + Index(j, l1) * l2 * l3 + Index(k, l2) * l3 + Index(l, l3);
    int r_index = Index(i, r0) * r1 * r2 * r3 + Index(j, r1) * r2 * r3 + Index(k, r2) * r3 + Index(l, r3);
    Func()(x1[l_index], x2[r_index], grad_x1, grad_x2, dy[pos], dx1 + l_index, dx2 + r_index);
  }
}

template <typename T>
__global__ void BroadcastGradKernel(const int l0, const int l1, const int l2, const int l3, const int r0, const int r1,
                                    const int r2, const int r3, const int d0, const int d1, const int d2, const int d3,
                                    const bool grad_x1, const bool grad_x2, enum BroadcastGradOpType op, const T *x1,
                                    const T *x2, const T *dy, T *dx1, T *dx2) {
  switch (op) {
    case BROADCAST_GRAD_TYPE_MINIMUM:
      return BroadcastGradOperator<T, MinimumGradFunc<T>>(l0, l1, l2, l3, r0, r1, r2, r3, d0, d1, d2, d3, grad_x1,
                                                          grad_x2, x1, x2, dy, dx1, dx2);
    case BROADCAST_GRAD_TYPE_MAXIMUM:
      return BroadcastGradOperator<T, MaximumGradFunc<T>>(l0, l1, l2, l3, r0, r1, r2, r3, d0, d1, d2, d3, grad_x1,
                                                          grad_x2, x1, x2, dy, dx1, dx2);
  }
}

template <typename T>
void BroadcastGrad(const int &l0, const int &l1, const int &l2, const int &l3, const int &r0, const int &r1,
                   const int &r2, const int &r3, const int &d0, const int &d1, const int &d2, const int &d3,
                   const bool &grad_x1, const bool &grad_x2, enum BroadcastGradOpType op, const T *x1, const T *x2,
                   const T *dy, T *dx1, T *dx2, hipStream_t stream) {
  int size = d0 * d1 * d2 * d3;
  BroadcastGradKernel<<<GET_BLOCKS(size), GET_THREADS, 0, stream>>>(l0, l1, l2, l3, r0, r1, r2, r3, d0, d1, d2, d3,
                                                                    grad_x1, grad_x2, op, x1, x2, dy, dx1, dx2);
}

template <typename T, typename Func>
__device__ __forceinline__ void NoBroadcastOperator(const int &nums, const bool &grad_x1, const bool &grad_x2,
                                                    const T *x1, const T *x2, const T *dy, T *dx1, T *dx2) {
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < nums; pos += blockDim.x * gridDim.x) {
    Func()(x1[pos], x2[pos], grad_x1, grad_x2, dy[pos], dx1 + pos, dx2 + pos);
  }
}

template <typename T>
__global__ void NoBroadcastGradKernel(const int nums, const bool grad_x1, const bool grad_x2,
                                      enum BroadcastGradOpType op, const T *x1, const T *x2, const T *dy, T *dx1,
                                      T *dx2) {
  switch (op) {
    case BROADCAST_GRAD_TYPE_MINIMUM:
      return NoBroadcastOperator<T, MinimumGradFunc<T>>(nums, grad_x1, grad_x2, x1, x2, dy, dx1, dx2);
    case BROADCAST_GRAD_TYPE_MAXIMUM:
      return NoBroadcastOperator<T, MaximumGradFunc<T>>(nums, grad_x1, grad_x2, x1, x2, dy, dx1, dx2);
  }
}

template <typename T>
void NoBroadcastGrad(const int &nums, const bool &grad_x1, const bool &grad_x2, enum BroadcastGradOpType op,
                     const T *x1, const T *x2, const T *dy, T *dx1, T *dx2, hipStream_t stream) {
  NoBroadcastGradKernel<<<GET_BLOCKS(nums), GET_THREADS, 0, stream>>>(nums, grad_x1, grad_x2, op, x1, x2, dy, dx1, dx2);
}

template CUDA_LIB_EXPORT void NoBroadcastGrad(const int &nums, const bool &grad_x1, const bool &grad_x2,
                                              enum BroadcastGradOpType op, const double *x1, const double *x2,
                                              const double *dy, double *dx1, double *dx2, hipStream_t stream);
template CUDA_LIB_EXPORT void NoBroadcastGrad(const int &nums, const bool &grad_x1, const bool &grad_x2,
                                              enum BroadcastGradOpType op, const float *x1, const float *x2,
                                              const float *dy, float *dx1, float *dx2, hipStream_t stream);
template CUDA_LIB_EXPORT void NoBroadcastGrad(const int &nums, const bool &grad_x1, const bool &grad_x2,
                                              enum BroadcastGradOpType op, const int *x1, const int *x2,
                                              const int *dy, int *dx1, int *dx2, hipStream_t stream);
template CUDA_LIB_EXPORT void NoBroadcastGrad(const int &nums, const bool &grad_x1, const bool &grad_x2,
                                              enum BroadcastGradOpType op, const half *x1, const half *x2,
                                              const half *dy, half *dx1, half *dx2, hipStream_t stream);
template CUDA_LIB_EXPORT void NoBroadcastGrad(const int &nums, const bool &grad_x1, const bool &grad_x2,
                                              enum BroadcastGradOpType op, const int64_t *x1, const int64_t *x2,
                                              const int64_t *dy, int64_t *dx1, int64_t *dx2, hipStream_t stream);
template CUDA_LIB_EXPORT void BroadcastGrad(const int &l0, const int &l1, const int &l2, const int &l3,
                                            const int &r0, const int &r1, const int &r2, const int &r3,
                                            const int &d0, const int &d1, const int &d2, const int &d3,
                                            const bool &grad_x1, const bool &grad_x2, enum BroadcastGradOpType op,
                                            const double *x1, const double *x2, const double *dy,
                                            double *dx1, double *dx2, hipStream_t stream);
template CUDA_LIB_EXPORT void BroadcastGrad(const int &l0, const int &l1, const int &l2, const int &l3,
                                            const int &r0, const int &r1, const int &r2, const int &r3,
                                            const int &d0, const int &d1, const int &d2, const int &d3,
                                            const bool &grad_x1, const bool &grad_x2, enum BroadcastGradOpType op,
                                            const float *x1, const float *x2, const float *dy, float *dx1, float *dx2,
                                            hipStream_t stream);
template CUDA_LIB_EXPORT void BroadcastGrad(const int &l0, const int &l1, const int &l2, const int &l3,
                                            const int &r0, const int &r1, const int &r2, const int &r3,
                                            const int &d0, const int &d1, const int &d2, const int &d3,
                                            const bool &grad_x1, const bool &grad_x2, enum BroadcastGradOpType op,
                                            const int *x1, const int *x2, const int *dy, int *dx1, int *dx2,
                                            hipStream_t stream);
template CUDA_LIB_EXPORT void BroadcastGrad(const int &l0, const int &l1, const int &l2, const int &l3,
                                            const int &r0, const int &r1, const int &r2, const int &r3,
                                            const int &d0, const int &d1, const int &d2, const int &d3,
                                            const bool &grad_x1, const bool &grad_x2, enum BroadcastGradOpType op,
                                            const half *x1, const half *x2, const half *dy, half *dx1, half *dx2,
                                            hipStream_t stream);
template CUDA_LIB_EXPORT void BroadcastGrad(const int &l0, const int &l1, const int &l2, const int &l3,
                                            const int &r0, const int &r1, const int &r2, const int &r3,
                                            const int &d0, const int &d1, const int &d2, const int &d3,
                                            const bool &grad_x1, const bool &grad_x2, enum BroadcastGradOpType op,
                                            const int64_t *x1, const int64_t *x2, const int64_t *dy,
                                            int64_t *dx1, int64_t *dx2, hipStream_t stream);
