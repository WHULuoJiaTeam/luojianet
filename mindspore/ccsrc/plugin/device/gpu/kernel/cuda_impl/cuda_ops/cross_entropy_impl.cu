#include "hip/hip_runtime.h"
/**
 * Copyright 2019-2021 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <math.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include "cross_entropy_impl.cuh"
#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/util.cuh"

template <typename T, typename S>
__global__ void CrossEntropyWithSparseKernel(const T *logits, const S *labels, const size_t batch_size,
                                             const size_t class_num, T *loss) {
  double total_loss = 0.0;
  T epsilon = 1e-6;
  for (size_t i = 0; i < batch_size; ++i) {
    T logit = logits[i * class_num + labels[i]];
    if (logit <= 0) {
      logit = epsilon;
    }
    total_loss += -logf(logit);
  }
  total_loss /= batch_size;
  loss[0] = static_cast<T>(total_loss);
}

template <typename T, typename S>
__global__ void LargeBatchCrossEntropyWithSparseKernel(const T *logits, const S *labels, const size_t batch_size,
                                                       const size_t class_num, T *loss) {
  *loss = 0;
  T epsilon = 1e-6;
  for (size_t index = blockIdx.x * blockDim.x + threadIdx.x; index < batch_size; index += blockDim.x * gridDim.x) {
    T logit = logits[index * class_num + labels[index]];
    if (logit <= 0) {
      logit = epsilon;
    }
    MsAtomicAdd(loss, -logf(logit) / batch_size);
  }
}

template <typename T, typename S>
__global__ void CrossEntropyGradWithSparseKernel(const T *logits, const S *labels, const size_t batch_size,
                                                 const size_t class_num, T *grad) {
  for (size_t i = 0; i < class_num; i++) {
    for (size_t j = blockIdx.x * blockDim.x + threadIdx.x; j < batch_size; j += blockDim.x * gridDim.x) {
      if (labels[j] == i) {
        grad[j * class_num + i] = (logits[j * class_num + i] - 1) / batch_size;
      } else {
        grad[j * class_num + i] = logits[j * class_num + i] / batch_size;
      }
    }
  }
}

template <typename T, typename S>
__global__ void CrossEntropyKernel(const T *logits, const S *labels, const size_t batch_size, const size_t class_num,
                                   T epsilon, T *losses, T *dlogits) {
  for (size_t index = blockIdx.x * blockDim.x + threadIdx.x; index < batch_size; index += blockDim.x * gridDim.x) {
    losses[index] = 0;
    const int start = index * class_num;
    const int end = (index + 1) * class_num;
    for (int i = start; i < end; ++i) {
      losses[index] -= logf((logits[i] <= 0 ? epsilon : logits[i])) * labels[i];
      dlogits[i] = logits[i] - labels[i];
    }
  }
}

template <typename T, typename S>
void CrossEntropyWithSparse(const T *logits, const S *labels, const size_t batch_size, const size_t class_num, T *loss,
                            hipStream_t hip_stream) {
  if (batch_size <= kLargeBatchLowLimit) {
    CrossEntropyWithSparseKernel<<<1, 1, 0, hip_stream>>>(logits, labels, batch_size, class_num, loss);
  } else {
    LargeBatchCrossEntropyWithSparseKernel<<<GET_BLOCKS(batch_size), GET_THREADS, 0, hip_stream>>>(
      logits, labels, batch_size, class_num, loss);
  }
}

template <typename T, typename S>
void CrossEntropyGradWithSparse(const T *logits, const S *labels, const size_t batch_size, const size_t class_num,
                                T *grad, hipStream_t hip_stream) {
  CrossEntropyGradWithSparseKernel<<<GET_BLOCKS(batch_size), GET_THREADS, 0, hip_stream>>>(logits, labels, batch_size,
                                                                                            class_num, grad);
}

template <typename T, typename S>
void CrossEntropy(const T *logits, const S *labels, const size_t batch_size, const size_t class_num, T *losses,
                  T *dlogits, hipStream_t hip_stream) {
  T epsilon = 1e-6;
  CrossEntropyKernel<<<GET_BLOCKS(batch_size), GET_THREADS, 0, hip_stream>>>(logits, labels, batch_size, class_num,
                                                                              epsilon, losses, dlogits);
}

template CUDA_LIB_EXPORT void CrossEntropyWithSparse<float, int>(const float *logits, const int *labels,
                                                                 const size_t batch_size, const size_t class_num,
                                                                 float *loss, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void CrossEntropyWithSparse<float, int64_t>(const float *logits, const int64_t *labels,
                                                                     const size_t batch_size, const size_t class_num,
                                                                     float *loss, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void CrossEntropyGradWithSparse<float, int>(const float *logits, const int *labels,
                                                                     const size_t batch_size, const size_t class_num,
                                                                     float *grad, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void CrossEntropyGradWithSparse<float, int64_t>(const float *logits, const int64_t *labels,
                                                                         const size_t batch_size,
                                                                         const size_t class_num, float *grad,
                                                                         hipStream_t hip_stream);
template CUDA_LIB_EXPORT void CrossEntropy<float, float>(const float *logits, const float *labels,
                                                         const size_t batch_size, const size_t class_num, float *losses,
                                                         float *dlogits, hipStream_t hip_stream);
