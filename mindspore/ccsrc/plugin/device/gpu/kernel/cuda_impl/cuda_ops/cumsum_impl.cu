#include "hip/hip_runtime.h"
/**
 * Copyright 2020-2021 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "cumsum_impl.cuh"
#include "include/hip/hip_fp16.h"

template <typename T>
__global__ void Copy(T *input, T *output, size_t size) {
  size_t step = blockDim.x * gridDim.x;
  for (size_t write_index = blockIdx.x * blockDim.x + threadIdx.x; write_index < size; write_index += step) {
    input[write_index] = output[write_index];
  }
}

template <typename T>
__global__ void LeftMoveSum(const T *input, T *output, size_t dim0, size_t dim1, size_t dim2, size_t stride,
                         size_t stride2) {
  size_t num = dim0 * dim2;
  size_t i, k, offset;
  size_t step = blockDim.x * gridDim.x;
  for (size_t write_index = blockIdx.x * blockDim.x + threadIdx.x; write_index < num; write_index += step) {
    i = write_index / dim2 % dim0;
    k = write_index % dim2;
    offset = i * stride + k;
    for (size_t j = 0; j < dim1; ++j) {
      size_t read_index = j * stride2 + offset;
      if (j == 0) {
        output[read_index] = 0;
      } else {
        size_t read_index2 = (j - 1) * stride2 + offset;
        output[read_index] = input[read_index2];
      }
    }
  }
}

template <typename T>
__global__ void RightMoveSum(const T *input, T *output, size_t dim0, size_t dim1, size_t dim2, size_t stride,
                          size_t stride2) {
  size_t num = dim0 * dim2;
  size_t i, k, offset;
  size_t step = blockDim.x * gridDim.x;
  for (size_t write_index = blockIdx.x * blockDim.x + threadIdx.x; write_index < num; write_index += step) {
    i = write_index / dim2 % dim0;
    k = write_index % dim2;
    offset = i * stride + k;
    for (int j = dim1 - 1; j >= 0; --j) {
      size_t read_index = j * stride2 + offset;
      if (j == dim1 - 1) {
        output[read_index] = 0;
      } else {
        size_t read_index2 = (j + 1) * stride2 + offset;
        output[read_index] = input[read_index2];
      }
    }
  }
}
template <typename T>
__global__ void CumSumKernelReverse(const T *input, T *output, size_t dim0, size_t dim1, size_t dim2, size_t stride,
                                    size_t stride2) {
  size_t num = dim0 * dim2;
  size_t i, k, offset;
  size_t step = blockDim.x * gridDim.x;
  for (size_t write_index = blockIdx.x * blockDim.x + threadIdx.x; write_index < num; write_index += step) {
    i = write_index / dim2 % dim0;
    k = write_index % dim2;
    offset = i * stride + k;
    for (int j = dim1 - 1; j >= 0; --j) {
      size_t read_index = j * stride2 + offset;
      if (j == dim1 - 1) {
        output[read_index] = input[read_index];
      } else {
        size_t read_index2 = (j + 1) * stride2 + offset;
        output[read_index] = output[read_index2] + input[read_index];
      }
    }
  }
}

template <typename T>
__global__ void CumSumKernel(const T *input, T *output, size_t dim0, size_t dim1, size_t dim2, size_t stride,
                             size_t stride2) {
  size_t num = dim0 * dim2;
  size_t i, k, offset;
  size_t step = blockDim.x * gridDim.x;
  for (size_t write_index = blockIdx.x * blockDim.x + threadIdx.x; write_index < num; write_index += step) {
    i = write_index / dim2 % dim0;
    k = write_index % dim2;
    offset = i * stride + k;
    for (size_t j = 0; j < dim1; ++j) {
      size_t read_index = j * stride2 + offset;
      if (j == 0) {
        output[read_index] = input[read_index];
      } else {
        size_t read_index2 = (j - 1) * stride2 + offset;
        output[read_index] = output[read_index2] + input[read_index];
      }
    }
  }
}
template <typename T>
void CumSum(const T *input, T *output, T *workspace, size_t dim0, size_t dim1, size_t dim2, size_t stride,
            size_t stride2, bool exclusive_, bool reverse_, hipStream_t stream) {
  int size = dim0 * dim2;
  if (exclusive_) {
    if (reverse_) {
      RightMoveSum<<<GET_BLOCKS(size), GET_THREADS, 0, stream>>>(input, output, dim0, dim1, dim2, stride, stride2);
      Copy<<<GET_BLOCKS(size * dim1), GET_THREADS, 0, stream>>>(workspace, output, size * dim1);
      CumSumKernelReverse<<<GET_BLOCKS(size), GET_THREADS, 0, stream>>>(workspace, output, dim0, dim1, dim2, stride,
                                                                        stride2);
    } else {
      LeftMoveSum<<<GET_BLOCKS(size), GET_THREADS, 0, stream>>>(input, output, dim0, dim1, dim2, stride, stride2);
      Copy<<<GET_BLOCKS(size * dim1), GET_THREADS, 0, stream>>>(workspace, output, size * dim1);
      CumSumKernel<<<GET_BLOCKS(size), GET_THREADS, 0, stream>>>(workspace, output, dim0, dim1, dim2, stride, stride2);
    }
  } else {
    if (reverse_) {
      CumSumKernelReverse<<<GET_BLOCKS(size), GET_THREADS, 0, stream>>>(input, output, dim0, dim1, dim2, stride,
                                                                        stride2);
    } else {
      CumSumKernel<<<GET_BLOCKS(size), GET_THREADS, 0, stream>>>(input, output, dim0, dim1, dim2, stride, stride2);
    }
  }
  return;
}

template CUDA_LIB_EXPORT void CumSum<uint8_t>(const uint8_t *input, uint8_t *output, uint8_t *workspace,
                                              size_t dim0, size_t dim1, size_t dim2, size_t stride, size_t stride2,
                                              bool exclusive_, bool reverse_, hipStream_t stream);
template CUDA_LIB_EXPORT void CumSum<int8_t>(const int8_t *input, int8_t *output, int8_t *workspace,
                                             size_t dim0, size_t dim1, size_t dim2, size_t stride, size_t stride2,
                                             bool exclusive_, bool reverse_, hipStream_t stream);
template CUDA_LIB_EXPORT void CumSum<int32_t>(const int32_t *input, int32_t *output, int32_t *workspace,
                                              size_t dim0, size_t dim1, size_t dim2, size_t stride, size_t stride2,
                                              bool exclusive_, bool reverse_, hipStream_t stream);
template CUDA_LIB_EXPORT void CumSum<double>(const double *input, double *output, double *workspace,
                                             size_t dim0, size_t dim1, size_t dim2, size_t stride, size_t stride2,
                                             bool exclusive_, bool reverse_, hipStream_t stream);
template CUDA_LIB_EXPORT void CumSum<float>(const float *input, float *output, float *workspace,
                                            size_t dim0, size_t dim1, size_t dim2, size_t stride, size_t stride2,
                                            bool exclusive_, bool reverse_, hipStream_t stream);
template CUDA_LIB_EXPORT void CumSum<half>(const half *input, half *output, half *workspace, size_t dim0, size_t dim1,
                                           size_t dim2, size_t stride, size_t stride2, bool exclusive_, bool reverse_,
                                           hipStream_t stream);
