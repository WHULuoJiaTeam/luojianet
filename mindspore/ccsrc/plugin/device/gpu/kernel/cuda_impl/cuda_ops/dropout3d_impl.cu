#include "hip/hip_runtime.h"
/**
 * Copyright 2021 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <stdint.h>
#include "dropout3d_impl.cuh"
#include "include/hip/hip_runtime.h"
#include "include/hip/hip_fp16.h"

template <typename T>
__global__ void Dropout3DForwardKernel(const T *input, bool *mask, T *output, float *rand_f, const size_t num_count,
                                       const float keep_prob, const float scale, const size_t num_per_chan) {
  size_t chan_idx;
  float drop_f;  // used in output calculations. Either 0.0 or 1.0.
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < num_count; i += blockDim.x * gridDim.x) {
    chan_idx = i / num_per_chan;  // get channel index over all samples

    drop_f = rand_f[chan_idx] <= keep_prob;
    output[i] = static_cast<T>(scale * input[i] * drop_f);
    mask[i] = static_cast<bool>(drop_f);
  }
}

template <>
__global__ void Dropout3DForwardKernel(const half *input, bool *mask, half *output, float *rand_f,
                                       const size_t num_count, const float keep_prob, const float scale,
                                       const size_t num_per_chan) {
  size_t chan_idx;
  float drop_f;   // used in output calculations. Acts as a single float mask (either 0.0 or 1.0).
  float input_f;  // used to temporarily convert input to float for calculations

  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < num_count; i += blockDim.x * gridDim.x) {
    chan_idx = i / num_per_chan;  // get channel index over all samples

    input_f = __half2float(input[i]);
    drop_f = rand_f[chan_idx] <= keep_prob;
    output[i] = __float2half(scale * input_f * drop_f);  // convert to half
    mask[i] = static_cast<bool>(drop_f);
  }
}

template <typename T>
void Dropout3DForward(const T *input, bool *mask, T *output, float *rand_f, const size_t num_count,
                      const float keep_prob, const size_t num_per_chan, hipStream_t hip_stream) {
  const float scale = 1.f / keep_prob;  // used to scale output, maintains expected value during training
  Dropout3DForwardKernel<<<GET_BLOCKS(num_count), GET_THREADS, 0, hip_stream>>>(input, mask, output, rand_f, num_count,
                                                                                 keep_prob, scale, num_per_chan);
}

template CUDA_LIB_EXPORT void Dropout3DForward<float>(const float *input, bool *mask, float *output, float *rand_f,
                                                      const size_t num_count, const float keep_prob,
                                                      const size_t num_per_chan, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void Dropout3DForward<half>(const half *input, bool *mask, half *output, float *rand_f,
                                                     const size_t num_count, const float keep_prob,
                                                     const size_t num_per_chan, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void Dropout3DForward<int8_t>(const int8_t *input, bool *mask, int8_t *output, float *rand_f,
                                                       const size_t num_count, const float keep_prob,
                                                       const size_t num_per_chan, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void Dropout3DForward<int16_t>(const int16_t *input, bool *mask, int16_t *output,
                                                        float *rand_f, const size_t num_count, const float keep_prob,
                                                        const size_t num_per_chan, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void Dropout3DForward<int32_t>(const int32_t *input, bool *mask, int32_t *output,
                                                        float *rand_f, const size_t num_count, const float keep_prob,
                                                        const size_t num_per_chan, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void Dropout3DForward<int64_t>(const int64_t *input, bool *mask, int64_t *output,
                                                        float *rand_f, const size_t num_count, const float keep_prob,
                                                        const size_t num_per_chan, hipStream_t hip_stream);
