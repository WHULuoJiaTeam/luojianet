#include "hip/hip_runtime.h"
/**
 * Copyright 2020-2021 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "dynamic_range_impl.cuh"
#include <hip/hip_runtime.h>

template <typename T>
__global__ void ValidateInputAndInferShape(const T *range_start, const T *range_end, const T *range_delta,
                                           int64_t *output_shape, DynamicRangeErrorCode *error_code,
                                           const int64_t max_output_size) {
  T start = range_start[0];
  T end = range_end[0];
  T delta = range_delta[0];
  *error_code = DynamicRangeErrorCode::kOk;

  if (delta == 0) {
    *error_code = DynamicRangeErrorCode::kDeltaIsZero;
    return;
  }

  if (start < end && delta < 0) {
    *error_code = DynamicRangeErrorCode::kInvalidNegativeDelta;
    return;
  }

  if (start > end && delta > 0) {
    *error_code = DynamicRangeErrorCode::kInvalidPositiveDelta;
    return;
  }

  if (*error_code == DynamicRangeErrorCode::kOk) {
    int64_t real_output_shape = static_cast<int64_t>(ceil(static_cast<double>(end - start) / delta));

    // verification in case of precision error during calculation of real_output_shape. one multiplication followed by
    // one addition is much more precise than the division that occurs when calculating real_output_shape.
    double last_value = start + (delta * (real_output_shape - 1));
    double epsilon = 1e-6;
    if ((end > start && last_value > end) || (start > end && last_value < end) || fabsf(last_value - end) < epsilon) {
      real_output_shape--;
    }

    if (real_output_shape > max_output_size) {
        *error_code = DynamicRangeErrorCode::kMaxSizeExceeded;
    }
    *output_shape = real_output_shape;
  }
}

template <typename T>
__global__ void Range(const T *range_start, const T *range_end, const T *range_delta, T *output, int64_t *output_shape,
                      const int64_t max_output_size) {
  T start = range_start[0];
  T delta = range_delta[0];

  size_t gt_id = blockIdx.x * blockDim.x + threadIdx.x;
  for (; gt_id < *output_shape; gt_id += blockDim.x * gridDim.x) {
    output[gt_id] = gt_id * delta + start;
  }
}

template <typename T>
void CudaValidateInputAndInferShape(const T *range_start, const T *range_end, const T *range_delta,
                                    int64_t *output_shape, DynamicRangeErrorCode *error_code,
                                    const int64_t max_output_size, hipStream_t hip_stream) {
  ValidateInputAndInferShape<<<1, 1, 0, hip_stream>>>(range_start, range_end, range_delta, output_shape, error_code,
                                                       max_output_size);
}

template <typename T>
void CalRange(const T *range_start, const T *range_end, const T *range_delta, T *output, int64_t *output_shape,
              DynamicRangeErrorCode *error_code, const int64_t max_output_size, hipStream_t hip_stream) {
  Range<<<GET_BLOCKS(max_output_size), GET_THREADS, 0, hip_stream>>>(range_start, range_end, range_delta,
                                                                             output, output_shape, max_output_size);
}

template CUDA_LIB_EXPORT void CudaValidateInputAndInferShape<int>(const int *range_start, const int *range_end,
                                                                  const int *range_delta, int64_t *output_shape,
                                                                  DynamicRangeErrorCode *error_code,
                                                                  const int64_t max_output_size,
                                                                  hipStream_t hip_stream);
template CUDA_LIB_EXPORT void CudaValidateInputAndInferShape<int64_t>(const int64_t *range_start,
                                                                      const int64_t *range_end,
                                                                      const int64_t *range_delta, int64_t *output_shape,
                                                                      DynamicRangeErrorCode *error_code,
                                                                      const int64_t max_output_size,
                                                                      hipStream_t hip_stream);
template CUDA_LIB_EXPORT void CudaValidateInputAndInferShape<float>(const float *range_start, const float *range_end,
                                                                    const float *range_delta, int64_t *output_shape,
                                                                    DynamicRangeErrorCode *error_code,
                                                                    const int64_t max_output_size,
                                                                    hipStream_t hip_stream);
template CUDA_LIB_EXPORT void CudaValidateInputAndInferShape<double>(const double *range_start, const double *range_end,
                                                                     const double *range_delta, int64_t *output_shape,
                                                                     DynamicRangeErrorCode *error_code,
                                                                     const int64_t max_output_size,
                                                                     hipStream_t hip_stream);

template CUDA_LIB_EXPORT void CalRange<int>(const int *range_start, const int *range_end, const int *range_delta,
                                            int *output, int64_t *output_shape, DynamicRangeErrorCode *error_code,
                                            const int64_t max_output_size, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void CalRange<int64_t>(const int64_t *range_start, const int64_t *range_end,
                                                const int64_t *range_delta, int64_t *output, int64_t *output_shape,
                                                DynamicRangeErrorCode *error_code, const int64_t max_output_size,
                                                hipStream_t hip_stream);
template CUDA_LIB_EXPORT void CalRange<float>(const float *range_start, const float *range_end,
                                              const float *range_delta, float *output, int64_t *output_shape,
                                              DynamicRangeErrorCode *error_code, const int64_t max_output_size,
                                              hipStream_t hip_stream);
template CUDA_LIB_EXPORT void CalRange<double>(const double *range_start, const double *range_end,
                                               const double *range_delta, double *output, int64_t *output_shape,
                                               DynamicRangeErrorCode *error_code, const int64_t max_output_size,
                                               hipStream_t hip_stream);
