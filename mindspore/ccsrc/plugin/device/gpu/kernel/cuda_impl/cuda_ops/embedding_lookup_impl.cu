#include "hip/hip_runtime.h"
/**
 * Copyright 2021 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/gatherv2.cuh"
#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/embedding_lookup_impl.cuh"
#include "include/hip/hip_fp16.h"

template <typename T>
__global__ void SubOffset(T *indices, size_t size, int64_t offset) {
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < size; pos += blockDim.x * gridDim.x) {
    indices[pos] -= static_cast<T>(offset);
  }
  return;
}

template <typename T, typename S>
void CalEmbeddingLookup(T *input, S *indices, T *output, size_t output_dim0, size_t output_dim1, size_t output_dim2,
                        size_t input_dim1, int64_t offset, hipStream_t stream) {
  size_t size = output_dim0 * output_dim1 * output_dim2;
  SubOffset<<<GET_BLOCKS(output_dim1), GET_THREADS, 0, stream>>>(indices, output_dim1, offset);
  GatherV2Kernel<<<GET_BLOCKS(size), GET_THREADS, 0, stream>>>(input, indices, output, output_dim0, output_dim1,
                                                               output_dim2, input_dim1);
  // restore indices
  SubOffset<<<GET_BLOCKS(output_dim1), GET_THREADS, 0, stream>>>(indices, output_dim1, -offset);
  return;
}

template CUDA_LIB_EXPORT void CalEmbeddingLookup<float, int>(float *input, int *indices, float *output,
                                                             size_t output_dim0, size_t output_dim1, size_t output_dim2,
                                                             size_t input_dim1, int64_t offset, hipStream_t stream);
template CUDA_LIB_EXPORT void CalEmbeddingLookup<float, int64_t>(float *input, int64_t *indices, float *output,
                                                                 size_t output_dim0, size_t output_dim1,
                                                                 size_t output_dim2, size_t input_dim1, int64_t offset,
                                                                 hipStream_t stream);
template CUDA_LIB_EXPORT void CalEmbeddingLookup<half, int>(half *input, int *indices, half *output, size_t output_dim0,
                                                            size_t output_dim1, size_t output_dim2, size_t input_dim1,
                                                            int64_t offset, hipStream_t stream);
template CUDA_LIB_EXPORT void CalEmbeddingLookup<half, int64_t>(half *input, int64_t *indices, half *output,
                                                                size_t output_dim0, size_t output_dim1,
                                                                size_t output_dim2, size_t input_dim1, int64_t offset,
                                                                hipStream_t stream);
template CUDA_LIB_EXPORT void CalEmbeddingLookup<double, int>(double *input, int *indices, double *output,
                                                              size_t output_dim0, size_t output_dim1,
                                                              size_t output_dim2, size_t input_dim1, int64_t offset,
                                                              hipStream_t stream);
template CUDA_LIB_EXPORT void CalEmbeddingLookup<double, int64_t>(double *input, int64_t *indices, double *output,
                                                                  size_t output_dim0, size_t output_dim1,
                                                                  size_t output_dim2, size_t input_dim1, int64_t offset,
                                                                  hipStream_t stream);
template CUDA_LIB_EXPORT void CalEmbeddingLookup<int, int>(int *input, int *indices, int *output, size_t output_dim0,
                                                           size_t output_dim1, size_t output_dim2, size_t input_dim1,
                                                           int64_t offset, hipStream_t stream);
template CUDA_LIB_EXPORT void CalEmbeddingLookup<int, int64_t>(int *input, int64_t *indices, int *output,
                                                               size_t output_dim0, size_t output_dim1,
                                                               size_t output_dim2, size_t input_dim1, int64_t offset,
                                                               hipStream_t stream);
template CUDA_LIB_EXPORT void CalEmbeddingLookup<int16_t, int>(int16_t *input, int *indices, int16_t *output,
                                                               size_t output_dim0, size_t output_dim1,
                                                               size_t output_dim2, size_t input_dim1, int64_t offset,
                                                               hipStream_t stream);
template CUDA_LIB_EXPORT void CalEmbeddingLookup<int16_t, int64_t>(int16_t *input, int64_t *indices, int16_t *output,
                                                                   size_t output_dim0, size_t output_dim1,
                                                                   size_t output_dim2, size_t input_dim1,
                                                                   int64_t offset, hipStream_t stream);
template CUDA_LIB_EXPORT void CalEmbeddingLookup<int8_t, int>(int8_t *input, int *indices, int8_t *output,
                                                              size_t output_dim0, size_t output_dim1,
                                                              size_t output_dim2, size_t input_dim1, int64_t offset,
                                                              hipStream_t stream);
template CUDA_LIB_EXPORT void CalEmbeddingLookup<int8_t, int64_t>(int8_t *input, int64_t *indices, int8_t *output,
                                                                  size_t output_dim0, size_t output_dim1,
                                                                  size_t output_dim2, size_t input_dim1, int64_t offset,
                                                                  hipStream_t stream);
template CUDA_LIB_EXPORT void CalEmbeddingLookup<uint8_t, int>(uint8_t *input, int *indices, uint8_t *output,
                                                               size_t output_dim0, size_t output_dim1,
                                                               size_t output_dim2, size_t input_dim1, int64_t offset,
                                                               hipStream_t stream);
template CUDA_LIB_EXPORT void CalEmbeddingLookup<uint8_t, int64_t>(uint8_t *input, int64_t *indices, uint8_t *output,
                                                                   size_t output_dim0, size_t output_dim1,
                                                                   size_t output_dim2, size_t input_dim1,
                                                                   int64_t offset, hipStream_t stream);
template CUDA_LIB_EXPORT void CalEmbeddingLookup<bool, int>(bool *input, int *indices, bool *output, size_t output_dim0,
                                                            size_t output_dim1, size_t output_dim2, size_t input_dim1,
                                                            int64_t offset, hipStream_t stream);
template CUDA_LIB_EXPORT void CalEmbeddingLookup<bool, int64_t>(bool *input, int64_t *indices, bool *output,
                                                                size_t output_dim0, size_t output_dim1,
                                                                size_t output_dim2, size_t input_dim1, int64_t offset,
                                                                hipStream_t stream);
