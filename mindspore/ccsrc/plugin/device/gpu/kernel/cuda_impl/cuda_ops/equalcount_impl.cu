#include "hip/hip_runtime.h"
/**
 * Copyright 2019 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "equalcount_impl.cuh"
#include "include/hip/hip_fp16.h"
template <typename T>
__global__ void EqualCount(const int size, const T* input1, const T* input2, T* output) {
  T equal_count = 0;

  for (int i = 0; i < size; i++) {
    if (input1[i] == input2[i]) {
      equal_count++;
    }
  }

  output[0] = equal_count;
  return;
}
template <typename T>
void CalEqualCount(const int size, const T* input1, const T* input2, T* output, hipStream_t hip_stream) {
  EqualCount<<<1, 1, 0, hip_stream>>>(size, input1, input2, output);
  return;
}

template CUDA_LIB_EXPORT void CalEqualCount<int>(const int size, const int* input1, const int* input2, int* output,
                                                 hipStream_t hip_stream);
template CUDA_LIB_EXPORT void CalEqualCount<float>(const int size, const float* input1, const float* input2,
                                                   float* output, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void CalEqualCount<half>(const int size, const half* input1, const half* input2, half* output,
                                                  hipStream_t hip_stream);
