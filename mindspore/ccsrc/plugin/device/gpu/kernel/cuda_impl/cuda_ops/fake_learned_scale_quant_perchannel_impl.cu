#include "hip/hip_runtime.h"
/**
 * Copyright 2021 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "fake_learned_scale_quant_perchannel_impl.cuh"
#include <thrust/extrema.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/reduce.h>
#include <thrust/pair.h>
#include <algorithm>
#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/util.cuh"

__global__ void FakeLearnedScaleQuantPerChannel(float *output, const int size, float *input_alpha,
                                                float *input_quant, const int channel_num) {
  int channel_idx = 0;
  int per_channel_num = size / channel_num;
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < size; i += blockDim.x * gridDim.x) {
    channel_idx = floor(static_cast<double>(i) / static_cast<double>(per_channel_num));
    // dequantize
    output[i] = input_quant[i] * input_alpha[channel_idx];
  }
  return;
}

__global__ void FakeLearnedScaleQuantPerChannelGrad(float *grad_input, float *grad_alpha, const float *gradient,
                                                    const int size, const float *input_div_alpha,
                                                    const float *input_quant, const bool neg_trunc,
                                                    const int channel_num) {
  int channel_idx = 0;
  int per_channel_num = size / channel_num;
  float lower_bound = -1.0 * !neg_trunc;

  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < size; i += blockDim.x * gridDim.x) {
    float grad_alpha_temp = 0.f;
    channel_idx = floor(static_cast<double>(i) / static_cast<double>(per_channel_num));
    if (input_div_alpha[i] > 1.0) {
      grad_alpha_temp = gradient[i];
      grad_input[i] = 0;
    } else if (input_div_alpha[i] < lower_bound) {
      grad_alpha_temp = -gradient[i];
      grad_input[i] = 0;
    } else {
      grad_input[i] = gradient[i];
      grad_alpha_temp = (gradient[i] * (input_quant[i] -  input_div_alpha[i]));
    }
    MsAtomicAdd(grad_alpha + channel_idx, grad_alpha_temp);
  }
  return;
}

__global__ void LSQNudgePerChannel(const float *input, const int size, float *input_alpha, float *input_quant_max,
                                   float *input_div_alpha, float *input_quant, const bool neg_trunc,
                                   const int channel_num) {
  float input_x;
  int channel_idx = 0;
  int per_channel_num = size / channel_num;
  float lower_bound = -1.0 * !neg_trunc;

  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < size; i += blockDim.x * gridDim.x) {
    channel_idx = floor(static_cast<double>(i) / static_cast<double>(per_channel_num));
    input_x = input[i] / input_alpha[channel_idx];
    input_div_alpha[i] = input_x;
    input_x = max(input_x, lower_bound);
    input_x = min(input_x, 1.0);

    // quantize
    input_quant[i] = floor(input_x * input_quant_max[0] + 0.5f) / input_quant_max[0];
  }
  return;
}

void CalFakeLearnedScaleQuantPerChannel(float *output, const int size, float *input_alpha, float *input_quant,
                                        const int channel_num, hipStream_t hip_stream) {
  FakeLearnedScaleQuantPerChannel<<<GET_BLOCKS(size), GET_THREADS, 0, hip_stream>>>(output, size, input_alpha,
                                                                                     input_quant, channel_num);
  return;
}

void CalFakeLearnedScaleQuantPerChannelGrad(float *grad_input, float *grad_alpha, const float *gradient, const int size,
                                            const float *input_div_alpha, const float *input_quant,
                                            const bool neg_trunc, const int channel_num, hipStream_t hip_stream) {
  FakeLearnedScaleQuantPerChannelGrad<<<GET_BLOCKS(size), GET_THREADS, 0, hip_stream>>>(grad_input,
                                                                                         grad_alpha,
                                                                                         gradient,
                                                                                         size,
                                                                                         input_div_alpha,
                                                                                         input_quant,
                                                                                         neg_trunc,
                                                                                         channel_num);
  return;
}

void CalLSQNudgePerChannel(const float *input, const int size, float *input_alpha, float *input_quant_max,
                           float *input_div_alpha, float *input_quant, const bool neg_trunc, const int channel_num,
                           hipStream_t hip_stream) {
  LSQNudgePerChannel<<<GET_BLOCKS(size), GET_THREADS, 0, hip_stream>>>(input, size, input_alpha, input_quant_max,
                                                                        input_div_alpha, input_quant, neg_trunc,
                                                                        channel_num);
  return;
}
