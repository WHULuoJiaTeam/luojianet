#include "hip/hip_runtime.h"
/**
 * Copyright 2020 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "include/hip/hip_runtime.h"
#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/float_status_impl.cuh"
#include "include/hip/hip_fp16.h"

template <typename T>
__global__ void IsNan(const size_t size, const T* input, bool* out) {
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < (size); pos += blockDim.x * gridDim.x) {
    if (isnan(input[pos])) {
      out[pos] = true;
    } else {
      out[pos] = false;
    }
  }
  return;
}
template <>
__global__ void IsNan(const size_t size, const half* input, bool* out) {
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < (size); pos += blockDim.x * gridDim.x) {
    if (__hisnan(input[pos])) {
      out[pos] = true;
    } else {
      out[pos] = false;
    }
  }
  return;
}

template <typename T>
__global__ void IsInf(const size_t size, const T* input, bool* out) {
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < (size); pos += blockDim.x * gridDim.x) {
    if (isinf(input[pos]) != 0) {
      out[pos] = true;
    } else {
      out[pos] = false;
    }
  }
  return;
}
template <>
__global__ void IsInf(const size_t size, const half* input, bool* out) {
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < (size); pos += blockDim.x * gridDim.x) {
    if (__hisinf(input[pos]) != 0) {
      out[pos] = true;
    } else {
      out[pos] = false;
    }
  }
  return;
}

template <typename T>
__global__ void IsFinite(const size_t size, const T* input, bool* out) {
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < (size); pos += blockDim.x * gridDim.x) {
    if (isinf(input[pos]) == 0 && !isnan(input[pos])) {
      out[pos] = true;
    } else {
      out[pos] = false;
    }
  }
  return;
}
template <>
__global__ void IsFinite(const size_t size, const half* input, bool* out) {
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < (size); pos += blockDim.x * gridDim.x) {
    if (__hisinf(input[pos]) == 0 && !__hisnan(input[pos])) {
      out[pos] = true;
    } else {
      out[pos] = false;
    }
  }
  return;
}

template <typename T>
__global__ void FloatStatus(const size_t size, const T* input, float* out) {
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < (size); pos += blockDim.x * gridDim.x) {
    if (isinf(input[pos]) != 0 || isnan(input[pos])) {
      out[0] = 1;
    }
  }
  return;
}
template <>
__global__ void FloatStatus(const size_t size, const half* input, float* out) {
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < (size); pos += blockDim.x * gridDim.x) {
    if (__hisinf(input[pos]) != 0 || __hisnan(input[pos])) {
      out[0] = 1;
    }
  }
  return;
}

template <typename T>
void CalFloatStatus(const size_t size, const T* input, float* output, hipStream_t hip_stream) {
  FloatStatus<<<GET_BLOCKS(size), GET_THREADS, 0, hip_stream>>>(size, input, output);
  return;
}
template <typename T>
void CalIsNan(const size_t size, const T* input, bool* output, hipStream_t hip_stream) {
  IsNan<<<GET_BLOCKS(size), GET_THREADS, 0, hip_stream>>>(size, input, output);
  return;
}
template <typename T>
void CalIsInf(const size_t size, const T* input, bool* output, hipStream_t hip_stream) {
  IsInf<<<GET_BLOCKS(size), GET_THREADS, 0, hip_stream>>>(size, input, output);
  return;
}
template <typename T>
void CalIsFinite(const size_t size, const T* input, bool* output, hipStream_t hip_stream) {
  IsFinite<<<GET_BLOCKS(size), GET_THREADS, 0, hip_stream>>>(size, input, output);
  return;
}

template CUDA_LIB_EXPORT void CalFloatStatus<float>(const size_t size, const float* input, float* output,
                                                    hipStream_t hip_stream);
template CUDA_LIB_EXPORT void CalFloatStatus<half>(const size_t size, const half* input, float* output,
                                                   hipStream_t hip_stream);
template CUDA_LIB_EXPORT void CalFloatStatus<double>(const size_t size, const double* input, float* output,
                                                     hipStream_t hip_stream);
template CUDA_LIB_EXPORT void CalIsInf<float>(const size_t size, const float* input, bool* output,
                                              hipStream_t hip_stream);
template CUDA_LIB_EXPORT void CalIsInf<half>(const size_t size, const half* input, bool* output,
                                             hipStream_t hip_stream);
template CUDA_LIB_EXPORT void CalIsInf<double>(const size_t size, const double* input, bool* output,
                                               hipStream_t hip_stream);
template CUDA_LIB_EXPORT void CalIsNan<float>(const size_t size, const float* input, bool* output,
                                              hipStream_t hip_stream);
template CUDA_LIB_EXPORT void CalIsNan<half>(const size_t size, const half* input, bool* output,
                                             hipStream_t hip_stream);
template CUDA_LIB_EXPORT void CalIsNan<double>(const size_t size, const double* input, bool* output,
                                               hipStream_t hip_stream);
template CUDA_LIB_EXPORT void CalIsFinite<float>(const size_t size, const float* input, bool* output,
                                                 hipStream_t hip_stream);
template CUDA_LIB_EXPORT void CalIsFinite<half>(const size_t size, const half* input, bool* output,
                                                hipStream_t hip_stream);
template CUDA_LIB_EXPORT void CalIsFinite<double>(const size_t size, const double* input, bool* output,
                                                  hipStream_t hip_stream);
