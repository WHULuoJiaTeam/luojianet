#include "hip/hip_runtime.h"
/**
 * Copyright 2021 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/hsigmoid_impl.cuh"
#include "include/hip/hip_fp16.h"

template <typename T>
__global__ void HsigmoidKernel(size_t size, const T *input, T *output) {
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < size; pos += blockDim.x * gridDim.x) {
    T value = (input[pos] + static_cast<T>(3)) / static_cast<T>(6);
    value = value > static_cast<T>(1) ? static_cast<T>(1) : value;
    output[pos] = value > static_cast<T>(0) ? value : static_cast<T>(0);
  }
}

template <typename T>
__global__ void HsigmoidGradKernel(size_t size, const T *dout, const T *x, T *output) {
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < size; pos += blockDim.x * gridDim.x) {
    T value = dout[pos] / static_cast<T>(6);
    output[pos] = (x[pos] > static_cast<T>(-3) && x[pos] < static_cast<T>(3)) ? value : static_cast<T>(0);
  }
}

template <typename T>
void CalHSigmoid(const size_t &size, const T *input, T *output, hipStream_t hip_stream) {
  HsigmoidKernel<<<GET_BLOCKS(size), GET_THREADS, 0, hip_stream>>>(size, input, output);
}

template <typename T>
void CalHSigmoidGrad(const size_t &size, const T *dout, const T *x, T *output, hipStream_t hip_stream) {
  HsigmoidGradKernel<<<GET_BLOCKS(size), GET_THREADS, 0, hip_stream>>>(size, dout, x, output);
}

template CUDA_LIB_EXPORT void CalHSigmoid<half>(const size_t &size, const half *input, half *output,
                                                hipStream_t hip_stream);
template CUDA_LIB_EXPORT void CalHSigmoid<float>(const size_t &size, const float *input, float *output,
                                                 hipStream_t hip_stream);

template CUDA_LIB_EXPORT void CalHSigmoidGrad<half>(const size_t &size, const half *dout, const half *x, half *output,
                                                    hipStream_t hip_stream);
template CUDA_LIB_EXPORT void CalHSigmoidGrad<float>(const size_t &size, const float *dout, const float *x,
                                                     float *output, hipStream_t hip_stream);
