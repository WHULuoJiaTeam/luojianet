#include "hip/hip_runtime.h"
/**
 * Copyright 2021 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "in_top_k_impl.cuh"
#include <hip/hip_runtime.h>
#include "include/hip/hip_fp16.h"

template <typename T>
__global__ void InTopK(const T *predictions, const int32_t *targets, bool *output, const T *top_k_output,
                       size_t batch_size, size_t class_id_count, int64_t k) {
  size_t gt_id = blockIdx.x * blockDim.x + threadIdx.x;
  for (; gt_id < batch_size; gt_id += blockDim.x * gridDim.x) {
    int32_t target_index = targets[gt_id];
    T predicted_value = predictions[gt_id * class_id_count + target_index];
    T top_k_smallest_value = top_k_output[gt_id * k + k - 1];

    output[gt_id] = predicted_value >= top_k_smallest_value;
  }
}

template <typename T>
void CalInTopK(const T *predictions, const int32_t *targets, bool *output, const T *top_k_output, size_t batch_size,
               size_t class_id_count, int64_t k, hipStream_t hip_stream) {
  InTopK<<<GET_BLOCKS(class_id_count), GET_THREADS, 0, hip_stream>>>(predictions, targets, output, top_k_output,
                                                                      batch_size, class_id_count, k);
}

template CUDA_LIB_EXPORT void CalInTopK<half>(const half *predictions, const int32_t *targets, bool *output,
                                              const half *top_k_output, size_t batch_size, size_t class_id_count,
                                              int64_t k, hipStream_t hip_stream);

template CUDA_LIB_EXPORT void CalInTopK<float>(const float *predictions, const int32_t *targets, bool *output,
                                               const float *top_k_output, size_t batch_size, size_t class_id_count,
                                               int64_t k, hipStream_t hip_stream);
