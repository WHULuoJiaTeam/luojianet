#include "hip/hip_runtime.h"
/**
 * Copyright 2020-2021 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "l2_loss.cuh"
#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/util.cuh"

template <typename T>
__global__ void L2LossKernel(const size_t input_size, const T *input , T *output) {
  T ret = 0;
  for (size_t id = blockIdx.x * blockDim.x + threadIdx.x; id < input_size; id += blockDim.x * gridDim.x) {
      ret = input[id] * input[id];
      ret /= static_cast<T>(2);
      MsAtomicAdd(output, ret);
  }
}

template <typename T>
__global__ void ClearOutputMem(T *output) {
    output[0] = static_cast<T>(0);
}

template <typename T>
void L2Loss(const size_t input_size, const T *input , T *output, hipStream_t stream) {
  ClearOutputMem<<<GET_BLOCKS(1), GET_THREADS, 0, stream>>>(output);
  L2LossKernel<<<GET_BLOCKS(input_size), GET_THREADS, 0, stream>>>(input_size, input, output);
}

template CUDA_LIB_EXPORT void L2Loss<float>(const size_t input_size, const float *input , float *output,
                                            hipStream_t stream);
template CUDA_LIB_EXPORT void L2Loss<half>(const size_t input_size, const half *input , half *output,
                                           hipStream_t stream);
