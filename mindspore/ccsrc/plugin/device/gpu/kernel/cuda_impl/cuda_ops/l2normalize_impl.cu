#include "hip/hip_runtime.h"
/**
 * Copyright 2020-2021 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "l2normalize_impl.cuh"
#include "include/hip/hip_fp16.h"
template <typename T>
__global__ void AssignEps(const size_t size, const float eps, T* value) {
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < (size); pos += blockDim.x * gridDim.x) {
    float v = static_cast<float>(value[pos]);
    float max = v > eps ? v : eps;
    value[pos] = static_cast<T>(max);
  }
}

template <typename T>
void GetMaxWithEpsAndValue(const size_t size, const float eps, T* value, hipStream_t hip_stream) {
  AssignEps<<<GET_BLOCKS(size), GET_THREADS, 0, hip_stream>>>(size, eps, value);
}

template CUDA_LIB_EXPORT void GetMaxWithEpsAndValue<float>(const size_t size, const float eps, float* value,
                                                           hipStream_t hip_stream);
template CUDA_LIB_EXPORT void GetMaxWithEpsAndValue<half>(const size_t size, const float eps, half* value,
                                                          hipStream_t hip_stream);
template CUDA_LIB_EXPORT void GetMaxWithEpsAndValue<int>(const size_t size, const float eps, int* value,
                                                         hipStream_t hip_stream);
