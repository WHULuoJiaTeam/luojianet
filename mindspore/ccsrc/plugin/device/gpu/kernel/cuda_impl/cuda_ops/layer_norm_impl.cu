#include "hip/hip_runtime.h"
/**
 * Copyright 2020 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <stdio.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/layer_norm_impl.cuh"

constexpr int NUM_PER_THREAD_REDUCE = 4;
constexpr int WARP_SIZE = 32;

template <typename T>
inline __device__ void MeanAndVarAccumulation(T *mean, T *var, T *num, const T &val) {
  // Welford Algorithm:
  // \mu_k = \mu_{k-1} + (x_k - \mu_{k-1})/k
  // \sigma_k^2 = \sigma_{k-1}^2 + (x_k - \mu_{k-1}) * (x_k - \mu_k)
  num[0]++;
  T mean_new = mean[0] + (val - mean[0]) / num[0];
  var[0] = var[0] + (val - mean[0]) * (val - mean_new);
  mean[0] = mean_new;
}

template <typename T>
inline __device__ void MeanAndVarMerge(T *m1, T *v1, T *n1, const T &m2, const T &v2, const T &n2) {
  T zero = 0;
  if (n2 == zero) {
    return;
  }

  T count = n1[0] + n2;
  v1[0] = v1[0] + v2 + (m1[0] - m2) * (m1[0] - m2) * n1[0] * n2 / count;
  m1[0] = (n1[0] * m1[0] + n2 * m2) / count;
  n1[0] = count;
}

template <typename T>
inline __device__ void ThreadReduce(const int &col_dim, const T *block_addr, T *mean, T *var, T *num) {
  int loop_num = (col_dim + NUM_PER_THREAD_REDUCE - 1) / NUM_PER_THREAD_REDUCE;
  for (int i = threadIdx.x; i < loop_num; i += blockDim.x) {
    for (int j = 0; j < NUM_PER_THREAD_REDUCE; j++) {
      int pos = NUM_PER_THREAD_REDUCE * i + j;
      if (pos >= col_dim) {
        return;
      }
      MeanAndVarAccumulation(mean, var, num, block_addr[pos]);
    }
  }
}

template <typename T>
inline __device__ void WarpReduce(T *mean, T *var, T *num) {
  for (int delta = (WARP_SIZE >> 1); delta > 0; delta >>= 1) {
    T mean_other = __shfl_down_sync(0xffffffff, mean[0], delta);
    T var_other = __shfl_down_sync(0xffffffff, var[0], delta);
    T num_other = __shfl_down_sync(0xffffffff, num[0], delta);
    MeanAndVarMerge(mean, var, num, mean_other, var_other, num_other);
  }
}

template <typename T>
inline __device__ void BlockReduce(const int &col_dim, T *mean, T *var, T *num, T *mean_addr, T *var_addr,
                                   T *share_mem) {
  // load data to share memory
  // thread(0, 32, 64, 96, ...) keep the data
  if (threadIdx.x % WARP_SIZE == 0) {
    int offset = threadIdx.x / WARP_SIZE * 3;
    share_mem[offset] = mean[0];
    share_mem[offset + 1] = var[0];
    share_mem[offset + 2] = num[0];
  }
  __syncthreads();

  for (int stride = blockDim.x / WARP_SIZE / 2; stride > 0; stride >>= 1) {
    if (threadIdx.x < stride) {
      int offset = (threadIdx.x + stride) * 3;
      MeanAndVarMerge(&share_mem[threadIdx.x * 3], &share_mem[threadIdx.x * 3 + 1], &share_mem[threadIdx.x * 3 + 2],
                      share_mem[offset], share_mem[offset + 1], share_mem[offset + 2]);
    }
  }
  __syncthreads();

  if (threadIdx.x == 0) {
    mean_addr[blockIdx.x] = share_mem[0];
    share_mem[1] /= col_dim;
    var_addr[blockIdx.x] = share_mem[1];
  }
}

template <typename T>
inline __device__ void LayerNorm(const int &row, const int &col_dim, const int &param_dim, const T *x,
                                 const T *share_mem, const T *gamma, const T *beta, const T epsilon, T *y) {
  for (int col = threadIdx.x; col < col_dim; col += blockDim.x) {
    int pos = row * col_dim + col;
    int i = pos % param_dim;
    y[pos] = (x[pos] - share_mem[0]) / sqrt(share_mem[1] + epsilon) * gamma[i] + beta[i];
  }
}

template <>
inline __device__ void LayerNorm(const int &row, const int &col_dim, const int &param_dim, const half *x,
                                 const half *share_mem, const half *gamma, const half *beta, const half epsilon,
                                 half *y) {
  for (int col = threadIdx.x; col < col_dim; col += blockDim.x) {
    int pos = row * col_dim + col;
    int i = pos % param_dim;
    y[pos] = (x[pos] - share_mem[0]) / hsqrt(share_mem[1] + epsilon) * gamma[i] + beta[i];
  }
}

template <typename T>
__global__ void LayerNormKernel(const int row_dim, const int col_dim, const int param_dim, const T epsilon, const T *x,
                                const T *gamma, const T *beta, T *y, T *mean_addr, T *var_addr) {
  for (auto row = blockIdx.x; row < row_dim; row += gridDim.x) {
    T mean = 0;
    T var = 0;
    T num = 0;
    const T *block_addr = x + row * col_dim;
    DynamicSharedMem<T> share_mem;

    ThreadReduce(col_dim, block_addr, &mean, &var, &num);
    WarpReduce(&mean, &var, &num);
    BlockReduce(col_dim, &mean, &var, &num, mean_addr, var_addr, share_mem.addr());

    __syncthreads();
    LayerNorm(row, col_dim, param_dim, x, share_mem.addr(), gamma, beta, epsilon, y);
  }
}

template <typename T>
void LayerNorm(const int &row_dim, const int &col_dim, const int &param_dim, const T &epsilon, const T *x,
               const T *gamma, const T *beta, T *y, T *mean, T *var, hipStream_t stream) {
  const int thread_per_block = 256;
  // keep the mean/var/num after warp reduce
  int share_mem_size = thread_per_block / WARP_SIZE * 3 * sizeof(T);
  LayerNormKernel<<<row_dim, thread_per_block, share_mem_size, stream>>>(row_dim, col_dim, param_dim, epsilon, x, gamma,
                                                                         beta, y, mean, var);
}

template CUDA_LIB_EXPORT void LayerNorm(const int &row_dim, const int &col_dim, const int &param_dim,
                                        const float &epsilon, const float *x, const float *gamma, const float *beta,
                                        float *y, float *mean, float *var, hipStream_t stream);
template CUDA_LIB_EXPORT void LayerNorm(const int &row_dim, const int &col_dim, const int &param_dim,
                                        const half &epsilon, const half *x, const half *gamma, const half *beta,
                                        half *y, half *mean, half *var, hipStream_t stream);
