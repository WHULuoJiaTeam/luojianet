#include "hip/hip_runtime.h"
/**
 * Copyright 2020 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/linspace.cuh"
#include <iostream>

template <typename T>
__global__ void LinSpaceKernel(const T *start, const T *stop, const size_t value_count, T *output) {
  T add_value = ((*stop - *start) / (value_count - 1));
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < value_count; i += gridDim.x * blockDim.x) {
    output[i] = *start + (add_value * i);
  }
}
template <typename T>
void calLinSpace(const T *start, const T *stop, const size_t value_count, T *output, hipStream_t hip_stream) {
  LinSpaceKernel<<<GET_BLOCKS(value_count), GET_THREADS, 0, hip_stream>>>(start, stop, value_count, output);
}
template CUDA_LIB_EXPORT void calLinSpace<float>(const float *start, const float *stop, const size_t value_count,
                                                 float *output, hipStream_t hip_stream);
