#include "hip/hip_runtime.h"
/**
 * Copyright 2020 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <algorithm>
#include "maxpool_with_argmax_grad_impl.cuh"
#include "include/hip/hip_fp16.h"
#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/util.cuh"

template <typename T, typename S>
__global__ void MaxPoolWithArgmaxGrad(const T* dy,
                                      const S* index,
                                      const int xCHW,
                                      const int dyCHW,
                                      const int dyNCHW,
                                      T* dx) {
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < (dyNCHW); pos += blockDim.x * gridDim.x) {
    const S idx = index[pos];
    const int posn = pos / dyCHW;
    MsAtomicAdd(dx + posn*xCHW + static_cast<int>(idx), dy[pos]);
  }
  return;
}

template <typename T>
__global__ void InitOutput(const int size, T *output) {
    T zero = 0;
    for (size_t id = blockIdx.x * blockDim.x + threadIdx.x; id < size; id += blockDim.x * gridDim.x) {
        output[id] = zero;
    }
    return;
}

template <typename T, typename S>
void CalMaxPoolWithArgmaxGrad(const T* dy,
                              const S* index,
                              const int n,
                              const int c,
                              const int xHeight,
                              const int xWidth,
                              const int dyHeight,
                              const int dyWidth,
                              T* dx,
                              hipStream_t hip_stream) {
  const int xHW = xHeight*xWidth;
  const int xCHW = c*xHW;
  const int xNCHW = n*xCHW;
  const int dyHW = dyHeight*dyWidth;
  const int dyCHW = c*dyHW;
  const int dyNCHW = n*dyCHW;
  InitOutput<<<GET_BLOCKS(xNCHW), GET_THREADS, 0, hip_stream>>>(xNCHW, dx);
  MaxPoolWithArgmaxGrad<<<GET_BLOCKS(dyNCHW),
                          GET_THREADS,
                          0,
                          hip_stream>>>(
                            dy,
                            index,
                            xCHW,
                            dyCHW,
                            dyNCHW,
                            dx);
  return;
}

template CUDA_LIB_EXPORT void CalMaxPoolWithArgmaxGrad<float, int>(const float* dy,
                                                                   const int* index,
                                                                   const int n,
                                                                   const int c,
                                                                   const int xHeight,
                                                                   const int xWidth,
                                                                   const int dyHeight,
                                                                   const int dyWidth,
                                                                   float* dx,
                                                                   hipStream_t hip_stream);
template CUDA_LIB_EXPORT void CalMaxPoolWithArgmaxGrad<half, int>(const half* dy,
                                                                  const int* index,
                                                                  const int n,
                                                                  const int c,
                                                                  const int xHeight,
                                                                  const int xWidth,
                                                                  const int dyHeight,
                                                                  const int dyWidth,
                                                                  half* dx,
                                                                  hipStream_t hip_stream);
