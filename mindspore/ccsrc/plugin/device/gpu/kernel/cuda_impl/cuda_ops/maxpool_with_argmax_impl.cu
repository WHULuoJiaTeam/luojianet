#include "hip/hip_runtime.h"
/**
 * Copyright 2020-2021 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <algorithm>
#include "maxpool_with_argmax_impl.cuh"
#include "include/hip/hip_fp16.h"
template <typename T, typename S>
__global__ void MaxPoolWithArgmax(const T* input,
                                  const int n,
                                  const int c,
                                  const int h,
                                  const int w,
                                  const int windowHeight,
                                  const int windowWidth,
                                  const int strideHeight,
                                  const int strideWidth,
                                  const int padTop,
                                  const int padLeft,
                                  const int outputHeight,
                                  const int outputWidth,
                                  const int outputNCHW,
                                  const int outputCHW,
                                  const int outputHW,
                                  T* output,
                                  S *index) {
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x;
       pos < (outputNCHW);
       pos += blockDim.x * gridDim.x) {
    const int posn = pos / outputCHW;
    const int posc = pos / outputHW % c;
    const int posh = pos / outputWidth % outputHeight;
    const int posw = pos % outputWidth;
    int hstart = posh * strideHeight - padTop;
    int wstart = posw * strideWidth - padLeft;
    const int hend = min(hstart + windowHeight, h);
    const int wend = min(wstart + windowWidth, w);
    hstart = max(hstart, 0);
    wstart = max(wstart, 0);
    S inputStart = posn*c*h*w;
    S maxIdx = posc*h*w + hstart*w + wstart;
    T maxData = input[inputStart+maxIdx];
    for (int hcur = hstart; hcur < hend; ++hcur) {
        for (int wcur = wstart; wcur < wend; ++wcur) {
            S inputIdx = posc*h*w + hcur*w + wcur;
            T inputData = input[inputStart+inputIdx];
            if (inputData > maxData) {
              maxIdx = inputIdx;
              maxData = inputData;
            }
        }
    }
    output[pos] = maxData;
    index[pos] = maxIdx;
  }
}

template <typename T, typename S>
void CalMaxPoolWithArgmax(const T* input,
                          const int n,
                          const int c,
                          const int h,
                          const int w,
                          const int windowHeight,
                          const int windowWidth,
                          const int strideHeight,
                          const int strideWidth,
                          const int padTop,
                          const int padLeft,
                          const int outputHeight,
                          const int outputWidth,
                          T* output,
                          S *index,
                          hipStream_t hip_stream) {
  const int outputNCHW = n*c*outputHeight*outputWidth;
  const int outputCHW = c*outputHeight*outputWidth;
  const int outputHW = outputHeight*outputWidth;
  MaxPoolWithArgmax<<<GET_BLOCKS(n*c*outputHeight*outputWidth),
                      GET_THREADS,
                      0,
                      hip_stream>>>(
                        input,
                        n,
                        c,
                        h,
                        w,
                        windowHeight,
                        windowWidth,
                        strideHeight,
                        strideWidth,
                        padTop,
                        padLeft,
                        outputHeight,
                        outputWidth,
                        outputNCHW,
                        outputCHW,
                        outputHW,
                        output,
                        index);
}

template CUDA_LIB_EXPORT void CalMaxPoolWithArgmax<float, int>(const float* input,
                                                               const int n,
                                                               const int c,
                                                               const int h,
                                                               const int w,
                                                               const int windowHeight,
                                                               const int windowWidth,
                                                               const int strideHeight,
                                                               const int strideWidth,
                                                               const int padTop,
                                                               const int padLeft,
                                                               const int outputHeight,
                                                               const int outputWidth,
                                                               float* output,
                                                               int* index,
                                                               hipStream_t hip_stream);

template CUDA_LIB_EXPORT void CalMaxPoolWithArgmax<half, int>(const half* input,
                                                              const int n,
                                                              const int c,
                                                              const int h,
                                                              const int w,
                                                              const int windowHeight,
                                                              const int windowWidth,
                                                              const int strideHeight,
                                                              const int strideWidth,
                                                              const int padTop,
                                                              const int padLeft,
                                                              const int outputHeight,
                                                              const int outputWidth,
                                                              half* output,
                                                              int* index,
                                                              hipStream_t hip_stream);
