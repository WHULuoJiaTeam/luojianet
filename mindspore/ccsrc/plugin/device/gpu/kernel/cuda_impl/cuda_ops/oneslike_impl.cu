#include "hip/hip_runtime.h"
/**
 * Copyright 2020-2021 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <hip/hip_runtime.h>
#include "oneslike_impl.cuh"
#include "include/hip/hip_fp16.h"
template <typename T>
__global__ void OnesLike(const size_t size, const T* input,  T* output) {
  int one = 1;
  T val = static_cast<T>(one);
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < size; pos += blockDim.x * gridDim.x) {
    output[pos] = val;
  }
  return;
}
template <typename T>
void CalOnesLike(const size_t size, const T* input, T* output, hipStream_t hip_stream) {
  OnesLike<<<GET_BLOCKS(size), GET_THREADS, 0, hip_stream>>>(size, input, output);
  return;
}

template CUDA_LIB_EXPORT void CalOnesLike<double>(const size_t size, const double* input, double* output,
                                                  hipStream_t hip_stream);
template CUDA_LIB_EXPORT void CalOnesLike<float>(const size_t size, const float* input, float* output,
                                                 hipStream_t hip_stream);
template CUDA_LIB_EXPORT void CalOnesLike<half>(const size_t size, const half* input, half* output,
                                                hipStream_t hip_stream);
template CUDA_LIB_EXPORT void CalOnesLike<int8_t>(const size_t size, const int8_t* input, int8_t* output,
                                                  hipStream_t hip_stream);
template CUDA_LIB_EXPORT void CalOnesLike<int16_t>(const size_t size, const int16_t* input, int16_t* output,
                                                   hipStream_t hip_stream);
template CUDA_LIB_EXPORT void CalOnesLike<int32_t>(const size_t size, const int32_t* input, int32_t* output,
                                                   hipStream_t hip_stream);
template CUDA_LIB_EXPORT void CalOnesLike<int64_t>(const size_t size, const int64_t* input, int64_t* output,
                                                   hipStream_t hip_stream);
template CUDA_LIB_EXPORT void CalOnesLike<uint8_t>(const size_t size, const uint8_t* input, uint8_t* output,
                                                   hipStream_t hip_stream);
template CUDA_LIB_EXPORT void CalOnesLike<uint16_t>(const size_t size, const uint16_t* input, uint16_t* output,
                                                    hipStream_t hip_stream);
template CUDA_LIB_EXPORT void CalOnesLike<uint32_t>(const size_t size, const uint32_t* input, uint32_t* output,
                                                    hipStream_t hip_stream);
template CUDA_LIB_EXPORT void CalOnesLike<uint64_t>(const size_t size, const uint64_t* input, uint64_t* output,
                                                    hipStream_t hip_stream);
