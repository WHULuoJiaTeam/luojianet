#include "hip/hip_runtime.h"
/**
 * Copyright 2020 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "random_op_impl.cuh"
template <typename T>
__global__ void NormalKernel(int seed, hiprandState *globalState, T *output, size_t count) {
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < (count); i += blockDim.x * gridDim.x) {
    hiprand_init(seed, i, 0, &globalState[i]);
    output[i] = (T)hiprand_normal(&globalState[i]);
  }
  return;
}

__device__ bool dev_error_res = false;

template <typename T>
__global__ void UniformIntKernel(int seed, hiprandState *globalState, T *input1, size_t input_size_1,
                                 T *input2, size_t input_size_2, T *output, size_t count) {
  if (!(input1[0] < input2[0])) {
    dev_error_res = false;
    return;
  }
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < (count); i += blockDim.x * gridDim.x) {
    hiprand_init(seed, i, 0, &globalState[i]);
    output[i] = (T)(hiprand_uniform(&globalState[i]) * (input2[0] - input1[0])) + input1[0];
  }
  dev_error_res = true;
  return;
}

template <typename T>
__global__ void UniformRealKernel(int seed, hiprandState *globalState, T *output, size_t count) {
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < (count); i += blockDim.x * gridDim.x) {
    hiprand_init(seed, i, 0, &globalState[i]);
    output[i] = (T)hiprand_uniform(&globalState[i]);
  }
  return;
}

template <typename T>
void StandardNormal(int seed, int seed2, hiprandState *globalState, T *output, size_t count, hipStream_t hip_stream) {
  int RNG_seed = 0;
  std::random_device rd;
  if (seed2 != 0) {
    RNG_seed = seed2;
  } else if (seed != 0) {
    RNG_seed = seed;
  } else {
    RNG_seed = static_cast<int>(rd());
  }
  NormalKernel<<<GET_BLOCKS(count), GET_THREADS, 0, hip_stream>>>(RNG_seed, globalState, output, count);
  return;
}

template <typename T>
bool UniformInt(int seed, int seed2, hiprandState *globalState, T *input1, size_t input_size_1,
                T *input2, size_t input_size_2, T *output, size_t count, hipStream_t hip_stream) {
  int RNG_seed = 0;
  std::random_device rd;
  if (seed2 != 0) {
    RNG_seed = seed2;
  } else if (seed != 0) {
    RNG_seed = seed;
  } else {
    RNG_seed = static_cast<int>(rd());
  }
  bool host_error_res = false;
  UniformIntKernel<<<GET_BLOCKS(count), GET_THREADS, 0, hip_stream>>>
               (RNG_seed, globalState, input1, input_size_1, input2, input_size_2, output, count);
  hipDeviceSynchronize();
  hipMemcpyFromSymbol(&host_error_res, HIP_SYMBOL(dev_error_res), sizeof(bool));
  return host_error_res;
}

template <typename T>
void UniformReal(int seed, int seed2, hiprandState *globalState, T *output, size_t count, hipStream_t hip_stream) {
  int RNG_seed = 0;
  std::random_device rd;
  if (seed2 != 0) {
    RNG_seed = seed2;
  } else if (seed != 0) {
    RNG_seed = seed;
  } else {
    RNG_seed = static_cast<int>(rd());
  }
  UniformRealKernel<<<GET_BLOCKS(count), GET_THREADS, 0, hip_stream>>>(RNG_seed, globalState, output, count);
  return;
}

template CUDA_LIB_EXPORT void StandardNormal<float>(int seed, int seed2, hiprandState *globalState,
                                                    float *output, size_t count, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void StandardNormal<int>(int seed, int seed2, hiprandState *globalState,
                                                  int *output, size_t count, hipStream_t hip_stream);
template CUDA_LIB_EXPORT bool UniformInt<float>(int seed, int seed2, hiprandState *globalState, float *input1,
                                                size_t input_size_1, float *input2, size_t input_size_2, float *output,
                                                size_t count, hipStream_t hip_stream);
template CUDA_LIB_EXPORT bool UniformInt<int>(int seed, int seed2, hiprandState *globalState, int *input1,
                                              size_t input_size_1, int *input2, size_t input_size_2, int *output,
                                              size_t count, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void UniformReal<float>(int seed, int seed2, hiprandState *globalState,
                                                 float *output, size_t count, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void UniformReal<int>(int seed, int seed2, hiprandState *globalState,
                                               int *output, size_t count, hipStream_t hip_stream);
