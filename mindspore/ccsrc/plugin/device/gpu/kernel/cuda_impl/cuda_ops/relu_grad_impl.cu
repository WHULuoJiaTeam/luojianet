#include "hip/hip_runtime.h"
/**
 * Copyright 2020-2021 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/relu_grad_impl.cuh"
#include "include/hip/hip_fp16.h"

template <typename T>
__global__ void CalReLUGradKernel(int size, T *dy, T *y, T *dx) {
  for (int pos = blockIdx.x * blockDim.x + threadIdx.x; pos < size; pos += blockDim.x * gridDim.x) {
    dx[pos] = y[pos] > static_cast<T>(0) ? dy[pos] : static_cast<T>(0);
  }
}

template <typename T>
void CalReLUGrad(int size, T *dy, T *y, T *dx, hipStream_t hip_stream) {
  CalReLUGradKernel<<<GET_BLOCKS(size), GET_THREADS, 0, hip_stream>>>(size, dy, y, dx);
  return;
}

template CUDA_LIB_EXPORT void CalReLUGrad(int size, double *dy, double *y, double *dx, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void CalReLUGrad(int size, float *dy, float *y, float *dx, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void CalReLUGrad(int size, half *dy, half *y, half *dx, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void CalReLUGrad(int size, int8_t *dy, int8_t *y, int8_t *dx, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void CalReLUGrad(int size, int16_t *dy, int16_t *y, int16_t *dx, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void CalReLUGrad(int size, int32_t *dy, int32_t *y, int32_t *dx, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void CalReLUGrad(int size, int64_t *dy, int64_t *y, int64_t *dx, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void CalReLUGrad(int size, uint8_t *dy, uint8_t *y, uint8_t *dx, hipStream_t hip_stream);
