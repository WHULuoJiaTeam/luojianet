#include "hip/hip_runtime.h"
/**
 * Copyright 2020-2021 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/relu_impl.cuh"
#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/util.cuh"

template <typename T>
__global__ void CalReLUKernel(int size, T *input_addr, T *output_addr) {
  for (int pos = blockIdx.x * blockDim.x + threadIdx.x; pos < size; pos += blockDim.x * gridDim.x) {
    output_addr[pos] = input_addr[pos] > static_cast<T>(0) ? input_addr[pos] : static_cast<T>(0);
  }
}

template <typename T>
void CalReLU(int size, T *input_addr, T *output_addr, hipStream_t hip_stream) {
  CalReLUKernel<<<GET_BLOCKS(size), GET_THREADS, 0, hip_stream>>>(size, input_addr, output_addr);
}

template CUDA_LIB_EXPORT void CalReLU(int size, double *input_addr, double *output_addr, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void CalReLU(int size, float *input_addr, float *output_addr, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void CalReLU(int size, half *input_addr, half *output_addr, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void CalReLU(int size, int8_t *input_addr, int8_t *output_addr, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void CalReLU(int size, int16_t *input_addr, int16_t *output_addr, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void CalReLU(int size, int32_t *input_addr, int32_t *output_addr, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void CalReLU(int size, int64_t *input_addr, int64_t *output_addr, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void CalReLU(int size, uint8_t *input_addr, uint8_t *output_addr, hipStream_t hip_stream);

template <typename T>
__global__ void ReluV2Kernel(const size_t num, const T *x, T *y, uint32_t *mask) {
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < num; i += blockDim.x * gridDim.x) {
    T v = x[i];
    bool p = v > static_cast<T>(0);
    y[i] = p ? v : static_cast<T>(0);

    auto warp_predict = BallotSync(p, __activemask());
    if (LaneId() == 0) {
      mask[WarpId(i)] = warp_predict;
    }
  }
}

template <typename T>
void ReluV2(const size_t num, const T *x, T *y, uint32_t *mask, hipStream_t hip_stream) {
  ReluV2Kernel<<<kBlocksPerGrid(num), kThreadsPerBlock, 0, hip_stream>>>(num, x, y, mask);
}

template <typename T>
__global__ void ReluGradV2Kernel(const size_t num, const T *dy, const uint32_t *mask, T *dx) {
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < num; i += blockDim.x * gridDim.x) {
    bool p = mask[WarpId(i)] & (1 << LaneId());
    dx[i] = p ? dy[i] : static_cast<T>(0);
  }
}

template <typename T>
void ReluGradV2(const size_t num, const T *dy, const uint32_t *mask, T *dx, hipStream_t hip_stream) {
  ReluGradV2Kernel<<<kBlocksPerGrid(num), kThreadsPerBlock, 0, hip_stream>>>(num, dy, mask, dx);
}

template CUDA_LIB_EXPORT void ReluV2(const size_t num, const double *x, double *y, uint32_t *mask,
                                     hipStream_t hip_stream);
template CUDA_LIB_EXPORT void ReluV2(const size_t num, const float *x, float *y, uint32_t *mask,
                                     hipStream_t hip_stream);
template CUDA_LIB_EXPORT void ReluV2(const size_t num, const half *x, half *y, uint32_t *mask,
                                     hipStream_t hip_stream);
template CUDA_LIB_EXPORT void ReluV2(const size_t num, const int8_t *x, int8_t *y, uint32_t *mask,
                                     hipStream_t hip_stream);
template CUDA_LIB_EXPORT void ReluV2(const size_t num, const int16_t *x, int16_t *y, uint32_t *mask,
                                     hipStream_t hip_stream);
template CUDA_LIB_EXPORT void ReluV2(const size_t num, const int32_t *x, int32_t *y, uint32_t *mask,
                                     hipStream_t hip_stream);
template CUDA_LIB_EXPORT void ReluV2(const size_t num, const int64_t *x, int64_t *y, uint32_t *mask,
                                     hipStream_t hip_stream);
template CUDA_LIB_EXPORT void ReluV2(const size_t num, const uint8_t *x, uint8_t *y, uint32_t *mask,
                                     hipStream_t hip_stream);

template CUDA_LIB_EXPORT void ReluGradV2(const size_t num, const double *dy, const uint32_t *mask, double *dx,
                                         hipStream_t hip_stream);
template CUDA_LIB_EXPORT void ReluGradV2(const size_t num, const float *dy, const uint32_t *mask, float *dx,
                                         hipStream_t hip_stream);
template CUDA_LIB_EXPORT void ReluGradV2(const size_t num, const half *dy, const uint32_t *mask, half *dx,
                                         hipStream_t hip_stream);
template CUDA_LIB_EXPORT void ReluGradV2(const size_t num, const int8_t *dy, const uint32_t *mask, int8_t *dx,
                                         hipStream_t hip_stream);
template CUDA_LIB_EXPORT void ReluGradV2(const size_t num, const int16_t *dy, const uint32_t *mask, int16_t *dx,
                                         hipStream_t hip_stream);
template CUDA_LIB_EXPORT void ReluGradV2(const size_t num, const int32_t *dy, const uint32_t *mask, int32_t *dx,
                                         hipStream_t hip_stream);
template CUDA_LIB_EXPORT void ReluGradV2(const size_t num, const int64_t *dy, const uint32_t *mask, int64_t *dx,
                                         hipStream_t hip_stream);
template CUDA_LIB_EXPORT void ReluGradV2(const size_t num, const uint8_t *dy, const uint32_t *mask, uint8_t *dx,
                                         hipStream_t hip_stream);

