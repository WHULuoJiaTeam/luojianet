#include "hip/hip_runtime.h"
/**
 * Copyright 2019 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <stdio.h>
#include <stdint.h>
#include <math.h>
#include <algorithm>
#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/resize_nearest_neighbor_impl.cuh"
#include "include/hip/hip_fp16.h"

template <typename T>
__global__ void ResizeNearestNeighbor(const int size, const T *input, const int s1, const int s2, const int s3,
                                      const int s4, T *output, const int d1, const int d2, const int d3, const int d4,
                                      bool align_corners, float h_scale, float w_scale) {
  // initialization
  // HalfPixelCenters false
  int input_pos;
  int pos_array[RESIZENEARESTNEIGHBOR_DIMENSION];
  int in_height = s3;
  int in_width = s4;
  // for example 4-D: pos = pos_array[0] * output_shape[1] * output_shape[2] * output_shape[3] +
  //                        pos_array[1] * output_shape[2] * output_shape[3] +
  //                        pos_array[2] * output_shape[3] +
  //                        pos_array[3]
  int out_h;
  int out_w;
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < (size); pos += blockDim.x * gridDim.x) {
    pos_array[0] = pos / (d2 * d3 * d4) % d1;
    pos_array[1] = pos / (d3 * d4) % d2;
    pos_array[2] = pos / (d4) % d3;
    pos_array[3] = pos % d4;
    out_h = pos_array[2];
    out_w = pos_array[3];
    const int in_y =
      min((align_corners) ? static_cast<int>(roundf(out_h * h_scale)) : static_cast<int>(floorf(out_h * h_scale)),
          in_height - 1);
    const int in_x =
      min((align_corners) ? static_cast<int>(roundf(out_w * w_scale)) : static_cast<int>(floorf(out_w * w_scale)),
          in_width - 1);
    // pos_array[0] N, pos_array[1] C, in_y H, in_x W
    input_pos = pos_array[0] * s2 * s3 * s4 + pos_array[1] * s3 * s4 + in_y * s4 + in_x;
    output[pos] = input[input_pos];
  }
  return;
}

template <typename T>
void CalResizeNearestNeighbor(const int size, const T *input, const int s1, const int s2, const int s3, const int s4,
                              T *output, const int d1, const int d2, const int d3, const int d4, bool align_corners,
                              float h_scale, float w_scale, hipStream_t hip_stream) {
  ResizeNearestNeighbor<<<GET_BLOCKS(size), GET_THREADS, 0, hip_stream>>>(size, input, s1, s2, s3, s4, output, d1, d2,
                                                                           d3, d4, align_corners, h_scale, w_scale);
  return;
}

template CUDA_LIB_EXPORT void CalResizeNearestNeighbor<float>(const int size, const float *input, const int s1,
                                                              const int s2, const int s3, const int s4, float *output,
                                                              const int d1, const int d2, const int d3, const int d4,
                                                              bool align_corners, float h_scale, float w_scale,
                                                              hipStream_t hip_stream);
template CUDA_LIB_EXPORT void CalResizeNearestNeighbor<half>(const int size, const half *input, const int s1,
                                                             const int s2, const int s3, const int s4, half *output,
                                                             const int d1, const int d2, const int d3, const int d4,
                                                             bool align_corners, float h_scale, float w_scale,
                                                             hipStream_t hip_stream);
template CUDA_LIB_EXPORT void CalResizeNearestNeighbor<int>(const int size, const int *input, const int s1,
                                                            const int s2, const int s3, const int s4, int *output,
                                                            const int d1, const int d2, const int d3, const int d4,
                                                            bool align_corners, float h_scale, float w_scale,
                                                            hipStream_t hip_stream);
