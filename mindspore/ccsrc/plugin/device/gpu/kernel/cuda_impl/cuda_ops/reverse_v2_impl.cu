#include "hip/hip_runtime.h"
/**
 * Copyright 2021 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <hip/hip_runtime.h>
#include "reverse_v2_impl.cuh"
#include "include/hip/hip_fp16.h"
template <typename T>
__global__ void ReverseV2(const T* input, T* output, const size_t* input_shape, const int64_t* strides,
                          const int64_t* axis, size_t input_size, size_t axis_size) {
  for (int64_t gt_id = blockIdx.x * blockDim.x + threadIdx.x; gt_id < input_size; gt_id += blockDim.x * gridDim.x) {
    int64_t intermediate_index = gt_id;
    for (size_t i = 0; i < axis_size; i++) {
      int64_t d = axis[i];
      int64_t pre_reverse_position = (gt_id / strides[d]) % input_shape[d];
      int64_t reversed_position = input_shape[d] - pre_reverse_position - 1;
      intermediate_index += ((reversed_position - pre_reverse_position) * strides[d]);
    }

    output[intermediate_index] = input[gt_id];
  }
  return;
}
template <typename T>
void CalReverseV2(const T* input, T* output, const size_t* input_shape, const int64_t* strides, const int64_t* axis,
                  size_t input_size, size_t axis_size, hipStream_t hip_stream) {
  ReverseV2<<<GET_BLOCKS(input_size), GET_THREADS, 0, hip_stream>>>(input, output, input_shape, strides, axis,
                                                                     input_size, axis_size);
  return;
}

template CUDA_LIB_EXPORT void CalReverseV2<half>(const half* input, half* output, const size_t* input_shape,
                                                 const int64_t* strides, const int64_t* axis, size_t input_size,
                                                 size_t axis_size, hipStream_t hip_stream);

template CUDA_LIB_EXPORT void CalReverseV2<float>(const float* input, float* output, const size_t* input_shape,
                                                  const int64_t* strides, const int64_t* axis, size_t input_size,
                                                  size_t axis_size, hipStream_t hip_stream);

template CUDA_LIB_EXPORT void CalReverseV2<uint8_t>(const uint8_t* input, uint8_t* output, const size_t* input_shape,
                                                    const int64_t* strides, const int64_t* axis, size_t input_size,
                                                    size_t axis_size, hipStream_t hip_stream);

template CUDA_LIB_EXPORT void CalReverseV2<int16_t>(const int16_t* input, int16_t* output, const size_t* input_shape,
                                                    const int64_t* strides, const int64_t* axis, size_t input_size,
                                                    size_t axis_size, hipStream_t hip_stream);

template CUDA_LIB_EXPORT void CalReverseV2<int32_t>(const int32_t* input, int32_t* output, const size_t* input_shape,
                                                    const int64_t* strides, const int64_t* axis, size_t input_size,
                                                    size_t axis_size, hipStream_t hip_stream);

template CUDA_LIB_EXPORT void CalReverseV2<int64_t>(const int64_t* input, int64_t* output, const size_t* input_shape,
                                                    const int64_t* strides, const int64_t* axis, size_t input_size,
                                                    size_t axis_size, hipStream_t hip_stream);
