#include "hip/hip_runtime.h"
/**
 * Copyright 2020-2021 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/sigmoid_cross_entropy_with_logits_impl.cuh"
#include "include/hip/hip_fp16.h"

template <typename T, typename S>
__global__ void SigmoidCrossEntropyWithLogitsKernel(const size_t size, const T *logits, const S *labels, T *outputs) {
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < size; i += gridDim.x * blockDim.x) {
    const T reverse_factor = static_cast<T>(logits[i] >= 0);
    outputs[i] =
      log1p(exp(logits[i] - static_cast<T>(2) * reverse_factor * logits[i])) - logits[i] * (labels[i] - reverse_factor);
  }
}

template <>
__global__ void SigmoidCrossEntropyWithLogitsKernel(const size_t size, const half *logits,
                                                    const half *labels, half *outputs) {
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < size; i += gridDim.x * blockDim.x) {
    const half reverse_factor = static_cast<half>(logits[i] >= static_cast<half>(0.));
    const float exp_logit = exp(__half2float(logits[i] - static_cast<half>(2) * reverse_factor * logits[i]));
    outputs[i] = __float2half(log1p(exp_logit)) - logits[i] * (labels[i] - reverse_factor);
  }
}

template <typename T, typename S>
void SigmoidCrossEntropyWithLogits(const size_t size, const T *logits, const S *labels, T *outputs,
                                   hipStream_t hip_stream) {
  SigmoidCrossEntropyWithLogitsKernel<<<GET_BLOCKS(size), GET_THREADS, 0, hip_stream>>>(size, logits, labels, outputs);
}

template CUDA_LIB_EXPORT void SigmoidCrossEntropyWithLogits<half, half>(const size_t size, const half *logits,
                                                                        const half *labels, half *outputs,
                                                                        hipStream_t hip_stream);
template CUDA_LIB_EXPORT void SigmoidCrossEntropyWithLogits<float, float>(const size_t size, const float *logits,
                                                                          const float *labels, float *outputs,
                                                                          hipStream_t hip_stream);
template CUDA_LIB_EXPORT void SigmoidCrossEntropyWithLogits<double, double>(const size_t size, const double *logits,
                                                                            const double *labels, double *outputs,
                                                                            hipStream_t hip_stream);
