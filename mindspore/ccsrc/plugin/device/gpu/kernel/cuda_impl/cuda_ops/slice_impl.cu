#include "hip/hip_runtime.h"
/**
 * Copyright 2019-2021 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>
#include <algorithm>
#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/slice_impl.cuh"
#include "include/hip/hip_fp16.h"

template <typename T>
__global__ void Slice1D(const size_t s1, const size_t l1, const size_t d1, const T *input, T *output) {
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < l1; pos += blockDim.x * gridDim.x) {
    output[pos] = input[pos + s1];
  }
}

template <typename T>
__global__ void Slice2D(const size_t s1, const size_t s2, const size_t l1, const size_t l2, const size_t d1,
                        const size_t d2, const T *input, T *output) {
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < l1 * l2; pos += blockDim.x * gridDim.x) {
    size_t i = pos / l2 % l1;
    size_t j = pos % l2;

    size_t offset = (i + s1) * d2 + (j + s2);
    output[pos] = input[offset];
  }
}

template <typename T>
__global__ void Slice3D(const size_t s1, const size_t s2, const size_t s3, const size_t l1, const size_t l2,
                        const size_t l3, const size_t d1, const size_t d2, const size_t d3, const T *input, T *output) {
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < l1 * l2 * l3; pos += blockDim.x * gridDim.x) {
    size_t i = pos / (l2 * l3) % l1;
    size_t j = pos / l3 % l2;
    size_t k = pos % l3;

    size_t offset = (i + s1) * (d2 * d3) + (j + s2) * d3 + (k + s3);
    output[pos] = input[offset];
  }
}

template <typename T>
__global__ void Slice4D(const size_t s1, const size_t s2, const size_t s3, const size_t s4, const size_t l1,
                        const size_t l2, const size_t l3, const size_t l4, const size_t d1, const size_t d2,
                        const size_t d3, const size_t d4, const T *input, T *output) {
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < l1 * l2 * l3 * l4; pos += blockDim.x * gridDim.x) {
    size_t i = pos / (l2 * l3 * l4) % l1;
    size_t j = pos / (l3 * l4) % l2;
    size_t k = pos / l4 % l3;
    size_t o = pos % l4;

    size_t offset = (i + s1) * (d2 * d3 * d4) + (j + s2) * (d3 * d4) + (k + s3) * d4 + (o + s4);
    output[pos] = input[offset];
  }
}

template <typename T>
__global__ void Slice5D(const size_t s1, const size_t s2, const size_t s3, const size_t s4, const size_t s5,
                        const size_t l1, const size_t l2, const size_t l3, const size_t l4, const size_t l5,
                        const size_t d1, const size_t d2, const size_t d3, const size_t d4, const size_t d5,
                        const T *input, T *output) {
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < l1 * l2 * l3 * l4 * l5;
       pos += blockDim.x * gridDim.x) {
    size_t i = pos / (l2 * l3 * l4 * l5) % l1;
    size_t j = pos / (l3 * l4 * l5) % l2;
    size_t k = pos / (l4 * l5) % l3;
    size_t o = pos / l5 % l4;
    size_t q = pos % l5;

    size_t offset =
      (i + s1) * (d2 * d3 * d4 * d5) + (j + s2) * (d3 * d4 * d5) + (k + s3) * (d4 * d5) + (o + s4) * d5 + (q + s5);
    output[pos] = input[offset];
  }
}

template <typename T>
__global__ void Slice6D(const size_t s1, const size_t s2, const size_t s3, const size_t s4, const size_t s5,
                        const size_t s6, const size_t l1, const size_t l2, const size_t l3, const size_t l4,
                        const size_t l5, const size_t l6, const size_t d1, const size_t d2, const size_t d3,
                        const size_t d4, const size_t d5, const size_t d6, const T *input, T *output) {
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < l1 * l2 * l3 * l4 * l5 * l6;
       pos += blockDim.x * gridDim.x) {
    size_t i = pos / (l2 * l3 * l4 * l5 * l6) % l1;
    size_t j = pos / (l3 * l4 * l5 * l6) % l2;
    size_t k = pos / (l4 * l5 * l6) % l3;
    size_t o = pos / (l5 * l6) % l4;
    size_t q = pos / l6 % l5;
    size_t r = pos % l6;

    size_t offset =
      (i + s1) * (d2 * d3 * d4 * d5 * d6) + (j + s2) * (d3 * d4 * d5 * d6) + (k + s3) * (d4 * d5 * d6) + (o + s4) *
      (d5 * d6) + (q + s5) * d6 + (r + s6);
    output[pos] = input[offset];
  }
}

template <typename T>
__global__ void Slice7D(const size_t s1, const size_t s2, const size_t s3, const size_t s4, const size_t s5,
                        const size_t s6, const size_t s7, const size_t l1, const size_t l2, const size_t l3,
                        const size_t l4, const size_t l5, const size_t l6, const size_t l7, const size_t d1,
                        const size_t d2, const size_t d3, const size_t d4, const size_t d5, const size_t d6,
                        const size_t d7, const T *input, T *output) {
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < l1 * l2 * l3 * l4 * l5 * l6 * l7;
       pos += blockDim.x * gridDim.x) {
    size_t i = pos / (l2 * l3 * l4 * l5 * l6 * l7) % l1;
    size_t j = pos / (l3 * l4 * l5 * l6 * l7) % l2;
    size_t k = pos / (l4 * l5 * l6 * l7) % l3;
    size_t o = pos / (l5 * l6 * l7) % l4;
    size_t q = pos / (l6 * l7) % l5;
    size_t r = pos / l7 % l6;
    size_t s = pos % l7;

    size_t offset =
      (i + s1) * (d2 * d3 * d4 * d5 * d6 * d7) + (j + s2) * (d3 * d4 * d5 * d6 * d7) + (k + s3) * (d4 * d5 * d6 * d7)+
      (o + s4) * (d5 * d6 * d7) + (q + s5) * (d6 * d7) + (r + s6) * d7 + (s + s7);
    output[pos] = input[offset];
  }
}

template <typename T>
__global__ void Slice4DGrad(const size_t s1, const size_t s2, const size_t s3, const size_t s4,
                        const size_t l1, const size_t l2, const size_t l3, const size_t l4,
                        const size_t d1, const size_t d2, const size_t d3, const size_t d4,
                        const T *dy, T *dx) {
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < (l1 * l2 * l3 * l4); pos += blockDim.x * gridDim.x) {
    size_t i = pos / (l2 * l3 * l4) % l1;
    size_t j = pos / (l3 * l4) % l2;
    size_t k = pos / l4 % l3;
    size_t o = pos % l4;
    size_t input_idx = (i + s1) * (d2 * d3 * d4) + (j + s2) * (d3 * d4) + (k + s3) * d4 + (o + s4);
    dx[input_idx] = dy[pos];
  }
}

template <typename T>
__global__ void FillArray(T *addr, const size_t len, const float value) {
  T value_ = static_cast<T>(value);
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < len; pos += blockDim.x * gridDim.x) {
    addr[pos] = value_;
  }
  return;
}
template <typename T>
void FillDeviceArray(const size_t input_size, T *addr, const float value, hipStream_t hip_stream) {
  FillArray<<<GET_BLOCKS(input_size), GET_THREADS, 0, hip_stream>>>(addr, input_size, value);
  return;
}
template <typename T>
void Slice1DKernel(const size_t s1, const size_t l1, const size_t d1, const T *input, T *output, hipStream_t stream) {
  Slice1D<<<GET_BLOCKS(l1), GET_THREADS, 0, stream>>>(s1, l1, d1, input, output);
}
template <typename T>
void Slice2DKernel(const size_t s1, const size_t s2, const size_t l1, const size_t l2, const size_t d1, const size_t d2,
                   const T *input, T *output, hipStream_t stream) {
  Slice2D<<<GET_BLOCKS(l1 * l2), GET_THREADS, 0, stream>>>(s1, s2, l1, l2, d1, d2, input, output);
}
template <typename T>
void Slice3DKernel(const size_t s1, const size_t s2, const size_t s3, const size_t l1, const size_t l2, const size_t l3,
                   const size_t d1, const size_t d2, const size_t d3, const T *input, T *output, hipStream_t stream) {
  Slice3D<<<GET_BLOCKS(l1 * l2 * l3), GET_THREADS, 0, stream>>>(s1, s2, s3, l1, l2, l3, d1, d2, d3, input, output);
}
template <typename T>
void Slice4DKernel(const size_t s1, const size_t s2, const size_t s3, const size_t s4, const size_t l1, const size_t l2,
                   const size_t l3, const size_t l4, const size_t d1, const size_t d2, const size_t d3, const size_t d4,
                   const T *input, T *output, hipStream_t stream) {
  Slice4D<<<GET_BLOCKS(l1 * l2 * l3 * l4), GET_THREADS, 0, stream>>>(s1, s2, s3, s4, l1, l2, l3, l4, d1, d2, d3, d4,
                                                                     input, output);
}
template <typename T>
void Slice5DKernel(const size_t s1, const size_t s2, const size_t s3, const size_t s4, const size_t s5, const size_t l1,
                   const size_t l2, const size_t l3, const size_t l4, const size_t l5, const size_t d1, const size_t d2,
                   const size_t d3, const size_t d4, const size_t d5, const T *input, T *output, hipStream_t stream) {
  Slice5D<<<GET_BLOCKS(l1 * l2 * l3 * l4 * l5), GET_THREADS, 0, stream>>>(s1, s2, s3, s4, s5, l1, l2, l3, l4, l5, d1,
                                                                          d2, d3, d4, d5, input, output);
}
template <typename T>
void Slice6DKernel(const size_t s1, const size_t s2, const size_t s3, const size_t s4, const size_t s5, const size_t s6,
                   const size_t l1, const size_t l2, const size_t l3, const size_t l4, const size_t l5, const size_t l6,
                   const size_t d1, const size_t d2, const size_t d3, const size_t d4, const size_t d5, const size_t d6,
                   const T *input, T *output, hipStream_t stream) {
  Slice6D<<<GET_BLOCKS(l1 * l2 * l3 * l4 * l5 * l6), GET_THREADS, 0, stream>>>(s1, s2, s3, s4, s5, s6, l1, l2, l3, l4,
                                                                               l5, l6, d1, d2, d3, d4, d5, d6, input,
                                                                               output);
}
template <typename T>
void Slice7DKernel(const size_t s1, const size_t s2, const size_t s3, const size_t s4, const size_t s5, const size_t s6,
                   const size_t s7, const size_t l1, const size_t l2, const size_t l3, const size_t l4, const size_t l5,
                   const size_t l6, const size_t l7, const size_t d1, const size_t d2, const size_t d3, const size_t d4,
                   const size_t d5, const size_t d6, const size_t d7, const T *input, T *output, hipStream_t stream) {
  Slice7D<<<GET_BLOCKS(l1 * l2 * l3 * l4 * l5 * l6 * l7), GET_THREADS, 0, stream>>>(s1, s2, s3, s4, s5, s6, s7, l1, l2,
                                                                                    l3, l4, l5, l6, l7, d1, d2, d3, d4,
                                                                                    d5, d6, d7, input, output);
}
template <typename T>
void CalSlice4DGrad(const size_t s1, const size_t s2, const size_t s3, const size_t s4, const size_t l1,
                   const size_t l2, const size_t l3, const size_t l4, const size_t d1, const size_t d2,
                   const size_t d3, const size_t d4, const T *dy, T *dx, hipStream_t stream) {
  Slice4DGrad<<<GET_BLOCKS(l1 * l2 * l3 * l4), GET_THREADS, 0, stream>>>(s1, s2, s3, s4, l1, l2, l3, l4, d1, d2, d3, d4,
                                                                     dy, dx);
}

template <typename T>
__global__ void StridedSliceKernel(const size_t b0, const size_t b1, const size_t b2, const size_t b3, const size_t b4,
                                   const size_t b5, const size_t b6, const size_t s0, const size_t s1, const size_t s2,
                                   const size_t s3, const size_t s4, const size_t s5, const size_t s6, const size_t i0,
                                   const size_t i1, const size_t i2, const size_t i3, const size_t i4, const size_t i5,
                                   const size_t i6, const size_t o0, const size_t o1, const size_t o2, const size_t o3,
                                   const size_t o4, const size_t o5, const size_t o6, const T *input_addr,
                                   T *output_addr) {
  size_t output_num = o0 * o1 * o2 * o3 * o4 * o5 * o6;
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < output_num; pos += blockDim.x * gridDim.x) {
    size_t i = pos / (o1 * o2 * o3 * o4 * o5 * o6) % o0;
    size_t j = pos / (o2 * o3 * o4 * o5 * o6) % o1;
    size_t k = pos / (o3 * o4 * o5 * o6) % o2;
    size_t l = pos / (o4 * o5 * o6) % o3;
    size_t m = pos / (o5 * o6) % o4;
    size_t n = pos / (o6) % o5;
    size_t o = pos % o6;

    size_t input_idx = (i * s0 + b0) * i1 * i2 * i3 * i4 * i5 * i6 + (j * s1 + b1) * i2 * i3 * i4 * i5 * i6 +
                       (k * s2 + b2) * i3 * i4 * i5 * i6 + (l * s3 + b3) * i4 * i5 * i6 + (m * s4 + b4) * i5 * i6 +
                       (n * s5 + b5) * i6 + (o * s6 + b6);
    output_addr[pos] = input_addr[input_idx];
  }
}

template <typename T>
void StridedSlice(const std::vector<size_t> &input_shape, const std::vector<int64_t> &begin,
                  const std::vector<int64_t> &strides, const std::vector<size_t> &output_shape, const T *input,
                  T *output, hipStream_t hip_stream) {
  size_t size = output_shape[0] * output_shape[1] * output_shape[2] * output_shape[3] * output_shape[4] *
                output_shape[5] * output_shape[6];
  StridedSliceKernel<<<GET_BLOCKS(size), GET_THREADS, 0, hip_stream>>>(
    begin[0], begin[1], begin[2], begin[3], begin[4], begin[5], begin[6], strides[0], strides[1], strides[2],
    strides[3], strides[4], strides[5], strides[6], input_shape[0], input_shape[1], input_shape[2], input_shape[3],
    input_shape[4], input_shape[5], input_shape[6], output_shape[0], output_shape[1], output_shape[2], output_shape[3],
    output_shape[4], output_shape[5], output_shape[6], input, output);
}

template <typename T>
__global__ void StridedSliceGradKernel(const size_t b0, const size_t b1, const size_t b2, const size_t b3,
                                       const size_t b4, const size_t b5, const size_t b6, const size_t s0,
                                       const size_t s1, const size_t s2, const size_t s3, const size_t s4,
                                       const size_t s5, const size_t s6, const size_t i0, const size_t i1,
                                       const size_t i2, const size_t i3, const size_t i4, const size_t i5,
                                       const size_t i6, const size_t o0, const size_t o1, const size_t o2,
                                       const size_t o3, const size_t o4, const size_t o5, const size_t o6, const T *dy,
                                       T *dx) {
  size_t output_num = o0 * o1 * o2 * o3 * o4 * o5 * o6;
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < output_num; pos += blockDim.x * gridDim.x) {
    size_t i = pos / (o1 * o2 * o3 * o4 * o5 * o6) % o0;
    size_t j = pos / (o2 * o3 * o4 * o5 * o6) % o1;
    size_t k = pos / (o3 * o4 * o5 * o6) % o2;
    size_t l = pos / (o4 * o5 * o6) % o3;
    size_t m = pos / (o5 * o6) % o4;
    size_t n = pos / (o6) % o5;
    size_t o = pos % o6;

    size_t input_idx = (i * s0 + b0) * i1 * i2 * i3 * i4 * i5 * i6 + (j * s1 + b1) * i2 * i3 * i4 * i5 * i6 +
                       (k * s2 + b2) * i3 * i4 * i5 * i6 + (l * s3 + b3) * i4 * i5 * i6 + (m * s4 + b4) * i5 * i6 +
                       (n * s5 + b5) * i6 + (o * s6 + b6);
                       (n * s5 + b5) * i6 + (o * s6 + b6);
    dx[input_idx] = dy[pos];
  }
  return;
}

template <typename T>
void StridedSliceGrad(const std::vector<size_t> &dy_shape, const std::vector<int64_t> &begin,
                      const std::vector<int64_t> &strides, const std::vector<size_t> &dx_shape, const T *dy, T *dx,
                      hipStream_t hip_stream) {
  size_t size = dy_shape[0] * dy_shape[1] * dy_shape[2] * dy_shape[3] * dy_shape[4] * dy_shape[5] * dy_shape[6];
  StridedSliceGradKernel<<<GET_BLOCKS(size), GET_THREADS, 0, hip_stream>>>(
    begin[0], begin[1], begin[2], begin[3], begin[4], begin[5], begin[6], strides[0], strides[1], strides[2],
    strides[3], strides[4], strides[5], strides[6], dx_shape[0], dx_shape[1], dx_shape[2], dx_shape[3], dx_shape[4],
    dx_shape[5], dx_shape[6], dy_shape[0], dy_shape[1], dy_shape[2], dy_shape[3], dy_shape[4], dy_shape[5], dy_shape[6],
    dy, dx);
}

template CUDA_LIB_EXPORT void Slice1DKernel(const size_t s1, const size_t l1, const size_t d1, const double *input,
                                            double *output, hipStream_t stream);
template CUDA_LIB_EXPORT void Slice1DKernel(const size_t s1, const size_t l1, const size_t d1, const float *input,
                                            float *output, hipStream_t stream);
template CUDA_LIB_EXPORT void Slice1DKernel(const size_t s1, const size_t l1, const size_t d1, const half *input,
                                            half *output, hipStream_t stream);
template CUDA_LIB_EXPORT void Slice1DKernel(const size_t s1, const size_t l1, const size_t d1, const int *input,
                                            int *output, hipStream_t stream);
template CUDA_LIB_EXPORT void Slice1DKernel(const size_t s1, const size_t l1, const size_t d1, const short *input,  // NOLINT
                                            short *output, hipStream_t stream);  // NOLINT
template CUDA_LIB_EXPORT void Slice1DKernel(const size_t s1, const size_t l1, const size_t d1,
                                            const unsigned char *input, unsigned char *output, hipStream_t stream);
template CUDA_LIB_EXPORT void Slice1DKernel(const size_t s1, const size_t l1, const size_t d1, const int64_t *input,
                                            int64_t *output, hipStream_t stream);
template CUDA_LIB_EXPORT void Slice1DKernel(const size_t s1, const size_t l1, const size_t d1, const bool *input,
                                            bool *output, hipStream_t stream);

template CUDA_LIB_EXPORT void Slice2DKernel(const size_t s1, const size_t s2, const size_t l1, const size_t l2,
                                            const size_t d1, const size_t d2, const double *input, double *output,
                                            hipStream_t stream);
template CUDA_LIB_EXPORT void Slice2DKernel(const size_t s1, const size_t s2, const size_t l1, const size_t l2,
                                            const size_t d1, const size_t d2, const float *input, float *output,
                                            hipStream_t stream);
template CUDA_LIB_EXPORT void Slice2DKernel(const size_t s1, const size_t s2, const size_t l1, const size_t l2,
                                            const size_t d1, const size_t d2, const half *input, half *output,
                                            hipStream_t stream);
template CUDA_LIB_EXPORT void Slice2DKernel(const size_t s1, const size_t s2, const size_t l1, const size_t l2,
                                            const size_t d1, const size_t d2, const int *input, int *output,
                                            hipStream_t stream);
template CUDA_LIB_EXPORT void Slice2DKernel(const size_t s1, const size_t s2, const size_t l1, const size_t l2,
                                            const size_t d1, const size_t d2, const short *input, short *output,  // NOLINT
                                            hipStream_t stream);
template CUDA_LIB_EXPORT void Slice2DKernel(const size_t s1, const size_t s2, const size_t l1, const size_t l2,
                                            const size_t d1, const size_t d2, const unsigned char *input,
                                            unsigned char *output, hipStream_t stream);
template CUDA_LIB_EXPORT void Slice2DKernel(const size_t s1, const size_t s2, const size_t l1, const size_t l2,
                                            const size_t d1, const size_t d2, const int64_t *input, int64_t *output,
                                            hipStream_t stream);
template CUDA_LIB_EXPORT void Slice2DKernel(const size_t s1, const size_t s2, const size_t l1, const size_t l2,
                                            const size_t d1, const size_t d2, const bool *input, bool *output,
                                            hipStream_t stream);

template CUDA_LIB_EXPORT void Slice3DKernel(const size_t s1, const size_t s2, const size_t s3, const size_t l1,
                                            const size_t l2, const size_t l3, const size_t d1, const size_t d2,
                                            const size_t d3, const double *input, double *output, hipStream_t stream);
template CUDA_LIB_EXPORT void Slice3DKernel(const size_t s1, const size_t s2, const size_t s3, const size_t l1,
                                            const size_t l2, const size_t l3, const size_t d1, const size_t d2,
                                            const size_t d3, const float *input, float *output, hipStream_t stream);
template CUDA_LIB_EXPORT void Slice3DKernel(const size_t s1, const size_t s2, const size_t s3, const size_t l1,
                                            const size_t l2, const size_t l3, const size_t d1, const size_t d2,
                                            const size_t d3, const half *input, half *output, hipStream_t stream);
template CUDA_LIB_EXPORT void Slice3DKernel(const size_t s1, const size_t s2, const size_t s3, const size_t l1,
                                            const size_t l2, const size_t l3, const size_t d1, const size_t d2,
                                            const size_t d3, const int *input, int *output, hipStream_t stream);
template CUDA_LIB_EXPORT void Slice3DKernel(const size_t s1, const size_t s2, const size_t s3, const size_t l1,
                                            const size_t l2, const size_t l3, const size_t d1, const size_t d2,
                                            const size_t d3, const short *input, short *output, hipStream_t stream);  // NOLINT
template CUDA_LIB_EXPORT void Slice3DKernel(const size_t s1, const size_t s2, const size_t s3, const size_t l1,
                                            const size_t l2, const size_t l3, const size_t d1, const size_t d2,
                                            const size_t d3, const unsigned char *input, unsigned char *output,
                                            hipStream_t stream);
template CUDA_LIB_EXPORT void Slice3DKernel(const size_t s1, const size_t s2, const size_t s3, const size_t l1,
                                            const size_t l2, const size_t l3, const size_t d1, const size_t d2,
                                            const size_t d3, const int64_t *input, int64_t *output,
                                            hipStream_t stream);
template CUDA_LIB_EXPORT void Slice3DKernel(const size_t s1, const size_t s2, const size_t s3, const size_t l1,
                                            const size_t l2, const size_t l3, const size_t d1, const size_t d2,
                                            const size_t d3, const bool *input, bool *output, hipStream_t stream);

template CUDA_LIB_EXPORT void Slice4DKernel(const size_t s1, const size_t s2, const size_t s3, const size_t s4,
                                            const size_t l1, const size_t l2, const size_t l3, const size_t l4,
                                            const size_t d1, const size_t d2, const size_t d3, const size_t d4,
                                            const double *input, double *output, hipStream_t stream);
template CUDA_LIB_EXPORT void Slice4DKernel(const size_t s1, const size_t s2, const size_t s3, const size_t s4,
                                            const size_t l1, const size_t l2, const size_t l3, const size_t l4,
                                            const size_t d1, const size_t d2, const size_t d3, const size_t d4,
                                            const float *input, float *output, hipStream_t stream);
template CUDA_LIB_EXPORT void Slice4DKernel(const size_t s1, const size_t s2, const size_t s3, const size_t s4,
                                            const size_t l1, const size_t l2, const size_t l3, const size_t l4,
                                            const size_t d1, const size_t d2, const size_t d3, const size_t d4,
                                            const half *input, half *output, hipStream_t stream);
template CUDA_LIB_EXPORT void Slice4DKernel(const size_t s1, const size_t s2, const size_t s3, const size_t s4,
                                            const size_t l1, const size_t l2, const size_t l3, const size_t l4,
                                            const size_t d1, const size_t d2, const size_t d3, const size_t d4,
                                            const int *input, int *output, hipStream_t stream);
template CUDA_LIB_EXPORT void Slice4DKernel(const size_t s1, const size_t s2, const size_t s3, const size_t s4,
                                            const size_t l1, const size_t l2, const size_t l3, const size_t l4,
                                            const size_t d1, const size_t d2, const size_t d3, const size_t d4,
                                            const short *input, short *output, hipStream_t stream);  // NOLINT
template CUDA_LIB_EXPORT void Slice4DKernel(const size_t s1, const size_t s2, const size_t s3, const size_t s4,
                                            const size_t l1, const size_t l2, const size_t l3, const size_t l4,
                                            const size_t d1, const size_t d2, const size_t d3, const size_t d4,
                                            const unsigned char *input, unsigned char *output, hipStream_t stream);
template CUDA_LIB_EXPORT void Slice4DKernel(const size_t s1, const size_t s2, const size_t s3, const size_t s4,
                                            const size_t l1, const size_t l2, const size_t l3, const size_t l4,
                                            const size_t d1, const size_t d2, const size_t d3, const size_t d4,
                                            const int64_t *input, int64_t *output, hipStream_t stream);
template CUDA_LIB_EXPORT void Slice4DKernel(const size_t s1, const size_t s2, const size_t s3, const size_t s4,
                                            const size_t l1, const size_t l2, const size_t l3, const size_t l4,
                                            const size_t d1, const size_t d2, const size_t d3, const size_t d4,
                                            const bool *input, bool *output, hipStream_t stream);

template CUDA_LIB_EXPORT void Slice5DKernel(const size_t s1, const size_t s2, const size_t s3, const size_t s4,
                                            const size_t s5, const size_t l1, const size_t l2, const size_t l3,
                                            const size_t l4, const size_t l5, const size_t d1, const size_t d2,
                                            const size_t d3, const size_t d4, const size_t d5, const double *input,
                                            double *output, hipStream_t stream);
template CUDA_LIB_EXPORT void Slice5DKernel(const size_t s1, const size_t s2, const size_t s3, const size_t s4,
                                            const size_t s5, const size_t l1, const size_t l2, const size_t l3,
                                            const size_t l4, const size_t l5, const size_t d1, const size_t d2,
                                            const size_t d3, const size_t d4, const size_t d5, const float *input,
                                            float *output, hipStream_t stream);
template CUDA_LIB_EXPORT void Slice5DKernel(const size_t s1, const size_t s2, const size_t s3, const size_t s4,
                                            const size_t s5, const size_t l1, const size_t l2, const size_t l3,
                                            const size_t l4, const size_t l5, const size_t d1, const size_t d2,
                                            const size_t d3, const size_t d4, const size_t d5, const half *input,
                                            half *output, hipStream_t stream);
template CUDA_LIB_EXPORT void Slice5DKernel(const size_t s1, const size_t s2, const size_t s3, const size_t s4,
                                            const size_t s5, const size_t l1, const size_t l2, const size_t l3,
                                            const size_t l4, const size_t l5, const size_t d1, const size_t d2,
                                            const size_t d3, const size_t d4, const size_t d5, const int64_t *input,
                                            int64_t *output, hipStream_t stream);
template CUDA_LIB_EXPORT void Slice5DKernel(const size_t s1, const size_t s2, const size_t s3, const size_t s4,
                                            const size_t s5, const size_t l1, const size_t l2, const size_t l3,
                                            const size_t l4, const size_t l5, const size_t d1, const size_t d2,
                                            const size_t d3, const size_t d4, const size_t d5, const int *input,
                                            int *output, hipStream_t stream);
template CUDA_LIB_EXPORT void Slice5DKernel(const size_t s1, const size_t s2, const size_t s3, const size_t s4,
                                            const size_t s5, const size_t l1, const size_t l2, const size_t l3,
                                            const size_t l4, const size_t l5, const size_t d1, const size_t d2,
                                            const size_t d3, const size_t d4, const size_t d5, const short *input,  // NOLINT
                                            short *output, hipStream_t stream);  // NOLINT
template CUDA_LIB_EXPORT void Slice5DKernel(const size_t s1, const size_t s2, const size_t s3, const size_t s4,
                                            const size_t s5, const size_t l1, const size_t l2, const size_t l3,
                                            const size_t l4, const size_t l5, const size_t d1, const size_t d2,
                                            const size_t d3, const size_t d4, const size_t d5,
                                            const unsigned char *input, unsigned char *output, hipStream_t stream);
template CUDA_LIB_EXPORT void Slice5DKernel(const size_t s1, const size_t s2, const size_t s3, const size_t s4,
                                            const size_t s5, const size_t l1, const size_t l2, const size_t l3,
                                            const size_t l4, const size_t l5, const size_t d1, const size_t d2,
                                            const size_t d3, const size_t d4, const size_t d5, const bool *input,
                                            bool *output, hipStream_t stream);

template CUDA_LIB_EXPORT void Slice6DKernel(const size_t s1, const size_t s2, const size_t s3, const size_t s4,
                                            const size_t s5, const size_t s6, const size_t l1, const size_t l2,
                                            const size_t l3, const size_t l4, const size_t l5, const size_t l6,
                                            const size_t d1, const size_t d2, const size_t d3, const size_t d4,
                                            const size_t d5, const size_t d6, const double *input, double *output,
                                            hipStream_t stream);
template CUDA_LIB_EXPORT void Slice6DKernel(const size_t s1, const size_t s2, const size_t s3, const size_t s4,
                                            const size_t s5, const size_t s6, const size_t l1, const size_t l2,
                                            const size_t l3, const size_t l4, const size_t l5, const size_t l6,
                                            const size_t d1, const size_t d2, const size_t d3, const size_t d4,
                                            const size_t d5, const size_t d6, const float *input, float *output,
                                            hipStream_t stream);
template CUDA_LIB_EXPORT void Slice6DKernel(const size_t s1, const size_t s2, const size_t s3, const size_t s4,
                                            const size_t s5, const size_t s6, const size_t l1, const size_t l2,
                                            const size_t l3, const size_t l4, const size_t l5, const size_t l6,
                                            const size_t d1, const size_t d2, const size_t d3, const size_t d4,
                                            const size_t d5, const size_t d6, const half *input, half *output,
                                            hipStream_t stream);
template CUDA_LIB_EXPORT void Slice6DKernel(const size_t s1, const size_t s2, const size_t s3, const size_t s4,
                                            const size_t s5, const size_t s6, const size_t l1, const size_t l2,
                                            const size_t l3, const size_t l4, const size_t l5, const size_t l6,
                                            const size_t d1, const size_t d2, const size_t d3, const size_t d4,
                                            const size_t d5, const size_t d6, const int64_t *input, int64_t *output,
                                            hipStream_t stream);
template CUDA_LIB_EXPORT void Slice6DKernel(const size_t s1, const size_t s2, const size_t s3, const size_t s4,
                                            const size_t s5, const size_t s6, const size_t l1, const size_t l2,
                                            const size_t l3, const size_t l4, const size_t l5, const size_t l6,
                                            const size_t d1, const size_t d2, const size_t d3, const size_t d4,
                                            const size_t d5, const size_t d6, const int *input, int *output,
                                            hipStream_t stream);
template CUDA_LIB_EXPORT void Slice6DKernel(const size_t s1, const size_t s2, const size_t s3, const size_t s4,
                                            const size_t s5, const size_t s6, const size_t l1, const size_t l2,
                                            const size_t l3, const size_t l4, const size_t l5, const size_t l6,
                                            const size_t d1, const size_t d2, const size_t d3, const size_t d4,
                                            const size_t d5, const size_t d6, const short *input, short *output,  // NOLINT
                                            hipStream_t stream);
template CUDA_LIB_EXPORT void Slice6DKernel(const size_t s1, const size_t s2, const size_t s3, const size_t s4,
                                            const size_t s5, const size_t s6, const size_t l1, const size_t l2,
                                            const size_t l3, const size_t l4, const size_t l5, const size_t l6,
                                            const size_t d1, const size_t d2, const size_t d3, const size_t d4,
                                            const size_t d5, const size_t d6, const unsigned char *input,
                                            unsigned char *output, hipStream_t stream);
template CUDA_LIB_EXPORT void Slice6DKernel(const size_t s1, const size_t s2, const size_t s3, const size_t s4,
                                            const size_t s5, const size_t s6, const size_t l1, const size_t l2,
                                            const size_t l3, const size_t l4, const size_t l5, const size_t l6,
                                            const size_t d1, const size_t d2, const size_t d3, const size_t d4,
                                            const size_t d5, const size_t d6, const bool *input, bool *output,
                                            hipStream_t stream);

template CUDA_LIB_EXPORT void Slice7DKernel(const size_t s1, const size_t s2, const size_t s3, const size_t s4,
                                            const size_t s5, const size_t s6, const size_t s7, const size_t l1,
                                            const size_t l2, const size_t l3, const size_t l4, const size_t l5,
                                            const size_t l6, const size_t l7, const size_t d1, const size_t d2,
                                            const size_t d3, const size_t d4, const size_t d5, const size_t d6,
                                            const size_t d7, const double *input, double *output, hipStream_t stream);
template CUDA_LIB_EXPORT void Slice7DKernel(const size_t s1, const size_t s2, const size_t s3, const size_t s4,
                                            const size_t s5, const size_t s6, const size_t s7, const size_t l1,
                                            const size_t l2, const size_t l3, const size_t l4, const size_t l5,
                                            const size_t l6, const size_t l7, const size_t d1, const size_t d2,
                                            const size_t d3, const size_t d4, const size_t d5, const size_t d6,
                                            const size_t d7, const float *input, float *output, hipStream_t stream);
template CUDA_LIB_EXPORT void Slice7DKernel(const size_t s1, const size_t s2, const size_t s3, const size_t s4,
                                            const size_t s5, const size_t s6, const size_t s7, const size_t l1,
                                            const size_t l2, const size_t l3, const size_t l4, const size_t l5,
                                            const size_t l6, const size_t l7, const size_t d1, const size_t d2,
                                            const size_t d3, const size_t d4, const size_t d5, const size_t d6,
                                            const size_t d7, const half *input, half *output, hipStream_t stream);
template CUDA_LIB_EXPORT void Slice7DKernel(const size_t s1, const size_t s2, const size_t s3, const size_t s4,
                                            const size_t s5, const size_t s6, const size_t s7, const size_t l1,
                                            const size_t l2, const size_t l3, const size_t l4, const size_t l5,
                                            const size_t l6, const size_t l7, const size_t d1, const size_t d2,
                                            const size_t d3, const size_t d4, const size_t d5, const size_t d6,
                                            const size_t d7, const int64_t *input, int64_t *output,
                                            hipStream_t stream);
template CUDA_LIB_EXPORT void Slice7DKernel(const size_t s1, const size_t s2, const size_t s3, const size_t s4,
                                            const size_t s5, const size_t s6, const size_t s7, const size_t l1,
                                            const size_t l2, const size_t l3, const size_t l4, const size_t l5,
                                            const size_t l6, const size_t l7, const size_t d1, const size_t d2,
                                            const size_t d3, const size_t d4, const size_t d5, const size_t d6,
                                            const size_t d7, const int *input, int *output, hipStream_t stream);
template CUDA_LIB_EXPORT void Slice7DKernel(const size_t s1, const size_t s2, const size_t s3, const size_t s4,
                                            const size_t s5, const size_t s6, const size_t s7, const size_t l1,
                                            const size_t l2, const size_t l3, const size_t l4, const size_t l5,
                                            const size_t l6, const size_t l7, const size_t d1, const size_t d2,
                                            const size_t d3, const size_t d4, const size_t d5, const size_t d6,
                                            const size_t d7, const short *input, short *output, hipStream_t stream);  // NOLINT
template CUDA_LIB_EXPORT void Slice7DKernel(const size_t s1, const size_t s2, const size_t s3, const size_t s4,
                                            const size_t s5, const size_t s6, const size_t s7, const size_t l1,
                                            const size_t l2, const size_t l3, const size_t l4, const size_t l5,
                                            const size_t l6, const size_t l7, const size_t d1, const size_t d2,
                                            const size_t d3, const size_t d4, const size_t d5, const size_t d6,
                                            const size_t d7, const unsigned char *input, unsigned char *output,
                                            hipStream_t stream);
template CUDA_LIB_EXPORT void Slice7DKernel(const size_t s1, const size_t s2, const size_t s3, const size_t s4,
                                            const size_t s5, const size_t s6, const size_t s7, const size_t l1,
                                            const size_t l2, const size_t l3, const size_t l4, const size_t l5,
                                            const size_t l6, const size_t l7, const size_t d1, const size_t d2,
                                            const size_t d3, const size_t d4, const size_t d5, const size_t d6,
                                            const size_t d7, const bool *input, bool *output, hipStream_t stream);

template CUDA_LIB_EXPORT void CalSlice4DGrad<double>(const size_t s1, const size_t s2, const size_t s3, const size_t s4,
                                                     const size_t l1, const size_t l2, const size_t l3, const size_t l4,
                                                     const size_t d1, const size_t d2, const size_t d3, const size_t d4,
                                                     const double *dy, double *dx, hipStream_t stream);
template CUDA_LIB_EXPORT void CalSlice4DGrad<float>(const size_t s1, const size_t s2, const size_t s3, const size_t s4,
                                                    const size_t l1, const size_t l2, const size_t l3, const size_t l4,
                                                    const size_t d1, const size_t d2, const size_t d3, const size_t d4,
                                                    const float *dy, float *dx, hipStream_t stream);
template CUDA_LIB_EXPORT void CalSlice4DGrad<half>(const size_t s1, const size_t s2, const size_t s3, const size_t s4,
                                                   const size_t l1, const size_t l2, const size_t l3, const size_t l4,
                                                   const size_t d1, const size_t d2, const size_t d3, const size_t d4,
                                                   const half *dy, half *dx, hipStream_t stream);
template CUDA_LIB_EXPORT void CalSlice4DGrad<int>(const size_t s1, const size_t s2, const size_t s3, const size_t s4,
                                                  const size_t l1, const size_t l2, const size_t l3, const size_t l4,
                                                  const size_t d1, const size_t d2, const size_t d3, const size_t d4,
                                                  const int *dy, int *dx, hipStream_t stream);
template CUDA_LIB_EXPORT void CalSlice4DGrad<short>(const size_t s1, const size_t s2, const size_t s3, const size_t s4,  // NOLINT
                                                    const size_t l1, const size_t l2, const size_t l3, const size_t l4,
                                                    const size_t d1, const size_t d2, const size_t d3, const size_t d4,
                                                    const short *dy, short *dx, hipStream_t stream);  // NOLINT
template CUDA_LIB_EXPORT void CalSlice4DGrad<unsigned char>(const size_t s1, const size_t s2, const size_t s3,
                                                            const size_t s4, const size_t l1, const size_t l2,
                                                            const size_t l3, const size_t l4, const size_t d1,
                                                            const size_t d2, const size_t d3, const size_t d4,
                                                            const unsigned char *dy, unsigned char *dx,
                                                            hipStream_t stream);
template CUDA_LIB_EXPORT void CalSlice4DGrad<int64_t>(const size_t s1, const size_t s2, const size_t s3,
                                                      const size_t s4, const size_t l1, const size_t l2,
                                                      const size_t l3, const size_t l4, const size_t d1,
                                                      const size_t d2, const size_t d3, const size_t d4,
                                                      const int64_t *dy, int64_t *dx, hipStream_t stream);
template CUDA_LIB_EXPORT void CalSlice4DGrad<bool>(const size_t s1, const size_t s2, const size_t s3, const size_t s4,
                                                   const size_t l1, const size_t l2, const size_t l3, const size_t l4,
                                                   const size_t d1, const size_t d2, const size_t d3, const size_t d4,
                                                   const bool *dy, bool *dx, hipStream_t stream);

template CUDA_LIB_EXPORT void FillDeviceArray<bool>(const size_t input_size, bool *addr, const float value,
                                                    hipStream_t hip_stream);
template CUDA_LIB_EXPORT void FillDeviceArray<int64_t>(const size_t input_size, int64_t *addr, const float value,
                                                       hipStream_t hip_stream);
template CUDA_LIB_EXPORT void FillDeviceArray<int>(const size_t input_size, int *addr, const float value,
                                                   hipStream_t hip_stream);
template CUDA_LIB_EXPORT void FillDeviceArray<short>(const size_t input_size, short *addr, const float value,  // NOLINT
                                                     hipStream_t hip_stream);
template CUDA_LIB_EXPORT void FillDeviceArray<int8_t>(const size_t input_size, int8_t *addr, const float value,
                                                      hipStream_t hip_stream);
template CUDA_LIB_EXPORT void FillDeviceArray<uint64_t>(const size_t input_size, uint64_t *addr, const float value,
                                                        hipStream_t hip_stream);
template CUDA_LIB_EXPORT void FillDeviceArray<uint32_t>(const size_t input_size, uint32_t *addr, const float value,
                                                        hipStream_t hip_stream);
template CUDA_LIB_EXPORT void FillDeviceArray<uint16_t>(const size_t input_size, uint16_t *addr, const float value,
                                                        hipStream_t hip_stream);
template CUDA_LIB_EXPORT void FillDeviceArray<unsigned char>(const size_t input_size, unsigned char *addr,
                                                             const float value, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void FillDeviceArray<half>(const size_t input_size, half *addr, const float value,
                                                    hipStream_t hip_stream);
template CUDA_LIB_EXPORT void FillDeviceArray<float>(const size_t input_size, float *addr, const float value,
                                                     hipStream_t hip_stream);
template CUDA_LIB_EXPORT void FillDeviceArray<double>(const size_t input_size, double *addr, const float value,
                                                      hipStream_t hip_stream);

template CUDA_LIB_EXPORT void StridedSlice(const std::vector<size_t> &input_shape, const std::vector<int64_t> &begin,
                                           const std::vector<int64_t> &strides, const std::vector<size_t> &output_shape,
                                           const bool *input, bool *output, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void StridedSlice(const std::vector<size_t> &input_shape, const std::vector<int64_t> &begin,
                                           const std::vector<int64_t> &strides, const std::vector<size_t> &output_shape,
                                           const double *input, double *output, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void StridedSlice(const std::vector<size_t> &input_shape, const std::vector<int64_t> &begin,
                                           const std::vector<int64_t> &strides, const std::vector<size_t> &output_shape,
                                           const float *input, float *output, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void StridedSlice(const std::vector<size_t> &input_shape, const std::vector<int64_t> &begin,
                                           const std::vector<int64_t> &strides, const std::vector<size_t> &output_shape,
                                           const half *input, half *output, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void StridedSlice(const std::vector<size_t> &input_shape, const std::vector<int64_t> &begin,
                                           const std::vector<int64_t> &strides, const std::vector<size_t> &output_shape,
                                           const int64_t *input, int64_t *output, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void StridedSlice(const std::vector<size_t> &input_shape, const std::vector<int64_t> &begin,
                                           const std::vector<int64_t> &strides, const std::vector<size_t> &output_shape,
                                           const int *input, int *output, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void StridedSlice(const std::vector<size_t> &input_shape, const std::vector<int64_t> &begin,
                                           const std::vector<int64_t> &strides, const std::vector<size_t> &output_shape,
                                           const short *input, short *output, hipStream_t hip_stream);  // NOLINT
template CUDA_LIB_EXPORT void StridedSlice(const std::vector<size_t> &input_shape, const std::vector<int64_t> &begin,
                                           const std::vector<int64_t> &strides, const std::vector<size_t> &output_shape,
                                           const int8_t *input, int8_t *output, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void StridedSlice(const std::vector<size_t> &input_shape, const std::vector<int64_t> &begin,
                                           const std::vector<int64_t> &strides, const std::vector<size_t> &output_shape,
                                           const uint64_t *input, uint64_t *output, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void StridedSlice(const std::vector<size_t> &input_shape, const std::vector<int64_t> &begin,
                                           const std::vector<int64_t> &strides, const std::vector<size_t> &output_shape,
                                           const uint32_t *input, uint32_t *output, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void StridedSlice(const std::vector<size_t> &input_shape, const std::vector<int64_t> &begin,
                                           const std::vector<int64_t> &strides, const std::vector<size_t> &output_shape,
                                           const uint16_t *input, uint16_t *output, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void StridedSlice(const std::vector<size_t> &input_shape, const std::vector<int64_t> &begin,
                                           const std::vector<int64_t> &strides, const std::vector<size_t> &output_shape,
                                           const unsigned char *input, unsigned char *output, hipStream_t hip_stream);

template CUDA_LIB_EXPORT void StridedSliceGrad(const std::vector<size_t> &dy_shape, const std::vector<int64_t> &begin,
                                               const std::vector<int64_t> &strides, const std::vector<size_t> &dx_shape,
                                               const bool *dy, bool *dx, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void StridedSliceGrad(const std::vector<size_t> &dy_shape, const std::vector<int64_t> &begin,
                                               const std::vector<int64_t> &strides, const std::vector<size_t> &dx_shape,
                                               const double *dy, double *dx, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void StridedSliceGrad(const std::vector<size_t> &dy_shape, const std::vector<int64_t> &begin,
                                               const std::vector<int64_t> &strides, const std::vector<size_t> &dx_shape,
                                               const float *dy, float *dx, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void StridedSliceGrad(const std::vector<size_t> &dy_shape, const std::vector<int64_t> &begin,
                                               const std::vector<int64_t> &strides, const std::vector<size_t> &dx_shape,
                                               const half *dy, half *dx, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void StridedSliceGrad(const std::vector<size_t> &dy_shape, const std::vector<int64_t> &begin,
                                               const std::vector<int64_t> &strides, const std::vector<size_t> &dx_shape,
                                               const int64_t *dy, int64_t *dx, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void StridedSliceGrad(const std::vector<size_t> &dy_shape, const std::vector<int64_t> &begin,
                                               const std::vector<int64_t> &strides, const std::vector<size_t> &dx_shape,
                                               const int *dy, int *dx, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void StridedSliceGrad(const std::vector<size_t> &dy_shape, const std::vector<int64_t> &begin,
                                               const std::vector<int64_t> &strides, const std::vector<size_t> &dx_shape,
                                               const short *dy, short *dx, hipStream_t hip_stream);  // NOLINT
template CUDA_LIB_EXPORT void StridedSliceGrad(const std::vector<size_t> &dy_shape, const std::vector<int64_t> &begin,
                                               const std::vector<int64_t> &strides, const std::vector<size_t> &dx_shape,
                                               const int8_t *dy, int8_t *dx, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void StridedSliceGrad(const std::vector<size_t> &dy_shape, const std::vector<int64_t> &begin,
                                               const std::vector<int64_t> &strides, const std::vector<size_t> &dx_shape,
                                               const uint64_t *dy, uint64_t *dx, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void StridedSliceGrad(const std::vector<size_t> &dy_shape, const std::vector<int64_t> &begin,
                                               const std::vector<int64_t> &strides, const std::vector<size_t> &dx_shape,
                                               const uint32_t *dy, uint32_t *dx, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void StridedSliceGrad(const std::vector<size_t> &dy_shape, const std::vector<int64_t> &begin,
                                               const std::vector<int64_t> &strides, const std::vector<size_t> &dx_shape,
                                               const uint16_t *dy, uint16_t *dx, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void StridedSliceGrad(const std::vector<size_t> &dy_shape, const std::vector<int64_t> &begin,
                                               const std::vector<int64_t> &strides, const std::vector<size_t> &dx_shape,
                                               const unsigned char *dy, unsigned char *dx, hipStream_t hip_stream);
