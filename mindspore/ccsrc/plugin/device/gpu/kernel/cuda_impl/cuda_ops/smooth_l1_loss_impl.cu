#include "hip/hip_runtime.h"
/**
 * Copyright 2020 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "smooth_l1_loss_impl.cuh"
#include "include/hip/hip_fp16.h"

template <typename T>
__global__ void SmoothL1LossKernel(const int input_size, const float beta, const T *prediction, const T *target,
                                   T *loss) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < input_size; i += blockDim.x * gridDim.x) {
    T value = fabsf(prediction[i] - target[i]);
    if (value < beta) {
      loss[i] = 0.5 * value * value / beta;
    } else {
      loss[i] = value - (0.5 * beta);
    }
  }
}

template <>
__global__ void SmoothL1LossKernel(const int input_size, const float beta, const half *prediction, const half *target,
                                   half *loss) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < input_size; i += blockDim.x * gridDim.x) {
    half value = fabsf(prediction[i] - target[i]);
    half h_beta = __float2half(beta);
    if (value < h_beta) {
      loss[i] = __float2half(0.5) * value * value / h_beta;
    } else {
      loss[i] = value - (__float2half(0.5) * h_beta);
    }
  }
}

template <typename T>
void SmoothL1Loss(const int &input_size, const float &beta, const T *prediction, const T *target, T *loss,
                  hipStream_t stream) {
  SmoothL1LossKernel<<<GET_BLOCKS(input_size), GET_THREADS, 0, stream>>>(input_size, beta, prediction, target, loss);
}

template <typename T>
__global__ void SmoothL1LossGradKernel(const int input_size, const float beta, const T *prediction, const T *target,
                                       const T *dloss, T *dx) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < input_size; i += blockDim.x * gridDim.x) {
    T value = prediction[i] - target[i];
    if (value > beta) {
      dx[i] = dloss[i];
    } else if (value < -beta) {
      dx[i] = -dloss[i];
    } else {
      dx[i] = (value / beta) * dloss[i];
    }
  }
}

template <>
__global__ void SmoothL1LossGradKernel(const int input_size, const float beta, const half *prediction,
                                       const half *target, const half *dloss, half *dx) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < input_size; i += blockDim.x * gridDim.x) {
    half value = prediction[i] - target[i];
    half h_beta = __float2half(beta);
    if (value > h_beta) {
      dx[i] = dloss[i];
    } else if (value < -h_beta) {
      dx[i] = -dloss[i];
    } else {
      dx[i] = (value / h_beta) * dloss[i];
    }
  }
}

template <typename T>
void SmoothL1LossGrad(const int &input_size, const float &beta, const T *prediction, const T *target, const T *dloss,
                      T *dx, hipStream_t stream) {
  SmoothL1LossGradKernel<<<GET_BLOCKS(input_size), GET_THREADS, 0, stream>>>(input_size, beta, prediction, target,
                                                                             dloss, dx);
}
template CUDA_LIB_EXPORT void SmoothL1Loss<double>(const int &input_size, const float &beta, const double *prediction,
                                                   const double *target, double *loss, hipStream_t stream);
template CUDA_LIB_EXPORT void SmoothL1LossGrad<double>(const int &input_size, const float &beta,
                                                       const double *prediction, const double *target,
                                                       const double *dloss, double *dx, hipStream_t stream);

template CUDA_LIB_EXPORT void SmoothL1Loss<float>(const int &input_size, const float &beta, const float *prediction,
                                                  const float *target, float *loss, hipStream_t stream);
template CUDA_LIB_EXPORT void SmoothL1LossGrad<float>(const int &input_size, const float &beta, const float *prediction,
                                                      const float *target, const float *dloss, float *dx,
                                                      hipStream_t stream);

template CUDA_LIB_EXPORT void SmoothL1Loss<half>(const int &input_size, const float &beta, const half *prediction,
                                                 const half *target, half *loss, hipStream_t stream);
template CUDA_LIB_EXPORT void SmoothL1LossGrad<half>(const int &input_size, const float &beta, const half *prediction,
                                                     const half *target, const half *dloss, half *dx,
                                                     hipStream_t stream);
