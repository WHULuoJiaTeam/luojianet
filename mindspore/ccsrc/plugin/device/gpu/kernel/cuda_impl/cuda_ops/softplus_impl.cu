#include "hip/hip_runtime.h"
/**
 * Copyright 2020-2021 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/softplus_impl.cuh"
#include "include/hip/hip_fp16.h"

template <typename T>
__global__ void SoftplusKernel(const size_t size, const T *input_addr, T *output_addr) {
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < size; pos += blockDim.x * gridDim.x) {
    float x = input_addr[pos];
    output_addr[pos] = logf(1. + exp(x));
  }
}

template <>
__global__ void SoftplusKernel(const size_t size, const half *input_addr, half *output_addr) {
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < size; pos += blockDim.x * gridDim.x) {
    float x = __half2float(input_addr[pos]);
    output_addr[pos] = __float2half(logf(1. + exp(x)));
  }
}

template <typename T>
void Softplus(const size_t size, const T *input_addr, T *output_addr, hipStream_t hip_stream) {
  SoftplusKernel<<<GET_BLOCKS(size), GET_THREADS, 0, hip_stream>>>(size, input_addr, output_addr);
}

template <>
void Softplus(const size_t size, const half *input_addr, half *output_addr, hipStream_t hip_stream) {
  SoftplusKernel<half><<<GET_BLOCKS(size), GET_THREADS, 0, hip_stream>>>(size, input_addr, output_addr);
}

template <typename T>
__global__ void SoftplusGradKernel(const size_t size, const T *dy_addr, const T *x_addr, T *dx_addr) {
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < (size); pos += blockDim.x * gridDim.x) {
    T exp_x = exp(x_addr[pos]);
    dx_addr[pos] = dy_addr[pos] * exp_x / (1. + exp_x);
  }
}

template <typename T>
__global__ void SoftplusGradKernel(const size_t size, const half *dy_addr, const half *x_addr, half *dx_addr) {
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < (size); pos += blockDim.x * gridDim.x) {
    float x = __half2float(x_addr[pos]);
    float dy = __half2float(dy_addr[pos]);
    float exp_x = exp(x);
    dx_addr[pos] = __float2half(dy * exp_x / (1. + exp_x));
  }
}

template <typename T>
void SoftplusGrad(const size_t size, const T *dy_addr, const T *x_addr, T *dx_addr, hipStream_t hip_stream) {
  SoftplusGradKernel<<<GET_BLOCKS(size), GET_THREADS, 0, hip_stream>>>(size, dy_addr, x_addr, dx_addr);
}

template <>
void SoftplusGrad(const size_t size, const half *dy_addr, const half *x_addr, half *dx_addr, hipStream_t hip_stream) {
  SoftplusGradKernel<half><<<GET_BLOCKS(size), GET_THREADS, 0, hip_stream>>>(size, dy_addr, x_addr, dx_addr);
}

template CUDA_LIB_EXPORT void Softplus(const size_t size, const float *input_addr, float *output_addr,
                                       hipStream_t hip_stream);
template CUDA_LIB_EXPORT void Softplus(const size_t size, const half *input_addr, half *output_addr,
                                       hipStream_t hip_stream);
template CUDA_LIB_EXPORT void SoftplusGrad(const size_t size, const float *dy_addr, const float *x_addr, float *dx_addr,
                                           hipStream_t hip_stream);
template CUDA_LIB_EXPORT void SoftplusGrad(const size_t size, const half *dy_addr, const half *x_addr, half *dx_addr,
                                           hipStream_t hip_stream);
