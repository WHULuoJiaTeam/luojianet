#include "hip/hip_runtime.h"
/**
 * Copyright 2020 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <stdio.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/split_impl.cuh"
#include "include/hip/hip_fp16.h"
template <typename T>
__global__ void Split(const size_t size, const int axis_step, const int all_size_before_axis,
                      const int all_size_axis, const T* input, T** outputs) {
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < size; pos += blockDim.x * gridDim.x) {
    int num = pos % all_size_before_axis / all_size_axis;
    int block = num / axis_step;
    int block_pos = pos / all_size_before_axis * axis_step * all_size_axis +
                    num % axis_step * all_size_axis + pos % all_size_axis;
    outputs[block][block_pos] = input[pos];
  }
  return;
}

template <typename T>
void SplitKernel(const size_t size, const int axis_step, const int all_size_before_axis,
                 const int all_size_axis, const T* input, T** outputs, hipStream_t hip_stream) {
  Split<<<GET_BLOCKS(size), GET_THREADS, 0, hip_stream>>>(size, axis_step, all_size_before_axis,
                                                           all_size_axis, input, outputs);
  return;
}

template CUDA_LIB_EXPORT void SplitKernel(const size_t size, const int axis_step, const int all_size_before_axis,
                                          const int all_size_axis, const half* input, half** outputs,
                                          hipStream_t hip_stream);
template CUDA_LIB_EXPORT void SplitKernel(const size_t size, const int axis_step, const int all_size_before_axis,
                                          const int all_size_axis, const float* input, float** outputs,
                                          hipStream_t hip_stream);
template CUDA_LIB_EXPORT void SplitKernel(const size_t size, const int axis_step, const int all_size_before_axis,
                                          const int all_size_axis, const double* input, double** outputs,
                                          hipStream_t hip_stream);
template CUDA_LIB_EXPORT void SplitKernel(const size_t size, const int axis_step, const int all_size_before_axis,
                                          const int all_size_axis, const int* input, int** outputs,
                                          hipStream_t hip_stream);
template CUDA_LIB_EXPORT void SplitKernel(const size_t size, const int axis_step, const int all_size_before_axis,
                                          const int all_size_axis, const uint32_t* input, uint32_t** outputs,
                                          hipStream_t hip_stream);
template CUDA_LIB_EXPORT void SplitKernel(const size_t size, const int axis_step, const int all_size_before_axis,
                                          const int all_size_axis, const int64_t* input, int64_t** outputs,
                                          hipStream_t hip_stream);
template CUDA_LIB_EXPORT void SplitKernel(const size_t size, const int axis_step, const int all_size_before_axis,
                                          const int all_size_axis, const bool* input, bool** outputs,
                                          hipStream_t hip_stream);
