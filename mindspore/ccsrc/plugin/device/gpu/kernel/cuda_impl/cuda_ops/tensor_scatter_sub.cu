#include "hip/hip_runtime.h"
/**
 * Copyright 2021 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/tensor_scatter_sub.cuh"
#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/util.cuh"

template <typename T, typename S>
__global__ void TensorScatterSubKernel(T *input, S *indices, T *update, T *output, const size_t block_size,
                                       const size_t input_size, const size_t output_size, const size_t indices_dim_0,
                                       const size_t indices_dim_1, S *indices_stride, S *work_shape) {
  int i, j;
  for (size_t read_index = blockIdx.x * blockDim.x + threadIdx.x; read_index < input_size;
       read_index += blockDim.x * gridDim.x) {
    size_t write_index = 0;
    bool out_bound = false;

    i = read_index / block_size;
    j = read_index % block_size;

    for (size_t k = 0; k < indices_dim_1; k++) {
      S indices_i = indices[i * indices_dim_1 + k];
      out_bound |= indices_i >= work_shape[k];
      write_index += indices_i * indices_stride[k];
    }

    write_index += j;
    out_bound |= write_index >= output_size;

    if (!out_bound) {
      MsAtomicSub(&output[write_index], update[read_index]);
    }
  }
}

template <typename T, typename S>
void TensorScatterSub(T *input, S *indices, T *update, T *output, const size_t &block_size, const size_t &input_size,
                      const size_t &output_size, const size_t &indices_dim_0, const size_t &indices_dim_1,
                      S *indices_stride, S *work_shape, hipStream_t stream) {
  TensorScatterSubKernel<<<GET_BLOCKS(output_size), GET_THREADS, 0, stream>>>(
    input, indices, update, output, block_size, input_size, output_size, indices_dim_0, indices_dim_1, indices_stride,
    work_shape);
  return;
}

// for int32 index
template CUDA_LIB_EXPORT void TensorScatterSub<half, int>(half *input, int *indices, half *update, half *output,
                                                          const size_t &block_size, const size_t &input_size,
                                                          const size_t &output_size, const size_t &indices_dim_0,
                                                          const size_t &indices_dim_1, int *indices_stride,
                                                          int *work_shape, hipStream_t stream);

template CUDA_LIB_EXPORT void TensorScatterSub<float, int>(float *input, int *indices, float *update, float *output,
                                                           const size_t &block_size, const size_t &input_size,
                                                           const size_t &output_size, const size_t &indices_dim_0,
                                                           const size_t &indices_dim_1, int *indices_stride,
                                                           int *work_shape, hipStream_t stream);

template CUDA_LIB_EXPORT void TensorScatterSub<char, int>(char *input, int *indices, char *update, char *output,
                                                          const size_t &block_size, const size_t &input_size,
                                                          const size_t &output_size, const size_t &indices_dim_0,
                                                          const size_t &indices_dim_1, int *indices_stride,
                                                          int *work_shape, hipStream_t stream);

template CUDA_LIB_EXPORT void TensorScatterSub<unsigned char, int>(unsigned char *input, int *indices,
                                                                   unsigned char *update, unsigned char *output,
                                                                   const size_t &block_size, const size_t &input_size,
                                                                   const size_t &output_size,
                                                                   const size_t &indices_dim_0,
                                                                   const size_t &indices_dim_1, int *indices_stride,
                                                                   int *work_shape, hipStream_t stream);

template CUDA_LIB_EXPORT void TensorScatterSub<int, int>(int *input, int *indices, int *update, int *output,
                                                         const size_t &block_size, const size_t &input_size,
                                                         const size_t &output_size, const size_t &indices_dim_0,
                                                         const size_t &indices_dim_1, int *indices_stride,
                                                         int *work_shape, hipStream_t stream);

// for int64 index
template CUDA_LIB_EXPORT void TensorScatterSub<half, int64_t>(half *input, int64_t *indices, half *update, half *output,
                                                              const size_t &block_size, const size_t &input_size,
                                                              const size_t &output_size, const size_t &indices_dim_0,
                                                              const size_t &indices_dim_1, int64_t *indices_stride,
                                                              int64_t *work_shape, hipStream_t stream);

template CUDA_LIB_EXPORT void TensorScatterSub<float, int64_t>(float *input, int64_t *indices, float *update,
                                                               float *output, const size_t &block_size,
                                                               const size_t &input_size, const size_t &output_size,
                                                               const size_t &indices_dim_0, const size_t &indices_dim_1,
                                                               int64_t *indices_stride, int64_t *work_shape,
                                                               hipStream_t stream);

template CUDA_LIB_EXPORT void TensorScatterSub<char, int64_t>(char *input, int64_t *indices, char *update, char *output,
                                                              const size_t &block_size, const size_t &input_size,
                                                              const size_t &output_size, const size_t &indices_dim_0,
                                                              const size_t &indices_dim_1, int64_t *indices_stride,
                                                              int64_t *work_shape, hipStream_t stream);

template CUDA_LIB_EXPORT void TensorScatterSub<unsigned char, int64_t>(unsigned char *input, int64_t *indices,
                                                                       unsigned char *update, unsigned char *output,
                                                                       const size_t &block_size,
                                                                       const size_t &input_size,
                                                                       const size_t &output_size,
                                                                       const size_t &indices_dim_0,
                                                                       const size_t &indices_dim_1,
                                                                       int64_t *indices_stride, int64_t *work_shape,
                                                                       hipStream_t stream);

template CUDA_LIB_EXPORT void TensorScatterSub<int, int64_t>(int *input, int64_t *indices, int *update, int *output,
                                                             const size_t &block_size, const size_t &input_size,
                                                             const size_t &output_size, const size_t &indices_dim_0,
                                                             const size_t &indices_dim_1, int64_t *indices_stride,
                                                             int64_t *work_shape, hipStream_t stream);
