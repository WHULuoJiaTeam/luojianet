#include "hip/hip_runtime.h"
/**
 * Copyright 2021 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/tile_impl.cuh"
#include "include/hip/hip_fp16.h"

template <typename T>
__global__ void Tile(const size_t output_size, const size_t input_size, const size_t shape_size,
                     const size_t *input_shape, const size_t *output_shape, const T *input, T *output) {
  // for example 4-D: pos = pos_array[0] * output_shape[1] * output_shape[2] * output_shape[3] +
  //                        pos_array[1] * output_shape[2] * output_shape[3] +
  //                        pos_array[2] * output_shape[3] +
  //                        pos_array[3]
  size_t pos_array[TILE_MAX_DIMENSION];
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < output_size; pos += blockDim.x * gridDim.x) {
    size_t tmp_pos = pos;
    size_t pos_size = output_size / output_shape[0];
    pos_array[0] = tmp_pos / pos_size;
    for (size_t i = 1; i < shape_size; i++) {
      tmp_pos -= pos_array[i - 1] * pos_size;
      pos_size = pos_size / output_shape[i];
      pos_array[i] = tmp_pos / pos_size;
    }
    for (size_t i = 0; i < shape_size; i++) {
      pos_array[i] = pos_array[i] % input_shape[i];
    }
    pos_size = input_size;
    size_t input_pos = 0;
    for (size_t i = 0; i < shape_size; i++) {
      pos_size /= input_shape[i];
      input_pos += (pos_array[i] * pos_size);
    }
    output[pos] = input[input_pos];
  }
}

template <typename T>
void CalTile(const size_t output_size, const size_t input_size, const size_t shape_size, const size_t *input_shape,
             const size_t *output_shape, const T *input, T *output, hipStream_t hip_stream) {
  Tile<<<GET_BLOCKS(output_size), GET_THREADS, 0, hip_stream>>>(output_size, input_size, shape_size, input_shape,
                                                                 output_shape, input, output);
  return;
}

template CUDA_LIB_EXPORT void CalTile<double>(const size_t output_size, const size_t input_size,
                                              const size_t shape_size, const size_t *input_shape,
                                              const size_t *output_shape, const double *input, double *output,
                                              hipStream_t hip_stream);
template CUDA_LIB_EXPORT void CalTile<float>(const size_t output_size, const size_t input_size, const size_t shape_size,
                                             const size_t *input_shape, const size_t *output_shape, const float *input,
                                             float *output, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void CalTile<half>(const size_t output_size, const size_t input_size, const size_t shape_size,
                                            const size_t *input_shape, const size_t *output_shape, const half *input,
                                            half *output, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void CalTile<int16_t>(const size_t output_size, const size_t input_size,
                                               const size_t shape_size, const size_t *input_shape,
                                               const size_t *output_shape, const int16_t *input, int16_t *output,
                                               hipStream_t hip_stream);
template CUDA_LIB_EXPORT void CalTile<int>(const size_t output_size, const size_t input_size, const size_t shape_size,
                                           const size_t *input_shape, const size_t *output_shape, const int *input,
                                           int *output, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void CalTile<int64_t>(const size_t output_size, const size_t input_size,
                                               const size_t shape_size, const size_t *input_shape,
                                               const size_t *output_shape, const int64_t *input, int64_t *output,
                                               hipStream_t hip_stream);
template CUDA_LIB_EXPORT void CalTile<bool>(const size_t output_size, const size_t input_size, const size_t shape_size,
                                            const size_t *input_shape, const size_t *output_shape, const bool *input,
                                            bool *output, hipStream_t hip_stream);
