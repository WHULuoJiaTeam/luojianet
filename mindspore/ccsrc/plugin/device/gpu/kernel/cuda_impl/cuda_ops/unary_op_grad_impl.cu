#include "hip/hip_runtime.h"
/**
 * Copyright 2020-2021 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "unary_op_grad_impl.cuh"
#include "include/hip/hip_fp16.h"

template <typename T>
__global__ void SqrtGradKernel(const T *input, const T *dout, T *output, const size_t count) {
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < (count); i += blockDim.x * gridDim.x) {
    float input_f = static_cast<float>(input[i]);
    float dout_f = static_cast<float>(dout[i]);
    float res_vmul = dout_f / (2.0 * input_f);
    output[i] = static_cast<T>(res_vmul);
  }
  return;
}

template <typename T>
__global__ void RsqrtGradKernel(const T *input, const T *dout, T *output, const size_t count) {
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < (count); i += blockDim.x * gridDim.x) {
    float input_f = static_cast<float>(input[i]);
    float dout_f = static_cast<float>(dout[i]);
    float res_vmul = input_f * input_f * input_f;
    res_vmul = -0.5 * res_vmul * dout_f;
    output[i] = static_cast<T>(res_vmul);
  }
  return;
}

template <typename T>
__global__ void AsinGradKernel(const T *input, const T *dout, T *output, const size_t count) {
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < (count); i += blockDim.x * gridDim.x) {
    T one = 1;
    T sqt = sqrtf(one - input[i] * input[i]);
    output[i] = dout[i] / sqt;
  }
  return;
}

template <>
__global__ void AsinGradKernel(const half *input, const half *dout, half *output, const size_t count) {
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < (count); i += blockDim.x * gridDim.x) {
    half one = 1;
    half sqt = hsqrt(one - input[i] * input[i]);
    output[i] = dout[i] / sqt;
  }
  return;
}

template <typename T>
__global__ void ACosGradKernel(const T *input, const T *dout, T *output, const size_t count) {
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < (count); i += blockDim.x * gridDim.x) {
    T neg_one = -1;
    T one = 1;
    T sqt = sqrtf(one - input[i] * input[i]);
    output[i] = neg_one * dout[i] / sqt;
  }
  return;
}

template <>
__global__ void ACosGradKernel(const half *input, const half *dout, half *output, const size_t count) {
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < (count); i += blockDim.x * gridDim.x) {
    half neg_one = -1;
    half one = 1;
    half sqt = hsqrt(one - input[i] * input[i]);
    output[i] = neg_one * dout[i] / sqt;
  }
  return;
}

template <typename T>
__global__ void AtanGradKernel(const T *input, const T *dout, T *output, const size_t count) {
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < (count); i += blockDim.x * gridDim.x) {
    T one = 1;
    T divisor = one + input[i] * input[i];
    output[i] = dout[i] / divisor;
  }
  return;
}

template <typename T>
__global__ void AsinhGradKernel(const T *input, const T *dout, T *output, const size_t count) {
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < (count); i += blockDim.x * gridDim.x) {
    float inputf = static_cast<float>(input[i]);
    T coshy = static_cast<T>(coshf(inputf));
    output[i] = dout[i] / coshy;
  }
  return;
}

template <typename T>
__global__ void AcoshGradKernel(const T *input, const T *dout, T *output, const size_t count) {
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < (count); i += blockDim.x * gridDim.x) {
    float inputf = static_cast<float>(input[i]);
    T sinhy = static_cast<T>(sinhf(inputf));
    output[i] = dout[i] / sinhy;
  }
  return;
}

template <typename T>
__global__ void ReciprocalGradKernel(const T *input, const T *dout, T *output, const size_t count) {
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < count; i += blockDim.x * gridDim.x) {
    float inputf = static_cast<float>(input[i]);
    float doutf = static_cast<float>(dout[i]);
    float res = -1 * doutf * inputf * inputf;
    output[i] = static_cast<T>(res);
  }
  return;
}

template <typename T>
void SqrtGrad(const T *input, const T *dout, T *output, const size_t count, hipStream_t hip_stream) {
  SqrtGradKernel<<<GET_BLOCKS(count), GET_THREADS, 0, hip_stream>>>(input, dout, output, count);
  return;
}

template <typename T>
void RsqrtGrad(const T *input, const T *dout, T *output, const size_t count, hipStream_t hip_stream) {
  RsqrtGradKernel<<<GET_BLOCKS(count), GET_THREADS, 0, hip_stream>>>(input, dout, output, count);
  return;
}

template <typename T>
void AsinGrad(const T *input, const T *dout, T *output, const size_t count, hipStream_t hip_stream) {
  AsinGradKernel<<<GET_BLOCKS(count), GET_THREADS, 0, hip_stream>>>(input, dout, output, count);
  return;
}

template <typename T>
void ACosGrad(const T *input, const T *dout, T *output, const size_t count, hipStream_t hip_stream) {
  ACosGradKernel<<<GET_BLOCKS(count), GET_THREADS, 0, hip_stream>>>(input, dout, output, count);
  return;
}

template <typename T>
void AtanGrad(const T *input, const T *dout, T *output, const size_t count, hipStream_t hip_stream) {
  AtanGradKernel<<<GET_BLOCKS(count), GET_THREADS, 0, hip_stream>>>(input, dout, output, count);
  return;
}

template <typename T>
void AsinhGrad(const T *input, const T *dout, T *output, const size_t count, hipStream_t hip_stream) {
  AsinhGradKernel<<<GET_BLOCKS(count), GET_THREADS, 0, hip_stream>>>(input, dout, output, count);
  return;
}

template <typename T>
void AcoshGrad(const T *input, const T *dout, T *output, const size_t count, hipStream_t hip_stream) {
  AcoshGradKernel<<<GET_BLOCKS(count), GET_THREADS, 0, hip_stream>>>(input, dout, output, count);
  return;
}

template <typename T>
void ReciprocalGrad(const T *input, const T *dout, T *output, const size_t count, hipStream_t hip_stream) {
  ReciprocalGradKernel<<<GET_BLOCKS(count), GET_THREADS, 0, hip_stream>>>(input, dout, output, count);
  return;
}

template CUDA_LIB_EXPORT void SqrtGrad<double>(const double *input, const double *dout, double *output,
                                               const size_t count, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void RsqrtGrad<double>(const double *input, const double *dout, double *output,
                                                const size_t count, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void AsinGrad<double>(const double *input, const double *dout, double *output,
                                               const size_t count, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void ACosGrad<double>(const double *input, const double *dout, double *output,
                                               const size_t count, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void AtanGrad<double>(const double *input, const double *dout, double *output,
                                               const size_t count, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void AsinhGrad<double>(const double *input, const double *dout, double *output,
                                                const size_t count, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void AcoshGrad<double>(const double *input, const double *dout, double *output,
                                                const size_t count, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void ReciprocalGrad<double>(const double *input, const double *dout, double *output,
                                                     const size_t count, hipStream_t hip_stream);

template CUDA_LIB_EXPORT void SqrtGrad<float>(const float *input, const float *dout, float *output,
                                              const size_t count, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void RsqrtGrad<float>(const float *input, const float *dout, float *output,
                                               const size_t count, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void AsinGrad<float>(const float *input, const float *dout, float *output,
                                              const size_t count, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void ACosGrad<float>(const float *input, const float *dout, float *output,
                                              const size_t count, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void AtanGrad<float>(const float *input, const float *dout, float *output,
                                              const size_t count, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void AsinhGrad<float>(const float *input, const float *dout, float *output,
                                               const size_t count, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void AcoshGrad<float>(const float *input, const float *dout, float *output,
                                               const size_t count, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void ReciprocalGrad<float>(const float *input, const float *dout, float *output,
                                                    const size_t count, hipStream_t hip_stream);
template CUDA_LIB_EXPORT void SqrtGrad<half>(const half *input, const half *dout, half *output, const size_t count,
                                             hipStream_t hip_stream);
template CUDA_LIB_EXPORT void RsqrtGrad<half>(const half *input, const half *dout, half *output, const size_t count,
                                              hipStream_t hip_stream);
template CUDA_LIB_EXPORT void AsinGrad<half>(const half *input, const half *dout, half *output, const size_t count,
                                             hipStream_t hip_stream);
template CUDA_LIB_EXPORT void ACosGrad<half>(const half *input, const half *dout, half *output, const size_t count,
                                             hipStream_t hip_stream);
template CUDA_LIB_EXPORT void AtanGrad<half>(const half *input, const half *dout, half *output, const size_t count,
                                             hipStream_t hip_stream);
template CUDA_LIB_EXPORT void AsinhGrad<half>(const half *input, const half *dout, half *output, const size_t count,
                                              hipStream_t hip_stream);
template CUDA_LIB_EXPORT void AcoshGrad<half>(const half *input, const half *dout, half *output, const size_t count,
                                              hipStream_t hip_stream);
template CUDA_LIB_EXPORT void ReciprocalGrad<half>(const half *input, const half *dout, half *output,
                                                   const size_t count, hipStream_t hip_stream);
