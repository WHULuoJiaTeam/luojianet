#include "hip/hip_runtime.h"
/**
 * Copyright 2020 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/unsorted_segment_max.cuh"
#include <limits>
#include "include/hip/hip_fp16.h"

template <typename T, typename S>
__global__ void UnsortedSegmentMax(const T *input, const S *segment_ids, const int64_t num_segments, size_t outer_size,
                                   size_t inner_size, bool fp16_flag, T init_K, T *output) {
  if (fp16_flag) {
    init_K = __int2half_rd(-65504);  // min value representable by float16
  }

  for (size_t t_idx = blockIdx.x * blockDim.x + threadIdx.x; t_idx < KWARPSIZE * num_segments * inner_size;
       t_idx += blockDim.x * gridDim.x) {
    size_t segment_id = t_idx / KWARPSIZE / inner_size;
    size_t inner_id = t_idx / KWARPSIZE % inner_size;
    size_t lane_id = threadIdx.x % KWARPSIZE;
    T threadK = init_K;

    for (size_t i = lane_id; i < outer_size; i += KWARPSIZE) {
      if (segment_ids[i] != segment_id) continue;
      T other_K = input[i * inner_size + inner_id];
      if (threadK < other_K) {
        threadK = other_K;
      }
    }
    __syncwarp();

    for (size_t offset = KWARPSIZE / 2; offset > 0; offset /= 2) {
      T other_K = __shfl_down_sync(0xffffffff, threadK, offset);
      if (threadK < other_K) {
        threadK = other_K;
      }
    }

    __syncwarp();

    if (lane_id == 0) {
      output[segment_id * inner_size + inner_id] = threadK;
    }
    __syncthreads();
  }
}

template <typename T, typename S>
void CalUnsortedSegmentMax(const T *input, const S *segment_ids, const int64_t num_segments, size_t outer_size,
                           size_t inner_size, T *output, hipStream_t stream) {
  size_t size = (inner_size * KWARPSIZE * num_segments);
  bool fp16_flag = false;
  // handle fp16 min value
  if (std::is_same<T, half>::value) {
    fp16_flag = true;
  }
  T init_K = std::numeric_limits<T>::lowest();
  UnsortedSegmentMax<<<GET_BLOCKS(size), GET_THREADS, 0, stream>>>(input, segment_ids, num_segments, outer_size,
                                                                   inner_size, fp16_flag, init_K, output);
  return;
}

template CUDA_LIB_EXPORT void CalUnsortedSegmentMax<float, int>(const float *input, const int *segment_ids,
                                                                const int64_t num_segments, size_t outer_size,
                                                                size_t inner_size, float *output, hipStream_t stream);
template CUDA_LIB_EXPORT void CalUnsortedSegmentMax<float, int64_t>(const float *input, const int64_t *segment_ids,
                                                                    const int64_t num_segments, size_t outer_size,
                                                                    size_t inner_size, float *output,
                                                                    hipStream_t stream);
template CUDA_LIB_EXPORT void CalUnsortedSegmentMax<half, int>(const half *input, const int *segment_ids,
                                                               const int64_t num_segments, size_t outer_size,
                                                               size_t inner_size, half *output, hipStream_t stream);
template CUDA_LIB_EXPORT void CalUnsortedSegmentMax<half, int64_t>(const half *input, const int64_t *segment_ids,
                                                                   const int64_t num_segments, size_t outer_size,
                                                                   size_t inner_size, half *output,
                                                                   hipStream_t stream);
template CUDA_LIB_EXPORT void CalUnsortedSegmentMax<int, int>(const int *input, const int *segment_ids,
                                                              const int64_t num_segments, size_t outer_size,
                                                              size_t inner_size, int *output, hipStream_t stream);
template CUDA_LIB_EXPORT void CalUnsortedSegmentMax<int, int64_t>(const int *input, const int64_t *segment_ids,
                                                                  const int64_t num_segments, size_t outer_size,
                                                                  size_t inner_size, int *output, hipStream_t stream);
