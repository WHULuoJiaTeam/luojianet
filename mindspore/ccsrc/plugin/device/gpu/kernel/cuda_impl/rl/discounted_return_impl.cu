#include "hip/hip_runtime.h"
/**
 * Copyright 2021 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "plugin/device/gpu/kernel/cuda_impl/rl/discounted_return_impl.cuh"
#include <hip/hip_fp16.h>
#include <algorithm>

template <typename T>
__global__ void DiscountedReturnKernel(const int timestep, const int num_env, const int num_element, const float gamma,
                                       const T *reward, const bool *done, const T *last_value, T *discouted_return) {
  int elements_per_timestep = num_env * num_element;
  int idx_in_timestep = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx_in_timestep >= num_env * num_element) {
    return;
  }

  T last_value_local = last_value[idx_in_timestep];
  int env_index = idx_in_timestep / num_element;
  for (int t = timestep - 1; t >= 0; t--) {
    int curr_timestep_index_offset = t * elements_per_timestep + idx_in_timestep;

    last_value_local = reward[curr_timestep_index_offset] +
                       static_cast<T>((1 - done[t * num_env + env_index]) * gamma) * last_value_local;
    discouted_return[curr_timestep_index_offset] = last_value_local;
  }
}

template <typename T>
void DiscountedReturn(const int &timestep, const int &num_env, const int &num_element, const float &gamma,
                      const T *reward, const bool *done, const T *last_value, T *discouted_return,
                      hipStream_t stream) {
  // Every block process M element, 256 is a common tile size.
  const int element_per_step = num_env * num_element;
  const int element_per_block = std::min(256, element_per_step);
  const int grid_dim = (element_per_step + element_per_block - 1) / element_per_block;

  DiscountedReturnKernel<<<grid_dim, element_per_block, 0, stream>>>(timestep, num_env, num_element, gamma, reward,
                                                                     done, last_value, discouted_return);
}

template void DiscountedReturn(const int &timestep, const int &num_env, const int &num_element, const float &gamma,
                               const float *reward, const bool *done, const float *last_value, float *discouted_return,
                               hipStream_t stream);
template void DiscountedReturn(const int &timestep, const int &num_env, const int &num_element, const float &gamma,
                               const half *reward, const bool *done, const half *last_value, half *discouted_return,
                               hipStream_t stream);
