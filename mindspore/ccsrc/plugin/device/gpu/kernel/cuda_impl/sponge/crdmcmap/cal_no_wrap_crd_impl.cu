#include "hip/hip_runtime.h"

/**
 * Copyright 2021 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "plugin/device/gpu/kernel/cuda_impl/sponge/crdmcmap/cal_no_wrap_crd_impl.cuh"
#include "plugin/device/gpu/kernel/cuda_impl/sponge/common_sponge.cuh"
#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/util.cuh"

__global__ void Calculate_No_Wrap_Crd(int atom_numbers, INT_VECTOR *box_map_times, VECTOR *box, VECTOR *crd,
                                      VECTOR *nowrap_crd) {
  for (int i = threadIdx.x; i < atom_numbers; i = i + blockDim.x) {
    nowrap_crd[i].x = static_cast<float>(box_map_times[i].int_x) * box[0].x + crd[i].x;
    nowrap_crd[i].y = static_cast<float>(box_map_times[i].int_y) * box[0].y + crd[i].y;
    nowrap_crd[i].z = static_cast<float>(box_map_times[i].int_z) * box[0].z + crd[i].z;
  }
}

void calculatenowrapcrd(int atom_numbers, int *box_map_times_f, float *box_f, float *crd_f, float *nowrap_crd_f,
                        hipStream_t stream) {
  Reset_List<<<ceilf(static_cast<float>(3. * atom_numbers) / 128), 128, 0, stream>>>(3 * atom_numbers, nowrap_crd_f,
                                                                                     0.);
  INT_VECTOR *box_map_times = reinterpret_cast<INT_VECTOR *>(box_map_times_f);
  VECTOR *box = reinterpret_cast<VECTOR *>(box_f);
  VECTOR *crd = reinterpret_cast<VECTOR *>(crd_f);
  VECTOR *nowrap_crd = reinterpret_cast<VECTOR *>(nowrap_crd_f);

  Calculate_No_Wrap_Crd<<<20, 256, 0, stream>>>(atom_numbers, box_map_times, box, crd,
                                                                         nowrap_crd);
  return;
}

void calculatenowrapcrd(int atom_numbers, int *box_map_times_f, float *box_f, float *crd_f, float *nowrap_crd_f,
                        hipStream_t stream);
