#include "hip/hip_runtime.h"
/**
 * Copyright 2021 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "plugin/device/gpu/kernel/cuda_impl/sponge/dihedral/dihedral_energy_impl.cuh"
#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/util.cuh"
#include "plugin/device/gpu/kernel/cuda_impl/sponge/common_sponge.cuh"

__global__ void DihedralEnergyKernel(int dihedral_numbers, const UNSIGNED_INT_VECTOR *uint_crd, const VECTOR *scaler,
                                     const int *atom_a, const int *atom_b, const int *atom_c, const int *atom_d,
                                     const int *ipn, const float *pk, const float *gamc, const float *gams,
                                     const float *pn, float *ene) {
  int dihedral_i = blockDim.x * blockIdx.x + threadIdx.x;
  if (dihedral_i < dihedral_numbers) {
    int atom_i = atom_a[dihedral_i];
    int atom_j = atom_b[dihedral_i];
    int atom_k = atom_c[dihedral_i];
    int atom_l = atom_d[dihedral_i];

    float temp_pk = pk[dihedral_i];
    float temp_pn = pn[dihedral_i];
    float temp_gamc = gamc[dihedral_i];
    float temp_gams = gams[dihedral_i];

    VECTOR drij = Get_Periodic_Displacement(uint_crd[atom_i], uint_crd[atom_j], scaler[0]);
    VECTOR drkj = Get_Periodic_Displacement(uint_crd[atom_k], uint_crd[atom_j], scaler[0]);
    VECTOR drkl = Get_Periodic_Displacement(uint_crd[atom_k], uint_crd[atom_l], scaler[0]);

    VECTOR r1 = drij ^ drkj;
    VECTOR r2 = drkl ^ drkj;

    float r1_1 = rnorm3df(r1.x, r1.y, r1.z);
    float r2_1 = rnorm3df(r2.x, r2.y, r2.z);
    float r1_1_r2_1 = r1_1 * r2_1;

    float phi = r1 * r2 * r1_1_r2_1;
    phi = fmaxf(-0.999999, fminf(phi, 0.999999));
    phi = acosf(phi);

    float sign = (r2 ^ r1) * drkj;
    copysignf(phi, sign);

    phi = CONSTANT_Pi - phi;

    float nphi = temp_pn * phi;

    float cos_nphi = cosf(nphi);
    float sin_nphi = sinf(nphi);

    ene[dihedral_i] = (temp_pk + cos_nphi * temp_gamc + sin_nphi * temp_gams);
  }
}

void DihedralEnergy(int dihedral_numbers, const int *uint_crd_f, const float *scaler_f, const int *atom_a,
                    const int *atom_b, const int *atom_c, const int *atom_d, const int *ipn, const float *pk,
                    const float *gamc, const float *gams, const float *pn, float *ene, hipStream_t stream) {
  size_t thread_per_block = 128;
  size_t block_per_grid = ceilf(static_cast<float>(dihedral_numbers) / 128);
  UNSIGNED_INT_VECTOR *uint_crd =
    const_cast<UNSIGNED_INT_VECTOR *>(reinterpret_cast<const UNSIGNED_INT_VECTOR *>(uint_crd_f));
  VECTOR *scaler = const_cast<VECTOR *>(reinterpret_cast<const VECTOR *>(scaler_f));

  DihedralEnergyKernel<<<block_per_grid, thread_per_block, 0, stream>>>(
    dihedral_numbers, uint_crd, scaler, atom_a, atom_b, atom_c, atom_d, ipn, pk, gamc, gams, pn, ene);
  return;
}
void DihedralEnergy(int dihedral_numbers, const int *uint_crd_f, const float *scaler_f, const int *atom_a,
                    const int *atom_b, const int *atom_c, const int *atom_d, const int *ipn, const float *pk,
                    const float *gamc, const float *gams, const float *pn, float *ene, hipStream_t stream);
