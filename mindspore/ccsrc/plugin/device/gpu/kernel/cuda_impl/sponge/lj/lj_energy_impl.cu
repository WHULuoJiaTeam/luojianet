#include "hip/hip_runtime.h"
/**
 * Copyright 2021 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "plugin/device/gpu/kernel/cuda_impl/sponge/lj/lj_energy_impl.cuh"
#include "plugin/device/gpu/kernel/cuda_impl/sponge/common_sponge.cuh"

__global__ void LJ_Energy_CUDA(const int atom_numbers, const NEIGHBOR_LIST *nl, const UINT_VECTOR_LJ_TYPE *uint_crd,
                               const VECTOR *boxlength, const float *LJ_type_A, const float *LJ_type_B,
                               const float cutoff_square, float *lj_ene) {
  int atom_i = blockDim.x * blockIdx.x + threadIdx.x;
  if (atom_i < atom_numbers) {
    NEIGHBOR_LIST nl_i = nl[atom_i];
    int N = nl_i.atom_numbers;
    int atom_j;
    int int_x;
    int int_y;
    int int_z;
    UINT_VECTOR_LJ_TYPE r1 = uint_crd[atom_i], r2;
    VECTOR dr;
    float dr2;
    float dr_2;
    float dr_4;
    float dr_6;
    float ene_lin = 0.;

    int x, y;
    int atom_pair_LJ_type;
    for (int j = threadIdx.y; j < N; j = j + blockDim.y) {
      atom_j = nl_i.atom_serial[j];
      r2 = uint_crd[atom_j];

      int_x = r2.uint_x - r1.uint_x;
      int_y = r2.uint_y - r1.uint_y;
      int_z = r2.uint_z - r1.uint_z;
      dr.x = boxlength[0].x * int_x;
      dr.y = boxlength[0].y * int_y;
      dr.z = boxlength[0].z * int_z;

      dr2 = dr.x * dr.x + dr.y * dr.y + dr.z * dr.z;
      if (dr2 < cutoff_square) {
        dr_2 = 1. / dr2;
        dr_4 = dr_2 * dr_2;
        dr_6 = dr_4 * dr_2;

        y = (r2.LJ_type - r1.LJ_type);
        x = y >> 31;
        y = (y ^ x) - x;
        x = r2.LJ_type + r1.LJ_type;
        r2.LJ_type = (x + y) >> 1;
        x = (x - y) >> 1;
        atom_pair_LJ_type = (r2.LJ_type * (r2.LJ_type + 1) >> 1) + x;

        dr_2 = (0.083333333 * LJ_type_A[atom_pair_LJ_type] * dr_6 - 0.166666666 * LJ_type_B[atom_pair_LJ_type]) * dr_6;
        ene_lin = ene_lin + dr_2;
      }
    }
    atomicAdd(&lj_ene[atom_i], ene_lin);
  }
}

void LJEnergy(const int atom_numbers, const float cutoff_square, const int *uint_crd_f, const int *LJtype,
              const float *charge, const float *scaler_f, float *uint_crd_with_LJ, int *nl_atom_numbers,
              int *nl_atom_serial, int *nl, const float *d_LJ_A, const float *d_LJ_B, float *d_LJ_energy_atom,
              hipStream_t stream) {
  VECTOR *scaler = const_cast<VECTOR *>(reinterpret_cast<const VECTOR *>(scaler_f));
  int max_neighbor_numbers = 800;
  NEIGHBOR_LIST *nl_a = reinterpret_cast<NEIGHBOR_LIST *>(nl);
  construct_neighbor_list_kernel<<<ceilf(static_cast<float>(atom_numbers) / 128), 128, 0, stream>>>(
    atom_numbers, max_neighbor_numbers, nl_atom_numbers, nl_atom_serial, nl_a);

  UINT_VECTOR_LJ_TYPE *uint_crd_with_LJ_a = reinterpret_cast<UINT_VECTOR_LJ_TYPE *>(uint_crd_with_LJ);

  UNSIGNED_INT_VECTOR *uint_crd =
    const_cast<UNSIGNED_INT_VECTOR *>(reinterpret_cast<const UNSIGNED_INT_VECTOR *>(uint_crd_f));

  Copy_Crd_To_New_Crd_Start<<<ceilf(static_cast<float>(atom_numbers) / 32), 32, 0, stream>>>(
    atom_numbers, uint_crd, uint_crd_with_LJ_a, LJtype, charge);

  Reset_List<<<ceilf(static_cast<float>(atom_numbers) / 32), 32, 0, stream>>>(atom_numbers, d_LJ_energy_atom, 0.);

  LJ_Energy_CUDA<<<ceilf(static_cast<float>(atom_numbers) / 8), thread_LJ, 0, stream>>>(
    atom_numbers, nl_a, uint_crd_with_LJ_a, scaler, d_LJ_A, d_LJ_B, cutoff_square, d_LJ_energy_atom);

  return;
}
void LJEnergy(const int atom_numbers, const float cutoff_square, const int *uint_crd_f, const int *LJtype,
              const float *charge, const float *scaler_f, float *uint_crd_with_LJ, int *nl_atom_numbers,
              int *nl_atom_serial, int *nl, const float *d_LJ_A, const float *d_LJ_B, float *d_LJ_energy_atom,
              hipStream_t stream);
