#include "hip/hip_runtime.h"
/**
 * Copyright 2021 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "plugin/device/gpu/kernel/cuda_impl/sponge/lj/lj_force_impl.cuh"
#include "plugin/device/gpu/kernel/cuda_impl/sponge/common_sponge.cuh"

__global__ void LJ_Force_CUDA(const int atom_numbers, const NEIGHBOR_LIST *nl, const UINT_VECTOR_LJ_TYPE *uint_crd,
                              const VECTOR *boxlength, const float *LJ_type_A, const float *LJ_type_B,
                              const float cutoff_square, VECTOR *frc) {
  int atom_i = blockDim.x * blockIdx.x + threadIdx.x;
  if (atom_i < atom_numbers) {
    NEIGHBOR_LIST nl_i = nl[atom_i];
    int N = nl_i.atom_numbers;
    int B = ceilf(static_cast<float>(N) / blockDim.y);
    int atom_j;
    int int_x;
    int int_y;
    int int_z;
    UINT_VECTOR_LJ_TYPE r1 = uint_crd[atom_i], r2;
    VECTOR dr;
    float dr2;
    float dr_2;
    float dr_4;
    float dr_8;
    float dr_14;
    float frc_abs = 0.;
    VECTOR frc_lin;
    VECTOR frc_record = {0., 0., 0.};

    int x, y;
    int atom_pair_LJ_type;
    for (int j = threadIdx.y * B; j < (threadIdx.y + 1) * B; j = j + 1) {
      if (j < N) {
        atom_j = nl_i.atom_serial[j];
        r2 = uint_crd[atom_j];
        int_x = r2.uint_x - r1.uint_x;
        int_y = r2.uint_y - r1.uint_y;
        int_z = r2.uint_z - r1.uint_z;
        dr.x = boxlength[0].x * int_x;
        dr.y = boxlength[0].y * int_y;
        dr.z = boxlength[0].z * int_z;
        dr2 = dr.x * dr.x + dr.y * dr.y + dr.z * dr.z;
        if (dr2 < cutoff_square) {
          dr_2 = 1. / dr2;
          dr_4 = dr_2 * dr_2;
          dr_8 = dr_4 * dr_4;
          dr_14 = dr_8 * dr_4 * dr_2;

          y = (r2.LJ_type - r1.LJ_type);
          x = y >> 31;
          y = (y ^ x) - x;
          x = r2.LJ_type + r1.LJ_type;
          r2.LJ_type = (x + y) >> 1;
          x = (x - y) >> 1;
          atom_pair_LJ_type = (r2.LJ_type * (r2.LJ_type + 1) >> 1) + x;

          frc_abs = -LJ_type_A[atom_pair_LJ_type] * dr_14 + LJ_type_B[atom_pair_LJ_type] * dr_8;
          frc_lin.x = frc_abs * dr.x;
          frc_lin.y = frc_abs * dr.y;
          frc_lin.z = frc_abs * dr.z;

          frc_record.x = frc_record.x + frc_lin.x;
          frc_record.y = frc_record.y + frc_lin.y;
          frc_record.z = frc_record.z + frc_lin.z;

          atomicAdd(&frc[atom_j].x, -frc_lin.x);
          atomicAdd(&frc[atom_j].y, -frc_lin.y);
          atomicAdd(&frc[atom_j].z, -frc_lin.z);
        }
      }
    }
    atomicAdd(&frc[atom_i].x, frc_record.x);
    atomicAdd(&frc[atom_i].y, frc_record.y);
    atomicAdd(&frc[atom_i].z, frc_record.z);
  }
}

void LJForce(const int atom_numbers, const float cutoff_square, const int *uint_crd_f, const int *LJtype,
             const float *charge, const float *scaler_f, float *uint_crd_with_LJ, int *nl_atom_numbers,
             int *nl_atom_serial, int *nl, const float *d_LJ_A, const float *d_LJ_B, float *frc_f,
             hipStream_t stream) {
  Reset_List<<<ceilf(static_cast<float>(3. * atom_numbers) / 128), 128, 0, stream>>>(3 * atom_numbers, frc_f, 0.);
  VECTOR *frc = reinterpret_cast<VECTOR *>(frc_f);
  VECTOR *scaler = const_cast<VECTOR *>(reinterpret_cast<const VECTOR *>(scaler_f));
  int max_neighbor_numbers = 800;
  NEIGHBOR_LIST *nl_a = reinterpret_cast<NEIGHBOR_LIST *>(nl);
  construct_neighbor_list_kernel<<<ceilf(static_cast<float>(atom_numbers) / 128), 128, 0, stream>>>(
    atom_numbers, max_neighbor_numbers, nl_atom_numbers, nl_atom_serial, nl_a);

  UINT_VECTOR_LJ_TYPE *uint_crd_with_LJ_a = reinterpret_cast<UINT_VECTOR_LJ_TYPE *>(uint_crd_with_LJ);

  UNSIGNED_INT_VECTOR *uint_crd =
    const_cast<UNSIGNED_INT_VECTOR *>(reinterpret_cast<const UNSIGNED_INT_VECTOR *>(uint_crd_f));

  Copy_Crd_To_New_Crd_Start<<<ceilf(static_cast<float>(atom_numbers) / 32), 32, 0, stream>>>(
    atom_numbers, uint_crd, uint_crd_with_LJ_a, LJtype, charge);

  LJ_Force_CUDA<<<ceilf(static_cast<float>(atom_numbers) / 8), thread_LJ, 0, stream>>>(
    atom_numbers, nl_a, uint_crd_with_LJ_a, scaler, d_LJ_A, d_LJ_B, cutoff_square, frc);
  return;
}
void LJForce(const int atom_numbers, const float cutoff_square, const int *uint_crd_f, const int *LJtype,
             const float *charge, const float *scaler_f, float *uint_crd_with_LJ, int *nl_atom_numbers,
             int *nl_atom_serial, int *nl, const float *d_LJ_A, const float *d_LJ_B, float *frc_f, hipStream_t stream);
