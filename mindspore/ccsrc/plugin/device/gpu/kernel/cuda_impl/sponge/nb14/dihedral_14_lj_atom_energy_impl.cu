#include "hip/hip_runtime.h"
/**
 * Copyright 2021 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "plugin/device/gpu/kernel/cuda_impl/sponge/nb14/dihedral_14_lj_atom_energy_impl.cuh"
#include "plugin/device/gpu/kernel/cuda_impl/sponge/common_sponge.cuh"

__global__ void Dihedral14LJAtomEnergyKernel(const int dihedral_14_numbers, const UINT_VECTOR_LJ_TYPE *uint_crd,
                                             const VECTOR *boxlength, const int *a_14, const int *b_14,
                                             const float *lj_scale_factor, const float *LJ_type_A,
                                             const float *LJ_type_B, float *ene) {
  int dihedral_14_i = blockDim.x * blockIdx.x + threadIdx.x;
  if (dihedral_14_i < dihedral_14_numbers) {
    int atom_i = a_14[dihedral_14_i];
    int atom_j = b_14[dihedral_14_i];

    UINT_VECTOR_LJ_TYPE r1 = uint_crd[atom_i];
    UINT_VECTOR_LJ_TYPE r2 = uint_crd[atom_j];

    int int_x;
    int int_y;
    int int_z;
    VECTOR dr;
    float dr2;
    float dr_2;
    float dr_4;
    float dr_6;
    float dr_12;
    float ene_lin = 0.;
    int x, y;
    int atom_pair_LJ_type;

    int_x = r2.uint_x - r1.uint_x;
    int_y = r2.uint_y - r1.uint_y;
    int_z = r2.uint_z - r1.uint_z;
    dr.x = boxlength[0].x * int_x;
    dr.y = boxlength[0].y * int_y;
    dr.z = boxlength[0].z * int_z;
    dr2 = dr.x * dr.x + dr.y * dr.y + dr.z * dr.z;

    dr_2 = 1. / dr2;
    dr_4 = dr_2 * dr_2;
    dr_6 = dr_4 * dr_2;
    dr_12 = dr_6 * dr_6;

    y = (r2.LJ_type - r1.LJ_type);
    x = y >> 31;
    y = (y ^ x) - x;
    x = r2.LJ_type + r1.LJ_type;
    r2.LJ_type = (x + y) >> 1;
    x = (x - y) >> 1;
    atom_pair_LJ_type = (r2.LJ_type * (r2.LJ_type + 1) >> 1) + x;

    ene_lin = 0.08333333 * LJ_type_A[atom_pair_LJ_type] * dr_12 -
              0.1666666 * LJ_type_B[atom_pair_LJ_type] * dr_6;  // LJ的A,B系数已经乘以12和6因此要反乘
    ene_lin *= lj_scale_factor[dihedral_14_i];

    atomicAdd(&ene[atom_i], ene_lin);
  }
}

void Dihedral14LJAtomEnergy(const int dihedral_14_numbers, const int atom_numbers, const int *uint_crd_f,
                            const int *LJtype, const float *charge, const float *boxlength_f, const int *a_14,
                            const int *b_14, const float *lj_scale_factor, const float *LJ_type_A,
                            const float *LJ_type_B, float *ene, hipStream_t stream) {
  size_t thread_per_block = 128;
  size_t block_per_grid = ceilf(static_cast<float>(atom_numbers) / 128);
  UINT_VECTOR_LJ_TYPE *uint_crd_with_LJ = NULL;
  Cuda_Malloc_Safely(reinterpret_cast<void **>(&uint_crd_with_LJ), sizeof(UINT_VECTOR_LJ_TYPE) * atom_numbers);

  UNSIGNED_INT_VECTOR *uint_crd =
    const_cast<UNSIGNED_INT_VECTOR *>(reinterpret_cast<const UNSIGNED_INT_VECTOR *>(uint_crd_f));

  Copy_Crd_To_New_Crd_Start<<<ceilf(static_cast<float>(atom_numbers) / 32), 32, 0, stream>>>(
    atom_numbers, uint_crd, uint_crd_with_LJ, LJtype, charge);

  VECTOR *boxlength = const_cast<VECTOR *>(reinterpret_cast<const VECTOR *>(boxlength_f));
  Reset_List<<<ceilf(static_cast<float>(3. * atom_numbers) / 128), 128, 0, stream>>>(atom_numbers, ene, 0.);
  Dihedral14LJAtomEnergyKernel<<<block_per_grid, thread_per_block, 0, stream>>>(
    dihedral_14_numbers, uint_crd_with_LJ, boxlength, a_14, b_14, lj_scale_factor, LJ_type_A, LJ_type_B, ene);

  hipStreamSynchronize(stream);

  return;
}

void Dihedral14LJAtomEnergy(const int dihedral_14_numbers, const int atom_numbers, const int *uint_crd_f,
                            const int *LJtype, const float *charge, const float *boxlength_f, const int *a_14,
                            const int *b_14, const float *lj_scale_factor, const float *LJ_type_A,
                            const float *LJ_type_B, float *ene, hipStream_t stream);
