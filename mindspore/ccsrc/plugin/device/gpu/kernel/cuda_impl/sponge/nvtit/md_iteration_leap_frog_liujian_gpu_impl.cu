#include "hip/hip_runtime.h"
/**
 * Copyright 2021 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "plugin/device/gpu/kernel/cuda_impl/sponge/nvtit/md_iteration_leap_frog_liujian_gpu_impl.cuh"
#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/util.cuh"
#include "plugin/device/gpu/kernel/cuda_impl/sponge/common_sponge.cuh"

__global__ void MD_Iteration_Leap_Frog_With_LiuJian_kernel(const int atom_numbers, const float half_dt, const float dt,
                                                           const float exp_gamma, float *inverse_mass,
                                                           float *sqrt_mass_inverse, VECTOR *vel, VECTOR *crd,
                                                           VECTOR *frc, VECTOR *acc, VECTOR *random_frc,
                                                           VECTOR *output) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i < atom_numbers) {
    acc[i].x = inverse_mass[i] * frc[i].x;
    acc[i].y = inverse_mass[i] * frc[i].y;
    acc[i].z = inverse_mass[i] * frc[i].z;

    vel[i].x = vel[i].x + dt * acc[i].x;
    vel[i].y = vel[i].y + dt * acc[i].y;
    vel[i].z = vel[i].z + dt * acc[i].z;

    output[i].x = crd[i].x + half_dt * vel[i].x;
    output[i].y = crd[i].y + half_dt * vel[i].y;
    output[i].z = crd[i].z + half_dt * vel[i].z;

    vel[i].x = exp_gamma * vel[i].x + sqrt_mass_inverse[i] * random_frc[i].x;
    vel[i].y = exp_gamma * vel[i].y + sqrt_mass_inverse[i] * random_frc[i].y;
    vel[i].z = exp_gamma * vel[i].z + sqrt_mass_inverse[i] * random_frc[i].z;

    output[i].x = output[i].x + half_dt * vel[i].x;
    output[i].y = output[i].y + half_dt * vel[i].y;
    output[i].z = output[i].z + half_dt * vel[i].z;
  }
}

void MD_Iteration_Leap_Frog_With_LiuJian(const int atom_numbers, const float half_dt, const float dt,
                                         const float exp_gamma, int float4_numbers, float *inverse_mass,
                                         float *sqrt_mass_inverse, float *vel, float *crd, float *frc, float *acc,
                                         hiprandStatePhilox4_32_10_t *rand_state, float *rand_frc, float *output,
                                         hipStream_t stream) {
  Rand_Normal<<<ceilf(static_cast<float>(float4_numbers) / 32.), 32, 0, stream>>>(float4_numbers, rand_state,
                                                                                  reinterpret_cast<float4 *>(rand_frc));
  VECTOR *d_vel = reinterpret_cast<VECTOR *>(vel);
  VECTOR *d_crd = reinterpret_cast<VECTOR *>(crd);
  VECTOR *d_frc = reinterpret_cast<VECTOR *>(frc);
  VECTOR *d_acc = reinterpret_cast<VECTOR *>(acc);
  VECTOR *d_rand_frc = reinterpret_cast<VECTOR *>(rand_frc);
  VECTOR *d_out = reinterpret_cast<VECTOR *>(output);
  MD_Iteration_Leap_Frog_With_LiuJian_kernel<<<ceilf(static_cast<float>(atom_numbers) / 32), 32, 0, stream>>>(
    atom_numbers, half_dt, dt, exp_gamma, inverse_mass, sqrt_mass_inverse, d_vel, d_crd, d_frc, d_acc, d_rand_frc,
    d_out);
}
