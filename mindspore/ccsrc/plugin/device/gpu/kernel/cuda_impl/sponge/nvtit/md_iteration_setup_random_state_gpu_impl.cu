#include "hip/hip_runtime.h"
/**
 * Copyright 2021 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "plugin/device/gpu/kernel/cuda_impl/sponge/nvtit/md_iteration_setup_random_state_gpu_impl.cuh"
#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/util.cuh"
#include "plugin/device/gpu/kernel/cuda_impl/sponge/common_sponge.cuh"

void MD_Iteration_Setup_Random_State(int float4_numbers, hiprandStatePhilox4_32_10_t *rand_state, int seed,
                                     hipStream_t stream) {
  Setup_Rand_Normal_Kernel<<<ceilf(static_cast<float>(float4_numbers) / 32.), 32, 0, stream>>>(float4_numbers,
                                                                                               rand_state, seed);
}

void MD_Iteration_Setup_Random_State(int float4_numbers, hiprandStatePhilox4_32_10_t *rand_state, int seed,
                                     hipStream_t stream);
