/**
 * Copyright 2021 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "plugin/device/gpu/kernel/cuda_impl/sponge/pme/fft_3d_impl.cuh"
#include "plugin/device/gpu/kernel/cuda_impl/sponge/pme/pme_common.cuh"

template <typename T>
void FFT3D(int Nfft, T *input_tensor, Complex<T> *output_tensor, const hipfftHandle &FFT_plan_r2c, hipStream_t stream) {
  hipfftExecR2C(FFT_plan_r2c, input_tensor, reinterpret_cast<hipfftComplex *>(output_tensor));
  return;
}

template void FFT3D<float>(int Nfft, float *input_tensor, Complex<float> *output_tensor,
                           const hipfftHandle &FFT_plan_r2c, hipStream_t stream);
