#include "hip/hip_runtime.h"
/**
 * Copyright 2021 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
/**
 * Note:
 *  PMEEnergyUpdate. This is an experimental interface that is subject to change and/or deletion.
 */
#include "plugin/device/gpu/kernel/cuda_impl/sponge/pme/pme_energy_update_impl.cuh"
#include "plugin/device/gpu/kernel/cuda_impl/sponge/pme/pme_common.cuh"
#include "plugin/device/gpu/kernel/cuda_impl/sponge/common_sponge.cuh"

__global__ void PME_Energy_Reciprocal_update(const int element_number, const hipfftComplex *FQ, const float *BC,
                                             float *sum) {
  if (threadIdx.x == 0) {
    sum[0] = 0.;
  }
  __syncthreads();
  float lin = 0.0;
  hipfftComplex FQ_i;
  for (int i = threadIdx.x; i < element_number; i = i + blockDim.x) {
    FQ_i = FQ[i];
    lin = lin + (FQ_i.x * FQ_i.x + FQ_i.y * FQ_i.y) * BC[i];
  }
  atomicAdd(sum, lin);
}

void PMEEnergyUpdate(int fftx, int ffty, int fftz, int atom_numbers, float beta, float *PME_BC, int *pme_uxyz,
                     float *pme_frxyz, float *PME_Q, float *pme_fq, int *PME_atom_near, int *pme_kxyz,
                     const int *uint_crd_f, const float *charge, int *nl_atom_numbers, int *nl_atom_serial, int *nl,
                     const float *scaler_f, const int *excluded_list_start, const int *excluded_list,
                     const int *excluded_atom_numbers, float *d_reciprocal_ene, float *d_self_ene, float *d_direct_ene,
                     float *d_correction_ene, dim3 thread_PME, int PME_Nin, int PME_Nfft, int PME_Nall,
                     const hipfftHandle &PME_plan_r2c, const hipfftHandle &PME_plan_c2r, float *neutralizing_factor,
                     float *charge_sum, int max_neighbor_numbers, hipStream_t stream) {
  UNSIGNED_INT_VECTOR *uint_crd =
    const_cast<UNSIGNED_INT_VECTOR *>(reinterpret_cast<const UNSIGNED_INT_VECTOR *>(uint_crd_f));
  VECTOR *scaler = const_cast<VECTOR *>(reinterpret_cast<const VECTOR *>(scaler_f));
  // int max_neighbor_numbers = 800;
  NEIGHBOR_LIST *nl_a = reinterpret_cast<NEIGHBOR_LIST *>(nl);
  construct_neighbor_list_kernel<<<ceilf(static_cast<float>(atom_numbers) / 128), 128, 0, stream>>>(
    atom_numbers, max_neighbor_numbers, nl_atom_numbers, nl_atom_serial, nl_a);

  UNSIGNED_INT_VECTOR *PME_uxyz = reinterpret_cast<UNSIGNED_INT_VECTOR *>(pme_uxyz);
  UNSIGNED_INT_VECTOR *PME_kxyz = reinterpret_cast<UNSIGNED_INT_VECTOR *>(pme_kxyz);
  VECTOR *PME_frxyz = reinterpret_cast<VECTOR *>(pme_frxyz);
  hipfftComplex *PME_FQ = reinterpret_cast<hipfftComplex *>(pme_fq);

  Reset_List<<<3 * atom_numbers / 32 + 1, 32, 0, stream>>>(3 * atom_numbers, reinterpret_cast<int *>(PME_uxyz),
                                                           1 << 30);
  PME_Atom_Near<<<atom_numbers / 32 + 1, 32, 0, stream>>>(
    uint_crd, PME_atom_near, PME_Nin, periodic_factor_inverse * fftx, periodic_factor_inverse * ffty,
    periodic_factor_inverse * fftz, atom_numbers, fftx, ffty, fftz, PME_kxyz, PME_uxyz, PME_frxyz);

  Reset_List<<<PME_Nall / 1024 + 1, 1024, 0, stream>>>(PME_Nall, PME_Q, 0);

  PME_Q_Spread<<<atom_numbers / thread_PME.x + 1, thread_PME, 0, stream>>>(PME_atom_near, charge, PME_frxyz, PME_Q,
                                                                           PME_kxyz, atom_numbers);

  hipfftExecR2C(PME_plan_r2c, reinterpret_cast<float *>(PME_Q), reinterpret_cast<hipfftComplex *>(PME_FQ));

  PME_Energy_Reciprocal_update<<<1, 1024, 0, stream>>>(PME_Nfft, PME_FQ, PME_BC, d_reciprocal_ene);

  PME_Energy_Product<<<1, 1024, 0, stream>>>(atom_numbers, charge, charge, d_self_ene);
  Scale_List<<<1, 1, 0, stream>>>(1, d_self_ene, -beta / sqrtf(PI));

  Sum_Of_List<<<1, 1024>>>(atom_numbers, charge, charge_sum);
  device_add<<<1, 1>>>(d_self_ene, neutralizing_factor, charge_sum);

  Reset_List<<<1, 1, 0, stream>>>(1, d_direct_ene, 0.0);
  PME_Direct_Energy<<<atom_numbers / thread_PME.x + 1, thread_PME, 0, stream>>>(
    atom_numbers, nl_a, uint_crd, scaler, charge, beta, cutoff * cutoff, d_direct_ene);

  Reset_List<<<1, 1, 0, stream>>>(1, d_correction_ene, 0.0);
  PME_Excluded_Energy_Correction<<<atom_numbers / 32 + 1, 32, 0, stream>>>(
    atom_numbers, uint_crd, scaler, charge, beta, sqrtf(PI), excluded_list_start, excluded_list, excluded_atom_numbers,
    d_correction_ene);
  return;
}
