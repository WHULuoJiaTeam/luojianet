#include "hip/hip_runtime.h"
/**
 * Copyright 2021 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
/**
 * Note:
 *  Constrain_Force_Cycle. This is an experimental interface that is subject to change and/or deletion.
 */

#include "plugin/device/gpu/kernel/cuda_impl/sponge/common_sponge.cuh"
#include "plugin/device/gpu/kernel/cuda_impl/sponge/simple_constrain/constrain_force_cycle_impl.cuh"

__global__ void Constrain_Force_Cycle_Kernel(const int constrain_pair_numbers, const UNSIGNED_INT_VECTOR *uint_crd,
                                             const VECTOR *scaler, const CONSTRAIN_PAIR *constrain_pair,
                                             const VECTOR *pair_dr, VECTOR *test_frc) {
  int pair_i = blockDim.x * blockIdx.x + threadIdx.x;
  if (pair_i < constrain_pair_numbers) {
    CONSTRAIN_PAIR cp = constrain_pair[pair_i];
    float r_1;
    VECTOR dr;
    float frc_abs;
    VECTOR frc_lin;

    dr.x = (static_cast<int>(uint_crd[cp.atom_i_serial].uint_x - uint_crd[cp.atom_j_serial].uint_x)) * scaler[0].x;
    dr.y = (static_cast<int>(uint_crd[cp.atom_i_serial].uint_y - uint_crd[cp.atom_j_serial].uint_y)) * scaler[0].y;
    dr.z = (static_cast<int>(uint_crd[cp.atom_i_serial].uint_z - uint_crd[cp.atom_j_serial].uint_z)) * scaler[0].z;
    r_1 = rnorm3df(dr.x, dr.y, dr.z);
    frc_abs = (1. - cp.constant_r * r_1) * cp.constrain_k;

    frc_lin.x = frc_abs * pair_dr[pair_i].x;
    frc_lin.y = frc_abs * pair_dr[pair_i].y;
    frc_lin.z = frc_abs * pair_dr[pair_i].z;

    atomicAdd(&test_frc[cp.atom_j_serial].x, frc_lin.x);
    atomicAdd(&test_frc[cp.atom_j_serial].y, frc_lin.y);
    atomicAdd(&test_frc[cp.atom_j_serial].z, frc_lin.z);

    atomicAdd(&test_frc[cp.atom_i_serial].x, -frc_lin.x);
    atomicAdd(&test_frc[cp.atom_i_serial].y, -frc_lin.y);
    atomicAdd(&test_frc[cp.atom_i_serial].z, -frc_lin.z);
  }
}

void Constrain_Force_Cycle(int atom_numbers, int constrain_pair_numbers, const unsigned int *uint_crd_f,
                           const float *scaler_f, float *constrain_pair_f, const float *pair_dr_f,
                           const int *atom_i_serials, const int *atom_j_serials, const float *constant_rs,
                           const float *constrain_ks, float *test_frc_f, hipStream_t stream) {
  Reset_List<<<ceilf(static_cast<float>(3 * atom_numbers) / 128), 128, 0, stream>>>(3 * atom_numbers, test_frc_f, 0.);
  size_t thread_per_block = 128;
  size_t block_per_grid = ceilf(static_cast<float>(atom_numbers) / 128);
  const UNSIGNED_INT_VECTOR *uint_crd = reinterpret_cast<const UNSIGNED_INT_VECTOR *>(uint_crd_f);
  const VECTOR *scaler = reinterpret_cast<const VECTOR *>(scaler_f);
  const VECTOR *pair_dr = reinterpret_cast<const VECTOR *>(pair_dr_f);

  VECTOR *test_frc = reinterpret_cast<VECTOR *>(test_frc_f);

  CONSTRAIN_PAIR *constrain_pair = reinterpret_cast<CONSTRAIN_PAIR *>(constrain_pair_f);

  construct_constrain_pair<<<ceilf(static_cast<float>(constrain_pair_numbers) / 128), 128, 0, stream>>>(
      constrain_pair_numbers, atom_i_serials, atom_j_serials, constant_rs, constrain_ks, constrain_pair);

  Constrain_Force_Cycle_Kernel<<<block_per_grid, thread_per_block, 0, stream>>>(
      constrain_pair_numbers, uint_crd, scaler, constrain_pair, pair_dr, test_frc);

  return;
}

void Constrain_Force_Cycle(int atom_numbers, int constrain_pair_numbers, const unsigned int *uint_crd_f,
                           const float *scaler_f, float *constrain_pair_f, const float *pair_dr_f,
                           const int *atom_i_serials, const int *atom_j_serials, const float *constant_rs,
                           const float *constrain_ks, float *test_frc_f, hipStream_t stream);
