#include "hip/hip_runtime.h"
/**
 * Copyright 2021 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
/**
 * Note:
 *  ConstrainForceCycleVirial. This is an experimental interface that is subject to change and/or deletion.
 */

#include "plugin/device/gpu/kernel/cuda_impl/sponge/common_sponge.cuh"
#include "plugin/device/gpu/kernel/cuda_impl/sponge/simple_constrain/constrain_force_cycle_with_virial_impl.cuh"

__global__ void Constrain_Force_Cycle_With_Virial(int constrain_pair_numbers, const UNSIGNED_INT_VECTOR *uint_crd,
                                                  const VECTOR *scaler, CONSTRAIN_PAIR *constrain_pair,
                                                  const VECTOR *pair_dr, VECTOR *test_frc, float *d_atom_virial) {
  int pair_i = blockDim.x * blockIdx.x + threadIdx.x;
  if (pair_i < constrain_pair_numbers) {
    CONSTRAIN_PAIR cp = constrain_pair[pair_i];
    VECTOR dr0 = pair_dr[pair_i];
    VECTOR dr = Get_Periodic_Displacement(uint_crd[cp.atom_i_serial], uint_crd[cp.atom_j_serial], scaler[0]);
    float r_1 = rnorm3df(dr.x, dr.y, dr.z);
    float frc_abs = (1. - cp.constant_r * r_1) * cp.constrain_k;
    VECTOR frc_lin = frc_abs * dr0;
    d_atom_virial[pair_i] -= frc_lin * dr0;

    atomicAdd(&test_frc[cp.atom_j_serial].x, frc_lin.x);
    atomicAdd(&test_frc[cp.atom_j_serial].y, frc_lin.y);
    atomicAdd(&test_frc[cp.atom_j_serial].z, frc_lin.z);

    atomicAdd(&test_frc[cp.atom_i_serial].x, -frc_lin.x);
    atomicAdd(&test_frc[cp.atom_i_serial].y, -frc_lin.y);
    atomicAdd(&test_frc[cp.atom_i_serial].z, -frc_lin.z);
  }
}

void Constrain_Force_Cycle_With_Virial(int atom_numbers, int constrain_pair_numbers, const unsigned int *uint_crd_f,
                                       const float *scaler_f, float *constrain_pair_f, const float *pair_dr_f,
                                       const int *atom_i_serials, const int *atom_j_serials, const float *constant_rs,
                                       const float *constrain_ks, float *test_frc_f, float *d_atom_virial,
                                       hipStream_t stream) {
  Reset_List<<<ceilf(static_cast<float>(3 * atom_numbers) / 128), 128, 0, stream>>>(3 * atom_numbers, test_frc_f, 0.);
  Reset_List<<<ceilf(static_cast<float>(constrain_pair_numbers) / 128), 128, 0, stream>>>(constrain_pair_numbers,
                                                                                          d_atom_virial, 0.);
  size_t thread_per_block = 128;
  size_t block_per_grid = ceilf(static_cast<float>(atom_numbers) / 128);
  const UNSIGNED_INT_VECTOR *uint_crd = reinterpret_cast<const UNSIGNED_INT_VECTOR *>(uint_crd_f);
  const VECTOR *scaler = reinterpret_cast<const VECTOR *>(scaler_f);
  const VECTOR *pair_dr = reinterpret_cast<const VECTOR *>(pair_dr_f);

  VECTOR *test_frc = reinterpret_cast<VECTOR *>(test_frc_f);

  CONSTRAIN_PAIR *constrain_pair = reinterpret_cast<CONSTRAIN_PAIR *>(constrain_pair_f);

  construct_constrain_pair<<<ceilf(static_cast<float>(constrain_pair_numbers) / 128), 128, 0, stream>>>(
      constrain_pair_numbers, atom_i_serials, atom_j_serials, constant_rs, constrain_ks, constrain_pair);

  Constrain_Force_Cycle_With_Virial<<<block_per_grid, thread_per_block, 0, stream>>>(
      constrain_pair_numbers, uint_crd, scaler, constrain_pair, pair_dr, test_frc, d_atom_virial);

  return;
}

void Constrain_Force_Cycle_With_Virial(int atom_numbers, int constrain_pair_numbers, const unsigned int *uint_crd_f,
                                       const float *scaler_f, float *constrain_pair_f, const float *pair_dr_f,
                                       const int *atom_i_serials, const int *atom_j_serials, const float *constant_rs,
                                       const float *constrain_ks, float *test_frc_f, float *d_atom_virial,
                                       hipStream_t stream);
