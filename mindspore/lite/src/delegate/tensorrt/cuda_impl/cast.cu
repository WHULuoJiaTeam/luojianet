#include "hip/hip_runtime.h"
/**
 * Copyright 2020 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "src/delegate/tensorrt/cuda_impl/cast.cuh"
#include "src/delegate/tensorrt/cuda_impl/cuda_helper.h"

// Generic cast
template <typename S, typename T>
__device__ __forceinline__ void CastBase(const S *input_addr, T *output_addr) {
  *output_addr = static_cast<T>((*input_addr));
}

template <typename S, typename T>
__global__ void CastKernel(const int input_size, const S *input_addr, T *output_addr) {
  for (size_t pos = blockIdx.x * blockDim.x + threadIdx.x; pos < input_size; pos += blockDim.x * gridDim.x) {
    CastBase(input_addr + pos, output_addr + pos);
  }
}

template <typename S, typename T>
void Cast(const int input_size, const S *input_addr, T *output_addr, hipStream_t stream) {
  CastKernel<<<GET_BLOCKS(input_size), GET_THREADS, 0, stream>>>(input_size, input_addr, output_addr);
}

template void Cast(const int input_size, const int8_t *input_addr, int8_t *output_addr, hipStream_t stream);
template void Cast(const int input_size, const int8_t *input_addr, int32_t *output_addr, hipStream_t stream);
template void Cast(const int input_size, const int8_t *input_addr, float *output_addr, hipStream_t stream);

template void Cast(const int input_size, const int32_t *input_addr, int8_t *output_addr, hipStream_t stream);
template void Cast(const int input_size, const int32_t *input_addr, int32_t *output_addr, hipStream_t stream);
template void Cast(const int input_size, const int32_t *input_addr, float *output_addr, hipStream_t stream);

template void Cast(const int input_size, const float *input_addr, int8_t *output_addr, hipStream_t stream);
template void Cast(const int input_size, const float *input_addr, int32_t *output_addr, hipStream_t stream);
template void Cast(const int input_size, const float *input_addr, float *output_addr, hipStream_t stream);
